#include "hip/hip_runtime.h"
#include "LSM.cuh"

#include "algorithms/binary_search.cuh"
#include "algorithms/for_each.cuh"
#include "algorithms/merge.cuh"
#include "algorithms/move.cuh"
#include "algorithms/set.cuh"
#include "utility/limits.cuh"
#include "utility/print.cuh"

#include "cub/block/block_radix_sort.cuh"

namespace gpu
{
	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::iterator lsm<Key, Value, N>::end()
	{
		return iterator(nullptr);
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::const_iterator lsm<Key, Value, N>::end() const
	{
		return const_iterator(nullptr);
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::const_iterator lsm<Key, Value, N>::cend() const
	{
		return const_iterator(nullptr);
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ lsm<Key, Value, N>::lsm(block_threads group, allocator_type& allocator, size_type expected_number_of_elements) :
		m_number_of_batches{ 0u },
		m_storage{},
		m_current_buffer{ false }
	{
		m_storage = allocator.allocate<value_type>(group, expected_number_of_elements);
		m_buffers[0] = allocator.allocate<value_type>(group, expected_number_of_elements / 2u);
		m_buffers[1] = allocator.allocate<value_type>(group, expected_number_of_elements / 2u);
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ lsm<Key, Value, N>::lsm(threads group, allocator_type& allocator, size_type expected_number_of_elements) :
		m_number_of_batches{ 0u },
		m_storage{},
		m_current_buffer{false}
	{
		m_storage = allocator.allocate<value_type>(group, expected_number_of_elements);
		m_buffers[0] = allocator.allocate<value_type>(group, expected_number_of_elements / 2u);
		m_buffers[1] = allocator.allocate<value_type>(group, expected_number_of_elements / 2u);
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ void lsm<Key, Value, N>::clear(block_threads group)
	{
		if (group.thread_rank() == 0)
			m_number_of_batches = 0u;
		group.sync();
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::iterator lsm<Key, Value, N>::find(const key_type& key)
	{
		for (unsigned int i = 0u; i != number_of_levels(); ++i)
		{
			if (m_number_of_batches & (1 << i))
			{
				pointer end_level = level(i + 1);
				pointer result = lower_bound(level(i), end_level, key, [](const value_type& lhs, const key_type& rhs) {
					return lhs.first < rhs;
				});
				if (result != end_level && result->first == key)
					return result;
			}
		}
		return end();
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::const_iterator lsm<Key, Value, N>::find(const key_type& key) const
	{
		for (unsigned int i = 0u; i != number_of_levels(); ++i)
		{
			if (m_number_of_batches & (1 << i))
			{
				const_pointer end_level = level(i + 1);
				const_pointer result = lower_bound(level(i), end_level, key, [](const value_type& lhs, const key_type& rhs) {
					return lhs.first < rhs;
				});
				if (result != end_level && result->first == key)
					return result;
			}
		}
		return end();
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::iterator lsm<Key, Value, N>::find(threads g, const key_type& key)
	{
		iterator it;
		if (g.thread_rank() == 0)
			it = find(key);
		return reinterpret_cast<pointer>(g.shfl(reinterpret_cast<std::uintptr_t>(it), 0));
	}

		template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::const_iterator lsm<Key, Value, N>::find(threads g, const key_type& key) const
	{
		const_iterator it;
		if (g.thread_rank() == 0)
			it = find(key);
		return reinterpret_cast<pointer>(g.shfl(reinterpret_cast<std::uintptr_t>(it), 0));
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ void lsm<Key, Value, N>::insert(block_threads group, value_type value)
	{
		sort(group, value);
		unsigned int i = 0u;
		unsigned int offset = 0u;
		unsigned int number_of_elements_at_level_i = N;

		while (is_level_full(i))
		{
			merge(group, offset, number_of_elements_at_level_i);
			full_empty(group, offset, number_of_elements_at_level_i);

			++i;
			offset += number_of_elements_at_level_i;
			number_of_elements_at_level_i <<= 1u;
		}

		gpu::move(group, current_buffer().begin(), current_buffer().begin() + number_of_elements_at_level_i, m_storage.begin() + offset);

		++m_number_of_batches;
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::size_type lsm<Key, Value, N>::number_of_batches() const
	{
		return m_number_of_batches;
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::iterator lsm<Key, Value, N>::predecessor(const key_type& key)
	{
		iterator previous_max = end();
		for (unsigned int i = 0u; i != number_of_levels(); ++i)
		{
			if (m_number_of_batches & (1 << i))
			{
				pointer start_level = level(i);
				pointer end_level = level(i + 1);
				pointer result = upper_bound(start_level, end_level, key, [](const key_type& lhs, const value_type& rhs) -> bool {
					return lhs < rhs.first;
				});
				if (result != start_level)
					--result;
				if (result->first <= key)
				{
					if (previous_max == end())
						previous_max = result;
					else if (result->first > previous_max->first)
						previous_max = result;
				}
			}
		}
		return previous_max;
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::iterator lsm<Key, Value, N>::predecessor(threads g, const key_type& key)
	{
		iterator it = end();
		if (g.thread_rank() == 0)
			it = predecessor(key);
		it = reinterpret_cast<iterator>(g.shfl(reinterpret_cast<std::uintptr_t>(it), 0));
		return it;
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::size_type lsm<Key, Value, N>::size() const
	{
		return m_number_of_batches * N;
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::iterator lsm<Key, Value, N>::successor(const key_type& key)
	{
		iterator successor_min = end();
		for (unsigned int i = 0u; i != number_of_levels(); ++i)
		{
			if (m_number_of_batches & (1 << i))
			{
				pointer end_level = level(i + 1);
				pointer result = lower_bound(level(i), end_level, key, [](const value_type& lhs, const key_type& rhs) -> bool {
					return lhs.first < rhs;
				});
				if (result != end_level)
				{
					if (successor_min == end())
						successor_min = result;
					else if (result->first < successor_min->first)
						successor_min = result;
				}
			}
		}
		return successor_min;
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::iterator lsm<Key, Value, N>::successor(threads g, const key_type& key)
	{
		return successor(key);
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ void lsm<Key, Value, N>::debug() const
	{
		print("Number of elements (", m_number_of_batches * N, ")\n");

		for (unsigned int i = 0u; i != number_of_levels(); ++i)
		{
			if (m_number_of_batches & (1 << i))
			{
				print("Dictonary number (", i, "): ");
				for (auto it = level(i); it != level(i + 1); ++it)
					print(" { ", it->first, ": ", it->second, " }, ");
				print("\n");
			}
		}
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::internal_storage& lsm<Key, Value, N>::current_buffer()
	{
		return m_buffers[m_current_buffer];
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ const typename lsm<Key, Value, N>::internal_storage& lsm<Key, Value, N>::current_buffer() const
	{
		return m_buffers[m_current_buffer];
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::key_type lsm<Key, Value, N>::FREE()
	{
		return numeric_limits<key_type>::max();
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ void lsm<Key, Value, N>::full_empty(block_threads group, unsigned int offset, unsigned int number_of_elements_at_level_i)
	{
		gpu::for_each(group, m_storage.begin() + offset, m_storage.begin() + offset + number_of_elements_at_level_i, [](value_type& value) {
			value.first = FREE();
		});
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ bool lsm<Key, Value, N>::is_level_full(unsigned int level) const
	{
		return m_number_of_batches & (1 << level);
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ unsigned int lsm<Key, Value, N>::number_of_levels() const
	{
		return 32 - __clz(m_number_of_batches);
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::pointer lsm<Key, Value, N>::level(unsigned int level)
	{
		return m_storage.begin() + N * ((1 << level) - 1u);
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ typename lsm<Key, Value, N>::const_pointer lsm<Key, Value, N>::level(unsigned int level) const
	{
		return m_storage.begin() + N * ((1 << level) - 1u);
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ void lsm<Key, Value, N>::merge(block_threads group, unsigned int offset, unsigned int number_of_elements_at_level_i)
	{
		auto& other_buffer = m_buffers[!m_current_buffer];
		auto start_next_buffer = other_buffer.begin();

		auto start_first_it = m_storage.begin() + offset;
		auto start_second_it = current_buffer().begin();
		gpu::merge(group, start_first_it, start_first_it + number_of_elements_at_level_i,
		      start_second_it, start_second_it + number_of_elements_at_level_i,
		      other_buffer.begin(),
		      [](const value_type& x, const value_type& y) {
		          return (x.first >> 1u) < (y.first >> 1u);
		});
		group.sync();

		if (group.thread_rank() == 0)
			m_current_buffer = !m_current_buffer;
	}

	template <typename Key, typename Value, unsigned int N>
	__device__ void lsm<Key, Value, N>::sort(block_threads group, value_type value)
	{
		// Specialize BlockRadixSort for a 1D block of N threads owning 1 integer items each
		typedef hipcub::BlockRadixSort<key_type, N, 1, mapped_type> BlockRadixSort;
		// Allocate shared memory for BlockRadixSort
		__shared__ typename BlockRadixSort::TempStorage temp_storage;

		key_type thread_key[1] = { std::move(value.first) };
		mapped_type thread_value[1] = { std::move(value.second) };

		// Collectively sort the keys
		BlockRadixSort(temp_storage).Sort(thread_key, thread_value);

		auto thid = group.thread_rank();
		current_buffer()[thid].first = std::move(thread_key[0]);
		current_buffer()[thid].second = std::move(thread_value[0]);
	}
}
