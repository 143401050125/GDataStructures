#include "hip/hip_runtime.h"
#include "LSM.cuh"
#include "Catch2/catch.hpp"
#include "cuda/api_wrappers.h"

#include "concurrent/allocators/default_allocator.cuh"
#include "containers/hash_tables/default_hash_function.cuh"
#include <hip/hip_cooperative_groups.h>

namespace
{
	using key_type = int;
	using mapped_type = int;
	constexpr unsigned int NUMBER_OF_WARPS = 1u;
	constexpr unsigned int NUMBER_OF_THREADS_PER_WARP = 32u;
	using LSM = gpu::lsm<key_type, mapped_type, NUMBER_OF_WARPS * NUMBER_OF_THREADS_PER_WARP>;
	using allocator_type = gpu::concurrent::default_allocator;

	inline __device__ void lsm_ensure_value(LSM* lsm, typename LSM::iterator it, int expected_value)
	{
		ENSURE(!(it == lsm->end()));
		ENSURE(it->second == expected_value);
	}

	__global__ void lsm_initialize_allocator_small(allocator_type* allocator, char* memory, int memory_size, LSM* lsm, unsigned int maximal_size)
	{
		cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
		if (block.thread_rank() == 0)
			new (allocator) allocator_type(memory, memory_size);
		block.sync();
		new (lsm) LSM(block, *allocator, maximal_size);
	}

	__global__ void lsm_add_increasing_order(LSM* lsm)
	{
		auto block = cooperative_groups::this_thread_block();
		auto thid = block.thread_rank();

		unsigned int offset = 0u;
		for (int i = 0; i != 1 << 2; ++i)
		{
			lsm->insert(block, gpu::make_pair<key_type, mapped_type>(offset + thid, thid));

			/*block.sync();
			if (block.thread_rank() == 0)
				lsm->debug();
			block.sync();*/

			auto it = lsm->find(offset + thid);
			ENSURE(it->second == thid);

			offset += block.size();
		}

		/*block.sync();
		if (block.thread_rank() == 0)
			lsm->debug();*/
		/*
		if (thid == 0)
			lsm->debug();

		auto it = lsm->find(201);
		ENSURE(it == lsm->end());

		auto pred = lsm->predecessor(offset);
		ENSURE(pred != lsm->end() && pred->first == offset - 1);

		pred = lsm->predecessor(1);
		ENSURE(pred != lsm->end() && pred->first == 1);

		auto succ = lsm->successor(1);
		ENSURE(succ != lsm->end() && succ->first == 1);

		succ = lsm->successor(offset);
		ENSURE(succ == lsm->end());
		*/
	}

	__global__ void lsm_add_decreasing_order(LSM* lsm)
	{
		auto block = cooperative_groups::this_thread_block();
		auto thid = block.thread_rank();

		unsigned int to_insert = 6u;
		unsigned int offset = (to_insert - 1u) * NUMBER_OF_WARPS * 32u;
		for (int i = (to_insert - 1u); i != -1; --i)
		{
			lsm->insert(block, gpu::make_pair<key_type, mapped_type>(offset + thid, thid));

			auto it = lsm->find(offset + thid);
			ENSURE(it->second == thid);

			offset -= block.size();
		}

		auto it = lsm->find(201);
		ENSURE(it == lsm->end());
	}

	__global__ void lsm_add_with_duplicates(LSM* lsm)
	{
		auto block = cooperative_groups::this_thread_block();
		auto thid = block.thread_rank();

		unsigned int offset = 0u;
		for (int i = 0; i != 6; ++i)
		{
			key_type key = offset + thid;
			key = (key % 32 == 0) ? 0 : key;
			lsm->insert(block, gpu::make_pair<key_type, mapped_type>(key, offset + thid));

			if (thid == 0)
				lsm->debug();
			block.sync();

			auto it = lsm->find(key);
			ENSURE(it->second == offset + thid);

			offset += block.size();
		}
	}

	__global__ void lsm_add_random_order(LSM* lsm)
	{
		auto block = cooperative_groups::this_thread_block();
		auto thid = block.thread_rank();

		unsigned int offset = 0u;
		for (int i = 0; i != 6; ++i)
		{
			int hashed_i = int(gpu::hash<int>{}(offset + thid));
			lsm->insert(block, gpu::make_pair<key_type, mapped_type>(hashed_i, thid));

			auto it = lsm->find(offset + thid);
			ENSURE(it->second == thid);

			offset += block.size();
		}
	}

	/*
	__global__ void lsm_test_predecessor_successor(LSM* lsm)
	{
		threads warp = cooperative_groups::tiled_partition<32>(cooperative_groups::this_thread_block());

		ENSURE(lsm->predecessor(warp, 128) == lsm->end());
		ENSURE(lsm->successor(warp, 128) == lsm->end());

		lsm->insert(warp, 2, 2);

		lsm_ensure_value(lsm, lsm->predecessor(warp, 128), 2);
		ENSURE(lsm->successor(warp, 128) == lsm->end());
		ENSURE(lsm->predecessor(warp, 1) == lsm->end());
		lsm_ensure_value(lsm, lsm->successor(warp, 1), 2);

		lsm->insert(warp, 13, 13);
		lsm_ensure_value(lsm, lsm->predecessor(warp, 128), 13);
		lsm_ensure_value(lsm, lsm->predecessor(warp, 13), 13);
		lsm_ensure_value(lsm, lsm->predecessor(warp, 12), 2);
		lsm_ensure_value(lsm, lsm->successor(warp, 3), 13);

		lsm->insert(warp, 251, 251);
		lsm_ensure_value(lsm, lsm->predecessor(warp, 128), 13);
		lsm_ensure_value(lsm, lsm->predecessor(warp, 253), 251);
		lsm_ensure_value(lsm, lsm->successor(warp, 128), 251);
		ENSURE(lsm->successor(warp, 252) == lsm->end());

		lsm->insert(warp, 190, 190);
		lsm_ensure_value(lsm, lsm->successor(warp, 191), 251);
		lsm_ensure_value(lsm, lsm->successor(warp, 190), 190);
		lsm_ensure_value(lsm, lsm->predecessor(warp, 189), 13);
		lsm_ensure_value(lsm, lsm->predecessor(warp, 250), 190);

		lsm->insert(warp, 17, 17);
		lsm->insert(warp, 35, 35);
		lsm->insert(warp, 51, 51);

		lsm_ensure_value(lsm, lsm->successor(warp, 51), 51);
		lsm_ensure_value(lsm, lsm->predecessor(warp, 51), 51);
		lsm_ensure_value(lsm, lsm->successor(warp, 34), 35);
		lsm_ensure_value(lsm, lsm->predecessor(warp, 36), 35);
		lsm_ensure_value(lsm, lsm->successor(warp, 36), 51);
		lsm_ensure_value(lsm, lsm->predecessor(warp, 34), 17);

		lsm_ensure_value(lsm, lsm->predecessor(warp, 190), 190); // It should be a split node
		lsm_ensure_value(lsm, lsm->successor(warp, 190), 190);
	}
	*/
}

SCENARIO("LSM", "[LSM]")
{
	int memory_size_allocated = 1u << 25u;
	unsigned int maximal_size = 1u << 20u;
	auto current_device = cuda::device::current::get();
	auto d_memory = cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated);
	auto d_allocator = cuda::memory::device::make_unique<allocator_type>(current_device);

	GIVEN("A LSM")
	{
		auto d_lsm = cuda::memory::device::make_unique<LSM>(current_device);

		cuda::launch(lsm_initialize_allocator_small,
			{ 1u, 1u },
			d_allocator.get(), d_memory.get(), memory_size_allocated, d_lsm.get(), maximal_size
		);

		WHEN("We add elements in increasing order")
		{
			THEN("It should be good")
			{
				cuda::launch(lsm_add_increasing_order,
					{ 1u, NUMBER_OF_WARPS * NUMBER_OF_THREADS_PER_WARP },
					d_lsm.get()
				);
			}
		}

		/*WHEN("We add elements in decreasing order")
		{
			THEN("It should be good")
			{
				cuda::launch(lsm_add_decreasing_order,
				{ 1u, NUMBER_OF_WARPS * 32u },
					d_lsm.get()
				);
			}
		}*/

		/*WHEN("We add elements with dulpicates")
		{
			THEN("It should be good")
			{
				cuda::launch(lsm_add_with_duplicates,
				{ 1u, NUMBER_OF_WARPS * 32u },
					d_lsm.get()
				);
			}
		}*/

		/*WHEN("We add elements in random order")
		{
			THEN("It should be good")
			{
				cuda::launch(lsm_add_random_order,
				{ 1u, NUMBER_OF_WARPS * 32u },
					d_lsm.get()
				);
			}
		}*/

		/*WHEN("We test for predecessor/successor")
		{
			THEN("It should be good")
			{
				cuda::launch(lsm_test_predecessor_successor,
				{ 1u, number_warps * 32u },
					d_lsm.get()
				);
			}
		}*/
	}
}
