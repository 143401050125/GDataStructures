#include "hip/hip_runtime.h"
#include "btree.cuh"
#include "Catch2/catch.hpp"
#include "cuda/api_wrappers.h"

#include "allocators/default_allocator.cuh"
#include "containers/hash_tables/default_hash_function.cuh"
#include <hip/hip_cooperative_groups.h>

using BTREE = gpu::BTree<gpu::Int64, gpu::Int64>;
using threads = cooperative_groups::thread_block_tile<32>;

inline __device__ void btree_ensure_value(BTREE* btree, typename BTREE::iterator it, int expected_value)
{
	ENSURE(!(it == btree->end()));
	ENSURE(it->second == expected_value);
}

__global__ void btree_initialize_allocator_small(gpu::default_allocator* allocator, char* memory, int memory_size, BTREE* btree)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	if (block.thread_rank() == 0)
		new (allocator) gpu::default_allocator(memory, memory_size);
	block.sync();
	new (btree) BTREE(block, *allocator);
}

__global__ void btree_add_increasing_order(BTREE* btree)
{
	threads warp = cooperative_groups::tiled_partition<32>(cooperative_groups::this_thread_block());

	for (int i = 0; i != 200; ++i)
	{
		btree->insert(warp, i, i);
		warp.sync();
	}

	/*auto it = btree->find(warp, 201);
	ENSURE(it == btree->end());

	it = btree->find(201);
	ENSURE(it == btree->end());*/
}

__global__ void btree_add_decreasing_order(BTREE* btree)
{
	threads warp = cooperative_groups::tiled_partition<32>(cooperative_groups::this_thread_block());

	for (int i = 200; i != 0; --i)
	{
		btree->insert(warp, i, i);
	}
}

__global__ void btree_add_random_order(BTREE* btree)
{
	threads warp = cooperative_groups::tiled_partition<32>(cooperative_groups::this_thread_block());

	for (int i = 500; i != 0; --i)
	{
		int hashed_i = int(gpu::hash<int>{}(i));
		btree->insert(warp, hashed_i, i);

		auto it = btree->find(warp, hashed_i);
		ENSURE(it->second == i);

		it = btree->find(hashed_i);
		ENSURE(it->second == i);
	}
}

__global__ void btree_test_predecessor_successor(BTREE* btree)
{
	threads warp = cooperative_groups::tiled_partition<32>(cooperative_groups::this_thread_block());

	ENSURE(btree->predecessor(warp, 128) == btree->end());
	ENSURE(btree->successor(warp, 128) == btree->end());

	btree->insert(warp, 2, 2);

	btree_ensure_value(btree, btree->predecessor(warp, 128), 2);
	ENSURE(btree->successor(warp, 128) == btree->end());
	ENSURE(btree->predecessor(warp, 1) == btree->end());
	btree_ensure_value(btree, btree->successor(warp, 1), 2);

	btree->insert(warp, 13, 13);
	btree_ensure_value(btree, btree->predecessor(warp, 128), 13);
	btree_ensure_value(btree, btree->predecessor(warp, 13), 13);
	btree_ensure_value(btree, btree->predecessor(warp, 12), 2);
	btree_ensure_value(btree, btree->successor(warp, 3), 13);

	btree->insert(warp, 251, 251);
	btree_ensure_value(btree, btree->predecessor(warp, 128), 13);
	btree_ensure_value(btree, btree->predecessor(warp, 253), 251);
	btree_ensure_value(btree, btree->successor(warp, 128), 251);
	ENSURE(btree->successor(warp, 252) == btree->end());

	btree->insert(warp, 190, 190);
	btree_ensure_value(btree, btree->successor(warp, 191), 251);
	btree_ensure_value(btree, btree->successor(warp, 190), 190);
	btree_ensure_value(btree, btree->predecessor(warp, 189), 13);
	btree_ensure_value(btree, btree->predecessor(warp, 250), 190);

	btree->insert(warp, 17, 17);
	btree->insert(warp, 35, 35);
	btree->insert(warp, 51, 51);

	btree_ensure_value(btree, btree->successor(warp, 51), 51);
	btree_ensure_value(btree, btree->predecessor(warp, 51), 51);
	btree_ensure_value(btree, btree->successor(warp, 34), 35);
	btree_ensure_value(btree, btree->predecessor(warp, 36), 35);
	btree_ensure_value(btree, btree->successor(warp, 36), 51);
	btree_ensure_value(btree, btree->predecessor(warp, 34), 17);

	btree_ensure_value(btree, btree->predecessor(warp, 190), 190); // It should be a split node
	btree_ensure_value(btree, btree->successor(warp, 190), 190);
}

SCENARIO("BTree", "[BTree]")
{
	int memory_size_allocated = 32 * 1024 * 1024;
	auto current_device = cuda::device::current::get();
	auto d_memory = cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated);
	auto d_allocator = cuda::memory::device::make_unique<gpu::default_allocator>(current_device);
	current_device.set_resource_limit(hipLimitStackSize, 4000);
	unsigned int number_warps = 1u;

	GIVEN("A BTree")
	{
		auto d_btree = cuda::memory::device::make_unique<BTREE>(current_device);

		cuda::launch(btree_initialize_allocator_small,
			{ 1u, 1u },
			d_allocator.get(), d_memory.get(), memory_size_allocated, d_btree.get()
		);

		/*WHEN("We add some elements")
		{
			THEN("We should retrieve them")
			{
				cuda::launch(btree_add,
				{ 1u, number_warps * 32u },
				d_btree.get());
			}
		}*/

		WHEN("We add elements in increasing order")
		{
			THEN("It should be good")
			{
				cuda::launch(btree_add_increasing_order,
					{ 1u, number_warps * 32u },
					d_btree.get()
				);
			}
		}

		WHEN("We add elements in decreasing order")
		{
			THEN("It should be good")
			{
				cuda::launch(btree_add_decreasing_order,
				{ 1u, number_warps * 32u },
					d_btree.get()
				);
			}
		}

		WHEN("We add elements in random order")
		{
			THEN("It should be good")
			{
				cuda::launch(btree_add_random_order,
				{ 1u, number_warps * 32u },
					d_btree.get()
				);
			}
		}

		WHEN("We test for predecessor/successor")
		{
			THEN("It should be good")
			{
				cuda::launch(btree_test_predecessor_successor,
				{ 1u, number_warps * 32u },
					d_btree.get()
				);
			}
		}
	}
}
