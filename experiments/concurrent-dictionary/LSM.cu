
#include <hip/hip_runtime.h>
//#include <hayai/hayai.hpp>
//
//#include "LSM.cuh"
//
//#include "concurrent-xfasttrie-fixture.cu"
//
//using LSM = gpu::lsm<key_type, mapped_type, 16u * 32u>;
//using LSMInsertionFixture = XTrieInsertionFixture<LSM, Structure::LSM>;
//using LSMGetThreadFixture = XTrieGetThreadFixture<LSM, Structure::LSM>;
//using LSMGetWorstThreadFixture = XTrieGetThreadFixture<LSM, Structure::LSM, true>;
//using LSMGetWarpFixture = XTrieGetWarpFixture<LSM, Structure::LSM>;
//using LSMGetWorstWarpFixture = XTrieGetWarpFixture<LSM, Structure::LSM, true>;
//using LSMPredecessorThreadFixture = XTriePredecessorFixture<LSM, Structure::LSM, false>;
//using LSMPredecessorWarpFixture = XTriePredecessorFixture<LSM, Structure::LSM, true>;
//using LSMPredecessorWorstWarpFixture = XTriePredecessorFixture<LSM, Structure::LSM, true, true>;
//using LSMSuccessorThreadFixture = XTrieSuccessorFixture<LSM, Structure::LSM, false>;
//using LSMSuccessorWarpFixture = XTrieSuccessorFixture<LSM, Structure::LSM, true>;
//
///*BENCHMARK_F(LSMInsertionFixture, LSM, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	insert();
//}*/
//
//BENCHMARK_F(LSMGetThreadFixture, GetThreadLSM, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get_thread();
//}
//
//BENCHMARK_F(LSMGetWorstThreadFixture, GetWorstThreadLSM, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get_thread();
//}
//
//BENCHMARK_F(LSMGetWarpFixture, GetWarpLSM, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get_warp();
//}
//
//BENCHMARK_F(LSMGetWorstWarpFixture, GetWorstWarpLSM, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get_warp();
//}
///*
//BENCHMARK_F(LSMPredecessorThreadFixture, PredecessorThreadLSM, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	predecessor();
//}
//*/
//BENCHMARK_F(LSMPredecessorWarpFixture, PredecessorWarpLSM, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	predecessor();
//}
//
//BENCHMARK_F(LSMPredecessorWorstWarpFixture, PredecessorWorstWarpLSM, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	predecessor();
//}
///*
//BENCHMARK_F(LSMSuccessorThreadFixture, SuccessorThreadLSM, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	successor();
//}
//
//BENCHMARK_F(LSMSuccessorWarpFixture, SuccessorWarpLSM, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	successor();
//}
//*/
