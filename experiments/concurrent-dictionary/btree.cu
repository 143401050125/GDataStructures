
#include <hip/hip_runtime.h>
//#include <hayai/hayai.hpp>
//
//#include "btree.cuh"
//
//#include "concurrent-xfasttrie-fixture.cu"
//
//using BTREE = gpu::BTree<key_type, mapped_type>;
//using BTreeInsertionFixture = XTrieInsertionFixture<BTREE, Structure::BTREE>;
//using BTreeGetThreadFixture = XTrieGetThreadFixture<BTREE, Structure::BTREE>;
//using BTreeGetWarpFixture = XTrieGetWarpFixture<BTREE, Structure::BTREE>;
//using BTreePredecessorFixture = XTriePredecessorFixture<BTREE, Structure::BTREE, true>;
//using BTreeSuccessorFixture = XTrieSuccessorFixture<BTREE, Structure::BTREE, true>;
//
//BENCHMARK_F(BTreeInsertionFixture, InsertionBtree, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	insert();
//}
///*
//BENCHMARK_F(BTreeGetThreadFixture, GetThreadBtree, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get_thread();
//}
//
//BENCHMARK_F(BTreeGetWarpFixture, GetWarpBtree, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get_warp();
//}
//
//BENCHMARK_F(BTreePredecessorFixture, PredecessorBtree, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	predecessor();
//}*/
///*
//BENCHMARK_F(BTreeSuccessorFixture, SuccessorBtree, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	successor();
//}*/
