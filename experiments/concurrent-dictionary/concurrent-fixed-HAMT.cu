
#include <hip/hip_runtime.h>
//#include <hayai/hayai.hpp>
//
//#include "fixed-HAMT.cuh"
//
//#include "concurrent-xfasttrie-fixture.cu"
//
//using HAMT5 = HAMT<key_type, mapped_type, 5>;
//using HAMT5InsertionFixture = XTrieInsertionFixture<HAMT5, Structure::XFASTTRIE>;
//using HAMT5GetWarpFixture = XTrieGetWarpFixture<HAMT5, Structure::XFASTTRIE>;
//using HAMT5PredecessorFixture = XTriePredecessorFixture<HAMT5, Structure::XFASTTRIE, true>;
//
//BENCHMARK_F(HAMT5InsertionFixture, HAMT5, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	insert();
//}
///*
//BENCHMARK_F(HAMT5GetWarpFixture, GetWarpHAMT5, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get_warp();
//}
//
//BENCHMARK_F(HAMT5PredecessorFixture, PredecessorHAMT5, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	predecessor();
//}*/
//
//using HAMT6 = HAMT<key_type, mapped_type, 6>;
//using HAMT6InsertionFixture = XTrieInsertionFixture<HAMT6, Structure::XFASTTRIE>;
//using HAMT6GetWarpFixture = XTrieGetWarpFixture<HAMT6, Structure::XFASTTRIE>;
//using HAMT6PredecessorFixture = XTriePredecessorFixture<HAMT6, Structure::XFASTTRIE, true>;
//
//BENCHMARK_F(HAMT6InsertionFixture, HAMT6, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	insert();
//}
///*
//BENCHMARK_F(HAMT6GetWarpFixture, GetWarpHAMT6, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get_warp();
//}
//
//BENCHMARK_F(HAMT6PredecessorFixture, PredecessorHAMT6, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	predecessor();
//}*/
