
#include <hip/hip_runtime.h>
//#include <hayai/hayai.hpp>
//
//#include "concurrent-xfasttrie-binary.cuh"
//
//#include "concurrent-xfasttrie-fixture.cu"
//
//using Binary = ConcurrentXFastTrieBinary<key_type, mapped_type, HEIGHT>;
//using BinaryInsertionFixture = XTrieInsertionFixture<Binary>;
//using BinaryGetThreadFixture = XTrieGetThreadFixture<Binary>;
//using BinaryGetWarpFixture = XTrieGetWarpFixture<Binary>;
//using BinaryPredecessorFixture = XTriePredecessorFixture<Binary>;
//using BinarySuccessorFixture = XTrieSuccessorFixture<Binary>;
//
//BENCHMARK_F(BinaryInsertionFixture, InsertionBinary, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	insert();
//}
//
///*
//BENCHMARK_F(BinaryGetThreadFixture, GetThreadBinary, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get_thread();
//}
//
//BENCHMARK_F(BinaryGetWarpFixture, GetWarpBinary, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get_warp();
//}
//*/
//
//BENCHMARK_F(BinaryPredecessorFixture, PredecessorBinary, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	predecessor();
//}
//
//BENCHMARK_F(BinarySuccessorFixture, SuccessorBinary, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	successor();
//}
