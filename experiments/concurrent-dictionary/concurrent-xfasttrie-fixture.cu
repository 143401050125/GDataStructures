#include "hip/hip_runtime.h"
#ifndef CONCURRENT_X_FAST_TRIE_FIXTURE_HPP
#define CONCURRENT_X_FAST_TRIE_FIXTURE_HPP

#include <cuda/api_wrappers.h>
#include <hayai/hayai.hpp>

#include "concurrent/allocators/default_allocator.cuh"
#include "concurrent/containers/hash_tables/default_hash_function.cuh"
#include "utility/pair.cuh"

#include <hip/hip_cooperative_groups.h>

enum class Structure
{
	BTREE,
	LSM,
	XFASTTRIE
};

using allocator_type = gpu::concurrent::default_allocator;

template <class XFastTrie>
inline __global__ void benchmark_xfasttrie_fixture_initialize_allocator(allocator_type* allocator, char* memory, unsigned int memory_size, XFastTrie* xtrie, unsigned int expected_number_of_elements)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	if (block.thread_rank() == 0)
		new (allocator) allocator_type(memory, memory_size);
	block.sync();
	new (xtrie) XFastTrie(block, *allocator, expected_number_of_elements);
}

template <class XFastTrie>
inline __global__ void benchmark_clear(allocator_type* allocator, XFastTrie* xtrie)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	allocator->clear(block);
	xtrie->clear(block);
}

template <class XFastTrie>
inline __global__ void benchmark_get_thread(XFastTrie* trie, unsigned int number_of_insertions, gpu::UInt64 upper_bound, unsigned int random_offset)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	cooperative_groups::thread_block_tile<32> warp = cooperative_groups::tiled_partition<32>(block);
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid > number_of_insertions)
		return;

	for (unsigned int offset = 0; offset < number_of_insertions; offset += blockDim.x * gridDim.x)
	{
		gpu::UInt64 hashed_i = gpu::UInt64(gpu::hash<unsigned int>{}(offset + thid + random_offset)) % upper_bound;
		volatile auto it = trie->find(hashed_i);
	}
}

template <class XFastTrie>
inline __global__ void benchmark_get_warp(XFastTrie* trie, unsigned int number_of_insertions, gpu::UInt64 upper_bound, unsigned int random_offset)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	cooperative_groups::thread_block_tile<32> warp = cooperative_groups::tiled_partition<32>(block);
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid > number_of_insertions)
		return;

	for (unsigned int i = thid / 32u; i < number_of_insertions; i += blockDim.x * gridDim.x / 32u)
	{
		gpu::UInt64 hashed_i = gpu::UInt64(gpu::hash<unsigned int>{}(i + random_offset)) % upper_bound;
		volatile auto it = trie->find(warp, hashed_i);
	}
}

template <class XFastTrie>
inline __global__ void benchmark_predecessor_thread(XFastTrie* trie, unsigned int number_of_insertions, gpu::UInt64 upper_bound, unsigned int random_offset)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	cooperative_groups::thread_block_tile<32> warp = cooperative_groups::tiled_partition<32>(block);
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid > number_of_insertions)
		return;

	for (unsigned int offset = 0; offset < number_of_insertions; offset += blockDim.x * gridDim.x)
	{
		gpu::UInt64 hashed_i = gpu::UInt64(gpu::hash<unsigned int>{}(offset + thid)) % upper_bound;
		hashed_i ^= random_offset;
		volatile auto it = trie->predecessor(hashed_i);
	}
}

template <class XFastTrie>
inline __global__ void benchmark_predecessor_warp(XFastTrie* trie, unsigned int number_of_insertions, gpu::UInt64 upper_bound, unsigned int random_offset)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	cooperative_groups::thread_block_tile<32> warp = cooperative_groups::tiled_partition<32>(block);
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid > number_of_insertions)
		return;

	for (unsigned int i = thid / 32u; i < number_of_insertions; i += blockDim.x * gridDim.x / 32u)
	{
		gpu::UInt64 hashed_i = gpu::UInt64(gpu::hash<unsigned int>{}(i + random_offset)) % upper_bound;
		hashed_i ^= random_offset;
		volatile auto it = trie->predecessor(warp, hashed_i);
	}
}

template <class XFastTrie>
inline __global__ void benchmark_successor_thread(XFastTrie* trie, unsigned int number_of_insertions, gpu::UInt64 upper_bound, unsigned int random_offset)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	cooperative_groups::thread_block_tile<32> warp = cooperative_groups::tiled_partition<32>(block);
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid > number_of_insertions)
		return;

	for (unsigned int offset = 0; offset < number_of_insertions; offset += blockDim.x * gridDim.x)
	{
		gpu::UInt64 hashed_i = gpu::UInt64(gpu::hash<unsigned int>{}(offset + thid)) % upper_bound;
		hashed_i ^= random_offset;
		volatile auto it = trie->successor(hashed_i);
	}
}

template <class XFastTrie>
inline __global__ void benchmark_successor_warp(XFastTrie* trie, unsigned int number_of_insertions, gpu::UInt64 upper_bound, unsigned int random_offset)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	cooperative_groups::thread_block_tile<32> warp = cooperative_groups::tiled_partition<32>(block);
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid > number_of_insertions)
		return;

	for (unsigned int i = thid / 32u; i < number_of_insertions; i += blockDim.x * gridDim.x / 32u)
	{
		gpu::UInt64 hashed_i = gpu::UInt64(gpu::hash<unsigned int>{}(i + random_offset)) % upper_bound;
		hashed_i ^= random_offset;
		volatile auto it = trie->successor(warp, hashed_i);
	}
}

template <class XFastTrie>
__global__ void benchmark_insert(XFastTrie* trie, unsigned int number_of_insertions, gpu::UInt64 upper_bound, unsigned int random_offset)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	cooperative_groups::thread_block_tile<32> warp = cooperative_groups::tiled_partition<32>(block);

	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid / 32 > number_of_insertions)
		return;

	for (unsigned int i = thid / 32u; i < number_of_insertions; i += blockDim.x * gridDim.x / 32u)
	{
		gpu::UInt64 hashed_i = gpu::UInt64(gpu::hash<unsigned int>{}(i + random_offset)) % upper_bound;
		trie->insert(warp, hashed_i, i);
	}
}

template <class XFastTrie>
__global__ void benchmark_insert_block(XFastTrie* trie, unsigned int number_of_insertions, gpu::UInt64 upper_bound, unsigned int random_offset)
{
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid > number_of_insertions)
		return;

	auto block = cooperative_groups::this_thread_block();
	for (unsigned int i = 0; i < number_of_insertions; i += blockDim.x * gridDim.x)
	{
		gpu::UInt64 hashed_i = gpu::UInt64(gpu::hash<unsigned int>{}(thid + i + random_offset)) % upper_bound;
		trie->insert(block, gpu::make_pair<typename XFastTrie::key_type, typename XFastTrie::mapped_type>(hashed_i, i));

		//printf("| %u %d | ", trie->number_of_batches(), trie->number_of_batches());
	}

	/*if (thid == 0)
		gpu::print(trie->size(), " ", blockDim.x);*/
}

template <class XFastTrie>
__global__ void benchmark_test(XFastTrie* trie)
{
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid == 0)
	{
		printf("%d ", trie->size());
		printf("%d ", trie->memory_consummed());
	}
}

template <class XFastTrie>
__global__ void benchmark_test_test(XFastTrie* trie)
{
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid == 0)
	{
		printf("%d ", trie->size());
		printf("%d \n", trie->number_of_batches());
	}
}

using key_type = gpu::UInt32;
using mapped_type = int;

constexpr unsigned int HEIGHT = 32u;
constexpr gpu::UInt64 UNIVERSE = (1u << HEIGHT) - 3u;
constexpr unsigned int MEMORY_ALLOCATED = 1u << 31u;
constexpr unsigned int NUMBER_OF_INSERTIONS = 1u << 19u;
constexpr unsigned int NUMBER_OF_ITERATIONS = 1u;// 5u;
constexpr unsigned int NUMBER_OF_RUNS = 30u;// 20u;
constexpr unsigned int NUMBER_OF_WARPS = 16u;
constexpr unsigned int NUMBER_OF_BLOCKS = 32u;

constexpr unsigned int STACK_SIZE = 4000u;
static unsigned int seed = 1;
static unsigned int random_offset = 1;

template <class DataStructure, Structure structure, bool WORST = false>
struct HelperInsert
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset);
};

template <class DataStructure>
struct HelperInsert<DataStructure, Structure::LSM, false>
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset)
	{
		gpu::UInt64 upper_bound = UNIVERSE;
		cuda::launch(
			benchmark_insert_block<DataStructure>,
			{ 1u, 16u * 32u },
			trie, NUMBER_OF_INSERTIONS, upper_bound, random_offset
		);
		cuda::device::current::get().synchronize();
	}
};

template <class DataStructure>
struct HelperInsert<DataStructure, Structure::LSM, true>
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset)
	{
		gpu::UInt64 upper_bound = UNIVERSE;
		cuda::launch(
			benchmark_insert_block<DataStructure>,
			{ 1u, 16u * 32u },
			trie, NUMBER_OF_INSERTIONS - 16 * 32u, upper_bound, random_offset
		);
		cuda::device::current::get().synchronize();
	}
};

template <class DataStructure>
struct HelperInsert<DataStructure, Structure::BTREE>
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset)
	{
		gpu::UInt64 upper_bound = UNIVERSE;
		cuda::launch(
			benchmark_insert<DataStructure>,
			{ 1u, 32u },
			trie, NUMBER_OF_INSERTIONS, upper_bound, random_offset
		);
		cuda::device::current::get().synchronize();
	}
};

template <class DataStructure>
struct HelperInsert<DataStructure, Structure::XFASTTRIE>
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset)
	{
		gpu::UInt64 upper_bound = UNIVERSE;
		cuda::launch(
			benchmark_insert<DataStructure>,
			{ NUMBER_OF_BLOCKS, NUMBER_OF_WARPS * 32u },
			trie, NUMBER_OF_INSERTIONS, upper_bound, random_offset
		);
		cuda::device::current::get().synchronize();
		/*cuda::launch(
			benchmark_test<DataStructure>,
			{ 1u, 1u },
			trie
		);
		cuda::device::current::get().synchronize();*/
	}
};

template <class DataStructure, bool WARP>
struct HelperPredecessor
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset);
};

template <class DataStructure>
struct HelperPredecessor<DataStructure, true>
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset)
	{
		gpu::UInt64 upper_bound = UNIVERSE;
		cuda::launch(
			benchmark_predecessor_warp<DataStructure>,
			{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
			trie, NUMBER_OF_INSERTIONS, upper_bound, random_offset
		);
		cuda::device::current::get().synchronize();
	}
};

template <class DataStructure>
struct HelperPredecessor<DataStructure, false>
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset)
	{
		gpu::UInt64 upper_bound = UNIVERSE;
		cuda::launch(
			benchmark_predecessor_thread<DataStructure>,
			{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
			trie, NUMBER_OF_INSERTIONS, upper_bound, random_offset
		);
		cuda::device::current::get().synchronize();
	}
};

template <class DataStructure, bool WARP>
struct HelperSuccessor
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset);
};

template <class DataStructure>
struct HelperSuccessor<DataStructure, true>
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset)
	{
		gpu::UInt64 upper_bound = UNIVERSE;
		cuda::launch(
			benchmark_successor_warp<DataStructure>,
			{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
			trie, NUMBER_OF_INSERTIONS, upper_bound, random_offset
		);
		cuda::device::current::get().synchronize();
	}
};

template <class DataStructure>
struct HelperSuccessor<DataStructure, false>
{
	void operator()(DataStructure* trie, unsigned int number_of_insertions, unsigned int random_offset)
	{
		gpu::UInt64 upper_bound = UNIVERSE;
		cuda::launch(
			benchmark_successor_thread<DataStructure>,
			{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
			trie, NUMBER_OF_INSERTIONS, upper_bound, random_offset
		);
		cuda::device::current::get().synchronize();
	}
};

template <class XFastTrie, Structure structure>
class XTrieInsertionFixture : public ::hayai::Fixture
{
	public:
		XTrieInsertionFixture() :
			::hayai::Fixture()
		{
			unsigned int memory_size_allocated = MEMORY_ALLOCATED;
			auto current_device = cuda::device::current::get();
			d_memory = std::move(cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated));
			d_allocator = std::move(cuda::memory::device::make_unique<allocator_type>(current_device));

			d_xtrie = std::move(cuda::memory::device::make_unique<XFastTrie>(current_device));
			cuda::launch(
				benchmark_xfasttrie_fixture_initialize_allocator<XFastTrie>,
				{ 1u, NUMBER_OF_WARPS * 32u },
				d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie.get(), NUMBER_OF_INSERTIONS
			);
			cuda::device::current::get().synchronize();

			random_offset = std::hash<unsigned int>{}(seed);
			seed = (seed % NUMBER_OF_RUNS) + 1u;
		}

		void insert()
		{
			HelperInsert<XFastTrie, structure>{}(d_xtrie.get(), NUMBER_OF_INSERTIONS, random_offset);
		}

		virtual void TearDown()
		{
			cuda::device::current::get().synchronize();
			cuda::launch(
				benchmark_clear<XFastTrie>,
				{ 1u, NUMBER_OF_WARPS * 32u },
				d_allocator.get(), d_xtrie.get()
			);
			cuda::device::current::get().synchronize();
		}

		cuda::memory::device::unique_ptr<char[]> d_memory;
		cuda::memory::device::unique_ptr<gpu::concurrent::default_allocator> d_allocator;
		cuda::memory::device::unique_ptr<XFastTrie> d_xtrie;
};

template <class XFastTrie, Structure structure, bool WORST = false>
class XTrieGetThreadFixture : public ::hayai::Fixture
{
	public:
		XTrieGetThreadFixture() :
			::hayai::Fixture()
		{
			unsigned int memory_size_allocated = MEMORY_ALLOCATED;
			auto current_device = cuda::device::current::get();
			d_memory = std::move(cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated));
			d_allocator = std::move(cuda::memory::device::make_unique<allocator_type>(current_device));
			current_device.set_resource_limit(hipLimitStackSize, STACK_SIZE);

			d_xtrie = std::move(cuda::memory::device::make_unique<XFastTrie>(current_device));
			cuda::launch(
				benchmark_xfasttrie_fixture_initialize_allocator<XFastTrie>,
				{ 1u, NUMBER_OF_WARPS * 32u },
				d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie.get(), NUMBER_OF_INSERTIONS
			);
			cuda::device::current::get().synchronize();

			random_offset = std::hash<unsigned int>{}(seed);
			seed = (seed % NUMBER_OF_RUNS) + 1u;

			insert();
		}

		void get_thread()
		{
			gpu::UInt64 upper_bound = UNIVERSE;
			cuda::launch(
				benchmark_get_thread<XFastTrie>,
				{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
				d_xtrie.get(), NUMBER_OF_INSERTIONS, upper_bound, random_offset
			);
			cuda::device::current::get().synchronize();
		}

		void insert()
		{
			HelperInsert<XFastTrie, structure, WORST>{}(d_xtrie.get(), NUMBER_OF_INSERTIONS, random_offset);
		}

		cuda::memory::device::unique_ptr<char[]> d_memory;
		cuda::memory::device::unique_ptr<gpu::concurrent::default_allocator> d_allocator;
		cuda::memory::device::unique_ptr<XFastTrie> d_xtrie;
};

template <class XFastTrie, Structure structure, bool WORST = false>
class XTrieGetWarpFixture : public ::hayai::Fixture
{
	public:
		XTrieGetWarpFixture() :
			::hayai::Fixture()
		{
			unsigned int memory_size_allocated = MEMORY_ALLOCATED;
			auto current_device = cuda::device::current::get();
			d_memory = std::move(cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated));
			d_allocator = std::move(cuda::memory::device::make_unique<allocator_type>(current_device));
			current_device.set_resource_limit(hipLimitStackSize, STACK_SIZE);

			d_xtrie = std::move(cuda::memory::device::make_unique<XFastTrie>(current_device));
			cuda::launch(
				benchmark_xfasttrie_fixture_initialize_allocator<XFastTrie>,
				{ 1u, NUMBER_OF_WARPS * 32u },
				d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie.get(), NUMBER_OF_INSERTIONS
			);
			cuda::device::current::get().synchronize();

			random_offset = std::hash<unsigned int>{}(seed);
			seed = (seed % NUMBER_OF_RUNS) + 1u;

			insert();
		}

		void get_warp()
		{
			gpu::UInt64 upper_bound = UNIVERSE;
			cuda::launch(
				benchmark_get_warp<XFastTrie>,
				{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
				d_xtrie.get(), NUMBER_OF_INSERTIONS, upper_bound, random_offset
			);
			cuda::device::current::get().synchronize();
		}

		void insert()
		{
			HelperInsert<XFastTrie, structure, WORST>{}(d_xtrie.get(), NUMBER_OF_INSERTIONS, random_offset);
		}

		cuda::memory::device::unique_ptr<char[]> d_memory;
		cuda::memory::device::unique_ptr<gpu::concurrent::default_allocator> d_allocator;
		cuda::memory::device::unique_ptr<XFastTrie> d_xtrie;
};

template <class XFastTrie, Structure structure, bool WARP = true, bool WORST = false>
class XTriePredecessorFixture : public ::hayai::Fixture
{
	public:
		XTriePredecessorFixture() :
			::hayai::Fixture()
		{
			unsigned int memory_size_allocated = MEMORY_ALLOCATED;
			auto current_device = cuda::device::current::get();
			d_memory = std::move(cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated));
			d_allocator = std::move(cuda::memory::device::make_unique<allocator_type>(current_device));
			current_device.set_resource_limit(hipLimitStackSize, STACK_SIZE);

			d_xtrie = std::move(cuda::memory::device::make_unique<XFastTrie>(current_device));
			cuda::launch(
				benchmark_xfasttrie_fixture_initialize_allocator<XFastTrie>,
				{ 1u, NUMBER_OF_WARPS * 32u },
				d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie.get(), NUMBER_OF_INSERTIONS
			);
			cuda::device::current::get().synchronize();

			random_offset = std::hash<unsigned int>{}(seed);
			seed = (seed % NUMBER_OF_RUNS) + 1u;

			insert();
		}

		void predecessor()
		{
			HelperPredecessor<XFastTrie, WARP>{}(d_xtrie.get(), NUMBER_OF_INSERTIONS, random_offset);
		}

		void insert()
		{
			HelperInsert<XFastTrie, structure, WORST>{}(d_xtrie.get(), NUMBER_OF_INSERTIONS, random_offset);
		}

		cuda::memory::device::unique_ptr<char[]> d_memory;
		cuda::memory::device::unique_ptr<gpu::concurrent::default_allocator> d_allocator;
		cuda::memory::device::unique_ptr<XFastTrie> d_xtrie;
};

template <class XFastTrie, Structure structure, bool WARP = true, bool WORST = false>
class XTrieSuccessorFixture : public ::hayai::Fixture
{
	public:
		XTrieSuccessorFixture() :
			::hayai::Fixture()
		{
			unsigned int memory_size_allocated = MEMORY_ALLOCATED;
			auto current_device = cuda::device::current::get();
			d_memory = std::move(cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated));
			d_allocator = std::move(cuda::memory::device::make_unique<allocator_type>(current_device));
			current_device.set_resource_limit(hipLimitStackSize, STACK_SIZE);

			d_xtrie = std::move(cuda::memory::device::make_unique<XFastTrie>(current_device));
			cuda::launch(
				benchmark_xfasttrie_fixture_initialize_allocator<XFastTrie>,
				{ 1u, NUMBER_OF_WARPS * 32u },
				d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie.get(), NUMBER_OF_INSERTIONS
			);
			cuda::device::current::get().synchronize();

			random_offset = std::hash<unsigned int>{}(seed);
			seed = (seed % NUMBER_OF_RUNS) + 1u;

			insert();
		}

		void successor()
		{
			HelperSuccessor<XFastTrie, WARP>{}(d_xtrie.get(), NUMBER_OF_INSERTIONS, random_offset);
		}

		void insert()
		{
			HelperInsert<XFastTrie, structure, WORST>{}(d_xtrie.get(), NUMBER_OF_INSERTIONS, random_offset);
		}

		cuda::memory::device::unique_ptr<char[]> d_memory;
		cuda::memory::device::unique_ptr<gpu::concurrent::default_allocator> d_allocator;
		cuda::memory::device::unique_ptr<XFastTrie> d_xtrie;
};

#endif // CONCURRENT_X_FAST_TRIE_FIXTURE_HPP
