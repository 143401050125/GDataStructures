
#include <hip/hip_runtime.h>
//#include <hayai/hayai.hpp>
//
//#include "concurrent/containers/hash_tables/chaining.cuh"
//
//#include "hash_map-fixture.cu"
//
//using Chaining = gpu::concurrent::chaining<key_type, mapped_type, gpu::hash<key_type>>;
//using ChainingInsertionFixture = HashMapInsertionFixture<Chaining>;
//using ChainingGetFixture = HashMapGetFixture<Chaining>;
//using ChainingGetUnsuccessfulFixture = HashMapGetUnsuccessfulFixture<Chaining>;
//
//BENCHMARK_F(ChainingInsertionFixture, Chaining, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	insert();
//}
//
//BENCHMARK_F(ChainingGetFixture, Chaining, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get();
//}
//
//BENCHMARK_F(ChainingGetUnsuccessfulFixture, Chaining, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	unsuccessful_get();
//}
