
#include <hip/hip_runtime.h>
//#include <hayai/hayai.hpp>
//
//#include "concurrent/containers/hash_tables/fixed_cuckoo.cuh"
//#include "concurrent/containers/hash_tables/fixed_bucket_cuckoo.cuh"
//
//#include "hash_map-fixture.cu"
///*
//using Cuckoo2 = gpu::concurrent::fixed_cuckoo<key_type, mapped_type, gpu::hash<key_type>, 2>;
//using Cuckoo2InsertionFixture = HashMapInsertionFixture<Cuckoo2>;
//using Cuckoo2GetFixture = HashMapGetFixture<Cuckoo2>;
//using Cuckoo2GetUnsuccessfulFixture = HashMapGetUnsuccessfulFixture<Cuckoo2>;
//
//BENCHMARK_F(Cuckoo2InsertionFixture, Cuckoo2, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	insert();
//}
//
//BENCHMARK_F(Cuckoo2GetFixture, Cuckoo2, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get();
//}
//
//BENCHMARK_F(Cuckoo2GetUnsuccessfulFixture, Cuckoo2, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	unsuccessful_get();
//}
//*/
//using Cuckoo3 = gpu::concurrent::fixed_cuckoo<key_type, mapped_type, gpu::hash<key_type>, 3>;
//using Cuckoo3InsertionFixture = HashMapInsertionFixture<Cuckoo3>;
//using Cuckoo3GetFixture = HashMapGetFixture<Cuckoo3>;
//using Cuckoo3GetUnsuccessfulFixture = HashMapGetUnsuccessfulFixture<Cuckoo3>;
//
//BENCHMARK_F(Cuckoo3InsertionFixture, Cuckoo3, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	insert();
//}
//
//BENCHMARK_F(Cuckoo3GetFixture, Cuckoo3, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get();
//}
//
//BENCHMARK_F(Cuckoo3GetUnsuccessfulFixture, Cuckoo3, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	unsuccessful_get();
//}
//
//using Cuckoo4 = gpu::concurrent::fixed_cuckoo<key_type, mapped_type, gpu::hash<key_type>, 4>;
//using Cuckoo4InsertionFixture = HashMapInsertionFixture<Cuckoo4>;
//using Cuckoo4GetFixture = HashMapGetFixture<Cuckoo4>;
//using Cuckoo4GetUnsuccessfulFixture = HashMapGetUnsuccessfulFixture<Cuckoo4>;
//
//BENCHMARK_F(Cuckoo4InsertionFixture, Cuckoo4, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	insert();
//}
//
//BENCHMARK_F(Cuckoo4GetFixture, Cuckoo4, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get();
//}
//
//BENCHMARK_F(Cuckoo4GetUnsuccessfulFixture, Cuckoo4, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	unsuccessful_get();
//}
//
//using BucketCuckoo4 = gpu::concurrent::fixed_bucket_cuckoo<key_type, mapped_type, gpu::hash<key_type>, 4>;
//using BucketCuckoo4InsertionFixture = HashMapInsertionFixture<BucketCuckoo4>;
//using BucketCuckoo4GetFixture = HashMapGetFixture<BucketCuckoo4>;
//using BucketCuckoo4GetUnsuccessfulFixture = HashMapGetUnsuccessfulFixture<BucketCuckoo4>;
//
//
//BENCHMARK_F(BucketCuckoo4InsertionFixture, BucketCuckoo4, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	insert();
//}
//
//BENCHMARK_F(BucketCuckoo4GetFixture, BucketCuckoo4, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	get();
//}
//
//BENCHMARK_F(BucketCuckoo4GetUnsuccessfulFixture, BucketCuckoo4, NUMBER_OF_RUNS, NUMBER_OF_ITERATIONS)
//{
//	unsuccessful_get();
//}
