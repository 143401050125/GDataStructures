#include "hip/hip_runtime.h"
#ifndef FAST_INTEGER_FIXTURE_HPP
#define FAST_INTEGER_FIXTURE_HPP

#include <cuda/api_wrappers.h>
#include <hayai/hayai.hpp>

#include "concurrent/containers/hash_tables/default_hash_function.cuh"
#include "concurrent/allocators/default_allocator.cuh"

#include "utility/pair.cuh"

using allocator_type = gpu::concurrent::default_allocator;

template <class HashMap>
inline __global__ void benchmark_hash_map_fixture_initialize_allocator(allocator_type* allocator, char* memory, unsigned int memory_size, HashMap* map, unsigned int number_of_insertions)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	cooperative_groups::thread_block_tile<32> warp = cooperative_groups::tiled_partition<32>(block);

	if (block.thread_rank() == 0)
		new (allocator) allocator_type(memory, memory_size);
	block.sync();
	new (map) HashMap(warp, *allocator, number_of_insertions);
}

template <class HashMap>
inline __global__ void benchmark_hash_map_fixture_clear(allocator_type* allocator, HashMap* map)
{
	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
	cooperative_groups::thread_block_tile<32> warp = cooperative_groups::tiled_partition<32>(cooperative_groups::this_thread_block());
	allocator->clear(block);
	map->clear(warp);
}

template <class HashMap>
inline __global__ void benchmark_hash_map_insert(HashMap* map, unsigned int number_of_insertions, unsigned int random_offset)
{
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid > number_of_insertions)
		return;

	for (unsigned int offset = 0; offset < number_of_insertions; offset += blockDim.x * gridDim.x)
	{
		unsigned int hashed_i = gpu::hash<unsigned int>{}(random_offset + offset + thid) % static_cast<unsigned int>(-3);
		map->insert(gpu::make_pair(hashed_i, thid));
	}
}

template <class HashMap>
inline __global__ void benchmark_hash_map_get(HashMap* map, unsigned int number_of_insertions, unsigned int random_offset)
{
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid > number_of_insertions)
		return;

	for (unsigned int offset = 0; offset < number_of_insertions; offset += blockDim.x * gridDim.x)
	{
		int hashed_i = int(gpu::hash<unsigned int>{}(random_offset + offset + thid));
		volatile auto it = map->find(hashed_i);
	}
}

template <class HashMap>
inline __global__ void benchmark_hash_map_get_unsuccessful(HashMap* map, unsigned int number_of_insertions, unsigned int random_offset)
{
	unsigned int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if (thid > number_of_insertions)
		return;

	for (unsigned int offset = 0; offset < number_of_insertions; offset += blockDim.x * gridDim.x)
	{
		volatile auto it = map->find(random_offset + offset + thid);
	}

}

using key_type = gpu::UInt32;
using mapped_type = gpu::UInt32;

constexpr unsigned int MEMORY_ALLOCATED = 1u << (26u + 5u);// 25u;
constexpr unsigned int NUMBER_OF_INSERTIONS = 1u << 20u;// 20u; // Min = 2^15
constexpr unsigned int NUMBER_OF_ITERATIONS = 10u;
constexpr unsigned int NUMBER_OF_RUNS = 30u;
constexpr unsigned int NUMBER_OF_WARPS = 32u;// 32u;
constexpr unsigned int NUMBER_OF_BLOCKS = 32u;// 32u;

static unsigned int seed = 1;
static unsigned int random_offset = 0;

template <class HashMap>
class HashMapInsertionFixture : public ::hayai::Fixture
{
	public:
		HashMapInsertionFixture() :
			::hayai::Fixture()
		{
			auto current_device = cuda::device::current::get();
			d_memory = std::move(cuda::memory::device::make_unique<char[]>(current_device, MEMORY_ALLOCATED));
			d_allocator = std::move(cuda::memory::device::make_unique<allocator_type>(current_device));

			d_hash_map = std::move(cuda::memory::device::make_unique<HashMap>(current_device));
			cuda::launch(
				benchmark_hash_map_fixture_initialize_allocator<HashMap>,
				{ 1u, 32u },
				d_allocator.get(), d_memory.get(), MEMORY_ALLOCATED, d_hash_map.get(), NUMBER_OF_INSERTIONS
			);
			cuda::device::current::get().synchronize();

			random_offset = std::hash<unsigned int>{}(seed);
			seed = (seed % NUMBER_OF_RUNS) + 1u;
		}

		virtual void TearDown()
		{
			cuda::device::current::get().synchronize();
			cuda::launch(
				benchmark_hash_map_fixture_clear<HashMap>,
				{ 1u, 32u },
				d_allocator.get(), d_hash_map.get()
			);
			cuda::device::current::get().synchronize();
		}

		void insert()
		{
			cuda::launch(
				benchmark_hash_map_insert<HashMap>,
				{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
				d_hash_map.get(), NUMBER_OF_INSERTIONS, random_offset
			);
			cuda::device::current::get().synchronize();
		}

		cuda::memory::device::unique_ptr<char[]> d_memory;
		cuda::memory::device::unique_ptr<allocator_type> d_allocator;
		cuda::memory::device::unique_ptr<HashMap> d_hash_map;
};

template <class HashMap>
class HashMapGetFixture : public ::hayai::Fixture
{
	public:
		HashMapGetFixture() :
			::hayai::Fixture()
		{
			auto current_device = cuda::device::current::get();
			d_memory = std::move(cuda::memory::device::make_unique<char[]>(current_device, MEMORY_ALLOCATED));
			d_allocator = std::move(cuda::memory::device::make_unique<allocator_type>(current_device));

			d_hash_map = std::move(cuda::memory::device::make_unique<HashMap>(current_device));
			cuda::launch(
				benchmark_hash_map_fixture_initialize_allocator<HashMap>,
				{ 1u, 32u },
				d_allocator.get(), d_memory.get(), MEMORY_ALLOCATED, d_hash_map.get(), NUMBER_OF_INSERTIONS
			);
			cuda::device::current::get().synchronize();

			insert();

			random_offset = std::hash<unsigned int>{}(seed);
			seed = (seed % NUMBER_OF_RUNS) + 1u;
		}

		virtual void TearDown()
		{
			cuda::device::current::get().synchronize();
			cuda::launch(
				benchmark_hash_map_fixture_clear<HashMap>,
				{ 1u, 32u },
				d_allocator.get(), d_hash_map.get()
			);
			cuda::device::current::get().synchronize();
		}

		void get()
		{
			cuda::launch(
				benchmark_hash_map_get<HashMap>,
				{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
				d_hash_map.get(), NUMBER_OF_INSERTIONS, random_offset
			);
			cuda::device::current::get().synchronize();
		}

	private:
		void insert()
		{
			cuda::launch(
				benchmark_hash_map_insert<HashMap>,
				{ 32u * 1u, 32u * 32u },
				d_hash_map.get(), NUMBER_OF_INSERTIONS, random_offset
			);
			cuda::device::current::get().synchronize();
		}

		cuda::memory::device::unique_ptr<char[]> d_memory;
		cuda::memory::device::unique_ptr<allocator_type> d_allocator;
		cuda::memory::device::unique_ptr<HashMap> d_hash_map;
};

template <class HashMap>
class HashMapGetUnsuccessfulFixture : public ::hayai::Fixture
{
	public:
		HashMapGetUnsuccessfulFixture() :
			::hayai::Fixture()
		{
			auto current_device = cuda::device::current::get();
			d_memory = std::move(cuda::memory::device::make_unique<char[]>(current_device, MEMORY_ALLOCATED));
			d_allocator = std::move(cuda::memory::device::make_unique<allocator_type>(current_device));

			d_hash_map = std::move(cuda::memory::device::make_unique<HashMap>(current_device));
			cuda::launch(
				benchmark_hash_map_fixture_initialize_allocator<HashMap>,
				{ 1u, 32u },
				d_allocator.get(), d_memory.get(), MEMORY_ALLOCATED, d_hash_map.get(), NUMBER_OF_INSERTIONS
			);
			cuda::device::current::get().synchronize();

			insert();

			random_offset = std::hash<unsigned int>{}(seed);
			seed = (seed % NUMBER_OF_RUNS) + 1u;
		}

		virtual void TearDown()
		{
			cuda::device::current::get().synchronize();
			cuda::launch(
				benchmark_hash_map_fixture_clear<HashMap>,
				{ 1u, 32u },
				d_allocator.get(), d_hash_map.get()
			);
			cuda::device::current::get().synchronize();
		}

		void unsuccessful_get()
		{
			cuda::launch(
				benchmark_hash_map_get_unsuccessful<HashMap>,
				{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
				d_hash_map.get(), NUMBER_OF_INSERTIONS, random_offset
			);
			cuda::device::current::get().synchronize();
		}

	private:
		void insert()
		{
			cuda::launch(
				benchmark_hash_map_insert<HashMap>,
				{ 32u * 1u, 32u * 32u },
				d_hash_map.get(), NUMBER_OF_INSERTIONS, random_offset
			);
			cuda::device::current::get().synchronize();
		}

		cuda::memory::device::unique_ptr<char[]> d_memory;
		cuda::memory::device::unique_ptr<allocator_type> d_allocator;
		cuda::memory::device::unique_ptr<HashMap> d_hash_map;
};

#endif // FAST_INTEGER_FIXTURE_HPP
