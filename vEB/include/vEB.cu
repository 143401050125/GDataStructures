#include "hip/hip_runtime.h"
#include "vEB.cuh"

#include "utility/print.cuh"

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::iterator vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::end()
{
	return m_bottom.end();
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::const_iterator vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::end() const
{
	return m_bottom.end();
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::vEB(block_threads block, allocator_type& allocator, unsigned int expected_number_of_elements) :
	m_allocator(&allocator)
{
	unsigned int power_of_two;
	if (expected_number_of_elements == 0)
		power_of_two = 10u;
	else
		power_of_two = __ffs(expected_number_of_elements) - 1u;

	auto warp = cooperative_groups::tiled_partition<32>(block);
	if (block.thread_rank() < warp.size())
		clear(warp, &m_vEB);

	new (&m_bottom) Map{ block, allocator, 1u << power_of_two };
	block.sync();
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::vEB(threads group, allocator_type& allocator, unsigned int expected_number_of_elements) :
	m_allocator(&allocator)
{
	unsigned int power_of_two;
	if (expected_number_of_elements == 0)
		power_of_two = 10u;
	else
		power_of_two = __ffs(expected_number_of_elements) - 1u;

	clear(group, &m_vEB);

	new (&m_bottom) Map{ block, allocator, 1u << power_of_two };
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ void vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::clear(block_threads block)
{
	threads tile32 = cooperative_groups::tiled_partition<32>(block);

	if (block.thread_rank() < tile32.size())
		clear(tile32);

	block.sync();
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ void vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::clear(threads group)
{
	clear(group, &m_vEB);

	m_bottom.clear(group);
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::iterator vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::find(key_type key)
{
	return m_bottom.find(key);
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::iterator vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::find(threads group, key_type key)
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::const_iterator vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::find(threads group, key_type key) const
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::iterator vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::insert(threads group, key_type key, mapped_type value)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	unsigned int depth = 0u;
	Node* current_node = &m_vEB;
#ifdef VAN_EMDE_BOAS_DEBUG
	if (group.thread_rank() == 0)
		printf("%d\n\n", key);
#endif // VAN_EMDE_BOAS_DEBUG
	while (true)
	{
	#ifdef VAN_EMDE_BOAS_DEBUG
		if (group.thread_rank() == 0)
			printf("%d %d %p ", depth, is_leaf(depth), current_node);
	#endif // VAN_EMDE_BOAS_DEBUG
		if (is_leaf(depth))
		{
			LeafNode* leaf_node = reinterpret_cast<LeafNode*>(current_node);
		#ifdef VAN_EMDE_BOAS_DEBUG
			if (group.thread_rank() == 0)
				printf("END: %p\n", leaf_node);
		#endif // VAN_EMDE_BOAS_DEBUG
			return set(group, leaf_node, key, value);
		}
		else
		{
			size_type bits = extract_bits(key, depth);
		#ifdef VAN_EMDE_BOAS_DEBUG
			if (group.thread_rank() == 0)
				printf("%d ", bits);
		#endif // VAN_EMDE_BOAS_DEBUG
			InternalNode* internal_node = reinterpret_cast<InternalNode*>(current_node);
			gpu::atomic<Node*>& ptr = internal_node->nodes[bits];
			do
			{
				if (!ptr)
				{
					if (ptr.compare_and_swap(group, nullptr, INSERTING()) == nullptr)
					{
						++depth;
						Node* next_node;
						if (is_leaf(depth))
						{
							LeafNode* leaf_node = m_allocator->allocate<LeafNode>(group);
						#ifdef VAN_EMDE_BOAS_DEBUG
							if (group.thread_rank() == 0)
								printf("Leaf %p\n", leaf_node);
						#endif // VAN_EMDE_BOAS_DEBUG
							clear(group, leaf_node);
							next_node = leaf_node;
						}
						else
						{
							InternalNode* internal_node = m_allocator->allocate<InternalNode>(group);
						#ifdef VAN_EMDE_BOAS_DEBUG
							if (group.thread_rank() == 0)
								printf("int %p\n", internal_node);
						#endif // VAN_EMDE_BOAS_DEBUG
							clear(group, internal_node);
							next_node = internal_node;
						}
						ptr.store(group, next_node);
						current_node = next_node;
					}
				}
				else
				{
					if (ptr != INSERTING())
					{
						++depth;
						current_node = ptr;
					#ifdef VAN_EMDE_BOAS_DEBUG
						if (group.thread_rank() == 0)
							printf("\n");
					#endif // VAN_EMDE_BOAS_DEBUG
						break;
					}
				}
			} while (ptr == INSERTING());
		}
	}
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::iterator vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::predecessor(threads group, key_type key)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	key_type discovered_bits = 0u;
	unsigned int depth = 0u;
	unsigned int predecessor_depth = 0u;
	Node* predecessor_node = nullptr;
	Node* current_node = &m_vEB;
	while (true)
	{
		if (is_leaf(depth))
		{
			LeafNode* leaf_node = reinterpret_cast<LeafNode*>(current_node);
			auto leaf_predecessor_info = predecessor_leaf(group, leaf_node, key);
			if (leaf_predecessor_info.in_leaf)
				return m_bottom.find(leaf_predecessor_info.key);
			else
				break;
		}
			
		else
		{
			size_type bits = extract_bits(key, depth);
			InternalNode* internal_node = reinterpret_cast<InternalNode*>(current_node);
			auto predecessor_info = find_predecessor(group, internal_node, bits);
			bool has_predecessor = predecessor_info.has_predecessor;
			Node* current_predecessor = predecessor_info.current_predecessor;
			Node* ptr = predecessor_info.ptr;
		#ifdef VAN_EMDE_BOAS_DEBUG
			if (group.thread_rank() == 0)
				printf("%p %p %d \n", ptr, current_predecessor, has_predecessor);
		#endif // VAN_EMDE_BOAS_DEBUG
			if (has_predecessor)
			{
				predecessor_depth = depth;
				predecessor_node = current_predecessor;
				if (depth == 0u)
					discovered_bits = predecessor_info.predecessor_index;
				else
				{
					discovered_bits = key >> (UNIVERSE - depth * NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO);
					discovered_bits = (discovered_bits << relative_shift(depth)) + predecessor_info.predecessor_index;
				}
			}

			++depth;

			if (!ptr || ptr == INSERTING())
				break;

			current_node = ptr;
		}
	}

	if (predecessor_node)
		return find_max(group, predecessor_node, discovered_bits, predecessor_depth + 1);
	else
		return m_bottom.end();
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::size_type vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::size() const
{
	return m_bottom.size();
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ void vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::debug() const
{
	internal_debug(&m_vEB, 0, 0u);
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ void vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::internal_debug(const Node* node, unsigned int depth, key_type key) const
{
	if (is_leaf(depth))
	{
		const LeafNode* leaf_node = reinterpret_cast<const LeafNode*>(node);

		for (unsigned int i = 0u; i != NUMBER_OF_ELEMENTS_AT_BOTTOM; ++i)
		{
			unsigned int data = leaf_node->bits[i];
			for (unsigned j = 0u; j != 8 * sizeof(gpu::UInt32); ++j)
			{
				if (data & (1u << j))
					gpu::print(key * NUMBER_OF_BITS_PER_NODE + i * 32 + j, " ");
			}
		}
		gpu::print("\n");
	}
	else
	{
		for (unsigned int i = 0u; i != NUMBER_OF_CHILDREN_PER_NODE; ++i)
		{
			const InternalNode* internal_node = reinterpret_cast<const InternalNode*>(node);
			const Node* ptr = internal_node->nodes[i];
			if (ptr && ptr != INSERTING())
			{
				unsigned int search_bits = UNIVERSE - NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO;
				unsigned int delta = (search_bits / NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO);
				delta = delta * NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO;
				delta = (UNIVERSE - NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO) - delta;

				if (!is_leaf(depth + 1))
					internal_debug(internal_node->nodes[i], depth + 1, (key << NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO) + i);
				else
					internal_debug(internal_node->nodes[i], depth + 1, (key << delta) + i);
				
			}
				
		}
	}
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::Node* vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::INSERTING() const
{
	return reinterpret_cast<Node*>(10);
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::size_type vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::extract_bits(key_type key, unsigned int depth) const
{
	unsigned int shift = (UNIVERSE - NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO * (depth + 1u));
	unsigned int shifted_key = key >> shift;
	unsigned int masked_key = shifted_key & (NUMBER_OF_CHILDREN_PER_NODE - 1u);
	if (depth + 1u == NUMBER_OF_LEVELS)
	{
		unsigned int search_bits = UNIVERSE - NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO;
		unsigned int delta = (search_bits / NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO);
		delta = delta * NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO;
		delta = (UNIVERSE - NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO) - delta;
		masked_key = masked_key >> (NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO - delta);
	}
	return masked_key;
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ bool vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::is_leaf(unsigned int depth) const
{
	return NUMBER_OF_LEVELS == depth;
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ void vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::clear(threads g, InternalNode* internal_node)
{
	unsigned int offset = 0u;
	while (offset < NUMBER_OF_CHILDREN_PER_NODE)
	{
		internal_node->nodes[offset + g.thread_rank()].store_unatomically(nullptr);
		offset += g.size();
	}
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ void vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::clear(threads g, LeafNode* leaf_node)
{
	for (unsigned int offset = 0u; offset != NUMBER_OF_ELEMENTS_AT_BOTTOM; offset += g.size())
		leaf_node->bits[offset + g.thread_rank()].store_unatomically(0u);
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ unsigned int vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::relative_shift(unsigned int depth) const
{
	if (depth + 1u == NUMBER_OF_LEVELS)
	{
		unsigned int search_bits = UNIVERSE - NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO;
		unsigned int delta = (search_bits / NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO);
		delta = delta * NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO;
		delta = (UNIVERSE - NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO) - delta;
		return delta;
	}
	else
		return NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO;
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::iterator vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::set(threads g, LeafNode* current_node, key_type key, mapped_type value)
{
	auto local_bits = key & (NUMBER_OF_BITS_PER_NODE - 1u);
	auto local_thid = local_bits / g.size();
	auto local_bit = local_bits % g.size();
	if (g.thread_rank() == (local_thid % g.size()))
		current_node->bits[local_thid].fetch_or(1u << local_bit);
	
	return m_bottom.insert(g, gpu::make_pair<key_type, mapped_type>(key, value));
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::iterator vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::find_max(threads g, Node* current_node, key_type discovered_bits, unsigned int depth)
{
	while (true)
	{
		if (is_leaf(depth))
		{
			LeafNode* leaf_node = reinterpret_cast<LeafNode*>(current_node);
			return find_max(g, leaf_node, discovered_bits);
		}
		else
		{
			InternalNode* internal_node = reinterpret_cast<InternalNode*>(current_node);
			auto result = find_max(g, internal_node);
			if (!result.next_node)
				return end();

			current_node = result.next_node;
			discovered_bits = (discovered_bits << relative_shift(depth)) + result.index;
			++depth;
		}
	}
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::MaxInfo vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::find_max(threads g, InternalNode* internal_node)
{
	unsigned int work_to_do = NUMBER_OF_CHILDREN_PER_NODE;
	unsigned int already_done = 0u;
	unsigned int pos = 0u;
	do
	{
		pos = work_to_do - g.size() - already_done + g.thread_rank();
		Node* ptr = internal_node->nodes[pos];
		unsigned int warp_result = g.ballot(ptr != nullptr);
		if (warp_result)
		{
			unsigned int winner_thid = 31u - __clz(warp_result);
			ptr = reinterpret_cast<Node*>(g.shfl(reinterpret_cast<std::uintptr_t>(ptr), winner_thid));
			pos = g.shfl(pos, winner_thid);
			return { ptr, pos };
		}
		already_done += g.size();
	} while (already_done < work_to_do);
	return { nullptr, -1 };
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::iterator vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::find_max(threads g, LeafNode* leaf_node, key_type key)
{
	unsigned int local_offset = (NUMBER_OF_ELEMENTS_AT_BOTTOM / g.size()) - 1u;
	do
	{
		unsigned int local_index = local_offset * g.size() + g.thread_rank();
		bool has_predecessor = leaf_node->bits[local_index];
		unsigned int warp_result = g.ballot(has_predecessor);
		if (!warp_result)
		{
			if (local_offset == 0u)
				return end();
			else
				--local_offset;
		}
		else
		{
			unsigned int winner_thid = __ffs(warp_result) - 1;
			unsigned int data = leaf_node->bits[local_offset * g.size() + winner_thid];
			unsigned int shift = 31 - __clz(data);
			key_type result_key = key * NUMBER_OF_BITS_PER_NODE + winner_thid * 32 + local_offset * 1024 + shift;

			return m_bottom.find(g, result_key);
		}
	} while (local_offset != 0);
	return end();
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::PredecessorInfo vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::find_predecessor(threads g, InternalNode* internal_node, size_type bits)
{
	Node* previous = nullptr;
	Node* current = nullptr;

	unsigned int local_offset = bits / g.size();
	do
	{
		unsigned int local_index = local_offset * g.size() + g.thread_rank();
		if (local_index < bits)
			previous = internal_node->nodes[local_index];
		else if (local_index == bits)
			current = internal_node->nodes[bits];

		current = reinterpret_cast<Node*>(g.shfl(reinterpret_cast<std::uintptr_t>(current), bits % g.size()));
		unsigned int warp_result = g.ballot(previous != nullptr);

		if (!warp_result)
		{
			if (local_offset == 0u)
				return { current, nullptr, -1, false };
			else
				--local_offset;
		}
		else
		{
			unsigned int pos = 31 - __clz(warp_result);
			previous = reinterpret_cast<Node*>(g.shfl(reinterpret_cast<std::uintptr_t>(previous), pos));
			return { current, previous, local_offset * g.size() + pos, true };
		}
	} while (local_offset != 0u);
	return { current, nullptr, -1, false };
}

template <typename Key, typename Value, unsigned int NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, unsigned int NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>
__device__ typename vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::PredecessorLeafInfo vEB<Key, Value, NUMBER_OF_CHILDREN_PER_NODE_POWER_OF_TWO, NUMBER_OF_BITS_PER_NODE_POWER_OF_TWO>::predecessor_leaf(threads g, LeafNode* leaf_node, key_type key)
{
	unsigned int local_bits = key & (NUMBER_OF_BITS_PER_NODE - 1u);
	unsigned int local_offset = local_bits / g.size();
	unsigned int local_bit = local_bits % g.size();
	bool has_predecessor = false;
	do
	{
		unsigned int local_index = local_offset + g.thread_rank();
		unsigned int msb = 0u;
		if (local_index < local_offset && leaf_node->bits[local_index])
		{
			auto data = leaf_node->bits[local_index];
			msb = 32u - __clz(data);
			has_predecessor = true;
		}
		else if (local_index == local_offset)
		{
			unsigned shift = 1u << local_bit;
			unsigned int mask = shift - 1u;
			unsigned int resulting_data = leaf_node->bits[local_index] & mask;
			msb = 31u - __clz(resulting_data);
			has_predecessor = resulting_data != 0u;
		}

		unsigned int warp_result = g.ballot(has_predecessor);
		if (!warp_result)
		{
			if (local_offset == 0u)
				return { {}, false };
			else
				--local_offset;
		}
		else
		{
			unsigned int winner_thid = __ffs(warp_result) - 1;
			msb = g.shfl(msb, winner_thid);
			key_type resulting_key = local_offset * g.size() + winner_thid * 32 + msb;
			key_type upper_bits = ~(NUMBER_OF_BITS_PER_NODE - 1u);
			resulting_key = resulting_key + (key & upper_bits);
			return { resulting_key, true };
		}

	} while (local_offset != 0u);
	return { {}, false };
}
