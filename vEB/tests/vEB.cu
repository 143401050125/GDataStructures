#include "hip/hip_runtime.h"
#include "vEB.cuh"
#include "Catch2/catch.hpp"
#include "cuda/api_wrappers.h"

#include "concurrent-xfasttrie-common.cuh"

#include <ctime>
#include <iostream>

using key_type = gpu::UInt32;
using mapped_type = int;
using vanEB = vEB<key_type, mapped_type, 6, 11>;

SCENARIO("VAN-EMDE-BOAS-TRIE", "[VEB]")
{
	unsigned int NUMBER_OF_BLOCKS = 64u;
	unsigned int NUMBER_OF_WARPS = 16u;
	const unsigned int memory_size_allocated = 1u << 31u;
	unsigned int to_insert = 1u << 15u;
	auto current_device = cuda::device::current::get();
	auto d_memory = cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated);
	auto d_allocator = cuda::memory::device::make_unique<allocator_type>(current_device);

	GIVEN("A vEB")
	{
		auto d_vEB = cuda::memory::device::make_unique<vanEB>(current_device);

		cuda::launch(initialize_allocator<vanEB>,
			{ 1u, NUMBER_OF_WARPS * 32u },
			d_allocator.get(), d_memory.get(), memory_size_allocated, d_vEB.get(), to_insert
		);

		WHEN("We add elements in random order")
		{
			THEN("It should be good")
			{
				std::cout << to_insert << std::endl;
				const std::clock_t begin_time = std::clock();
				cuda::launch(test_insert_random<vanEB>,
					{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
					d_vEB.get(), to_insert
				);
				cuda::device::current::get().synchronize();
				std::cout << float(std::clock() - begin_time) / CLOCKS_PER_SEC;
				fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(hipPeekAtLastError()), __FILE__, __LINE__);
				cuda::device::current::get().synchronize();
				/*cuda::launch(test_retrieve_size<vanEB>,
					{ 1u, 1u },
					d_vEB.get(), to_insert
				);
				cuda::device::current::get().synchronize();*/
				/*cuda::launch(test_post_condition<XFastTrie>,
					{ 1u, 32u },
					d_xfasttrie.get()
				);*/
			}
		}

		WHEN("We add elements in random order")
		{
			THEN("Ask for predecessors")
			{
				cuda::launch(test_predecessor_random<vanEB>,
					{ 1u, 32u },
					d_vEB.get(), to_insert
				);
				cuda::device::current::get().synchronize();
				fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(hipPeekAtLastError()), __FILE__, __LINE__);
				cuda::device::current::get().synchronize();
				/*cuda::launch(test_retrieve_size<XFastTrie>,
				{ 1u, 1u },
				d_xfasttrie.get(), to_insert
				);*/
				cuda::device::current::get().synchronize();
				/*cuda::launch(test_post_condition<XFastTrie>,
				{ 1u, 32u },
				d_xfasttrie.get()
				);*/
			}
		}
	}
}
