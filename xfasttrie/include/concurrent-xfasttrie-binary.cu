#include "hip/hip_runtime.h"
#include "concurrent-xfasttrie-binary.cuh"

#include "utility/limits.cuh"
#include "utility/print.cuh"

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::begin()
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::const_iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::begin() const
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::end()
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::const_iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::end() const
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::ConcurrentXFastTrieBinary(block_threads block, allocator_type& allocator, unsigned int expected_number_of_elements)
{
	unsigned int power_of_two;
	if (expected_number_of_elements == 0)
		power_of_two = 10u;
	else
		power_of_two = __ffs(expected_number_of_elements) - 1u;

	for (int rank = 0; rank != SUBRANK; ++rank)
	{
		unsigned int preallocate = rank < power_of_two ? 1u << (rank + 2u) : 1u << power_of_two;
		new (&m_maps[rank]) Keyset{ block, allocator, preallocate };
	}
	new (&m_bottom) Map{ block, allocator, 1u << power_of_two };

	if (block.thread_rank() == 0)
	{
		m_head = INVALID_PREDECESSOR();
		m_tail = INVALID_SUCCESSOR();
	}
	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::ConcurrentXFastTrieBinary(threads group, allocator_type& allocator, unsigned int expected_number_of_elements)
{
	unsigned int max_allocate = 1 << 19u;
	for (int rank = 0; rank != SUBRANK; ++rank)
	{
		unsigned int preallocate = rank < 17u ? 1u << (rank + 2u) : max_allocate;
		m_maps[rank] = Keyset{ group, allocator, preallocate };
	}
	m_bottom = Map{ group, allocator, max_allocate };

	m_head = INVALID_PREDECESSOR();
	m_tail = INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::clear(block_threads block)
{
	threads tile32 = cooperative_groups::tiled_partition<32>(block);

	if (block.thread_rank() < 32)
		clear(tile32);

	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::clear(threads group)
{
	for (int rank = 0; rank != SUBRANK; ++rank)
		m_maps[rank].clear(group);

	m_bottom.clear(group);

	m_head = INVALID_PREDECESSOR();
	m_tail = INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::find(key_type key)
{
	return m_bottom.find(key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::find(threads group, key_type key)
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::const_iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::find(threads group, key_type key) const
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::insert(threads group, key_type key, value_type value)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	auto u = binary_search(group, key);
	auto old_it = m_maps[u.second].end(); // since when we reallocate data, pointer will change

	size_type loop = u.second;
	while (loop != SUBRANK)
	{
		key_type bits = extract_i_upper_bits(key, loop);
		auto found_it = m_maps[loop].find(group, bits);
		auto end_it = m_maps[loop].end();
		if (found_it == end_it)
		{
			m_maps[loop].insert_or_update(gpu::make_pair<key_type, Children>(bits, { key, key }),
				[](Children& lhs, Children&& rhs) {
				lhs.minimal_left.min(rhs.minimal_left);
				lhs.maximal_right.min(rhs.maximal_right);
			});
		}
		++loop;
	}

	walk_up(group, key, u.second);

	auto old_minimal = u.first->second.minimal_left;
	auto old_maximal = u.first->second.maximal_right;

	if (!has_data())
	{
		auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, INVALID_PREDECESSOR(), INVALID_SUCCESSOR() }));
		key_type old_head = m_head.min(group, key);
		if (old_head != INVALID_PREDECESSOR())
			return insert_around(group, old_head, current_it);
		key_type old_tail = m_tail.max(group, key);
		if (old_tail != INVALID_SUCCESSOR())
			return insert_around(group, old_tail, current_it);
		post_condition(group);
		return current_it;
	}
	else
	{
		if (key < m_head)
		{
			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, INVALID_PREDECESSOR(), m_head }));
			return insert_around(group, m_head, current_it);
		}
		else if (key > m_tail)
		{
			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, m_tail, INVALID_SUCCESSOR() }));
			return insert_around(group, m_tail, current_it);
		}
		else
		{
			if (key < old_minimal)
			{
				Map_iterator successor_it = spinlock_for_value(group, old_minimal);
				auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, successor_it->second.predecessor, old_minimal }));
				return insert_around(group, old_minimal, current_it);
			}
			else if (key > old_maximal)
			{
				Map_iterator predecessor_it = spinlock_for_value(group, old_maximal);
				auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, old_maximal, predecessor_it->second.successor }));
				return insert_around(group, old_maximal, current_it);
			}
			else
			{
				/*if (group.thread_rank() == 3)
				{
				gpu::print(" > ", key, " o ", old_minimal, " p ", old_maximal, "\n");
				}
				group.sync();
				if (group.thread_rank() == 0)
				{
				gpu::print(" > ", key, " o ", old_minimal, " p ", old_maximal, "\n");
				}
				group.sync();

				ENSURE(false);*/
				return end();
			}
		}
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::size_type ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::maximal_size() const
{
	return 1 << RANK;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::predecessor(threads group, key_type key)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	if (!has_data())
		return end();

	if (key < m_head)
		return end();
	if (key >= m_tail)
		return m_bottom.find(group, m_tail);

	auto u = binary_search(group, key);
	if (key < u.first->second.minimal_left)
	{
		auto predecessor_it = m_bottom.find(group, u.first->second.minimal_left);
		return m_bottom.find(group, predecessor_it->second.predecessor);
	}
	else
		return m_bottom.find(group, u.first->second.maximal_right);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::size_type ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::size() const
{
	return m_bottom.size();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::successor(threads group, key_type key)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	if (!has_data())
		return end();

	if (key > m_tail)
		return end();
	if (key <= m_head)
		return m_bottom.find(group, m_head);

	auto u = binary_search(group, key);
	if (key < u.first->second.minimal_left)
		return m_bottom.find(group, u.first->second.minimal_left);
	else
	{
		auto predecessor_it = m_bottom.find(group, u.first->second.minimal_left);
		return m_bottom.find(group, predecessor_it->second.successor);
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::debug() const
{
	using gpu::print;
	for (int i = 0; i != SUBRANK; ++i)
	{
		print("HashMap (", i, "): ");
		const auto& map = m_maps[i];
		map.for_each([](const auto& it) {
			print("{", key_type(it->first), "|", key_type(it->second.minimal_left), ", ", key_type(it->second.maximal_right), "}");
		});
		print("\n");
	}

	print("Bottom: ");
	m_bottom.for_each([this](const auto& it) {
		auto& value = it->second;
		if (value.predecessor != INVALID_PREDECESSOR() && value.successor != INVALID_SUCCESSOR())
			print("{", key_type(it->first), "|", value.value, "=>[", key_type(value.predecessor), "|", key_type(value.successor), "]}");
		else if (value.predecessor != INVALID_PREDECESSOR())
			print("{", key_type(it->first), "|", value.value, "=>[", key_type(value.predecessor), "|#]}");
		else if (value.successor != INVALID_SUCCESSOR())
			print("{", key_type(it->first), "|", value.value, "=>[#|", key_type(value.successor), "]}");
		else
			print("{", key_type(it->first), "|", value.value, "=>[#|#]}");
	});

	print("\nHead/Tail: ");
	if (m_head != INVALID_PREDECESSOR())
		print("Head: ", key_type(m_head), " ");
	if (m_tail != INVALID_SUCCESSOR())
		print("Tail: ", key_type(m_tail), " ");
	print("\n");
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ auto ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::binary_search(threads group, key_type key) -> gpu::pair<keyset_iterator, size_type>
{
	int low = 0;
	int high = SUBRANK - 1;
	keyset_iterator u = m_maps[low].end();
	while (low <= high)
	{
		int mid = (low + high) / 2;
		key_type bits = extract_i_upper_bits(key, mid);
		auto v = m_maps[mid].find(group, bits);
		if (v == m_maps[mid].end())
		{
			high = mid - 1;
		}
		else
		{
			u = v;
			low = mid + 1;
		}
	}

	return { u, low };
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::key_type ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::extract_i_upper_bits(key_type key, int number_of_bits) const
{
	key_type result = key >> (SUBRANK - number_of_bits);
	return result;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::get_predecessor(threads group, key_type key, keyset_iterator u)
{
	auto& value = u->second;
	return m_bottom.find(group, value.minimal_left);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::get_successor(threads group, key_type key, keyset_iterator u)
{
	auto& value = u->second;
	return m_bottom.find(group, value.maximal_right);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ bool ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::has_data() const
{
	return m_head != INVALID_PREDECESSOR() && m_tail != INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::key_type ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::INVALID_PREDECESSOR() const
{
	return gpu::numeric_limits<key_type>::max();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::key_type ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::INVALID_SUCCESSOR() const
{
	return gpu::numeric_limits<key_type>::min();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::insert_around(threads group, const key_type& key, Map_iterator to_insert)
{
	while (true)
	{
		Map_iterator predecessor = m_bottom.end();
		Map_iterator successor = m_bottom.end();

		predecessor = spinlock_for_value(group, key);
		while (predecessor->second.predecessor != INVALID_PREDECESSOR() && predecessor->second.predecessor > to_insert->first)
		{
			successor = predecessor;
			predecessor = spinlock_for_value(group, predecessor->second.predecessor);
		}
		while (predecessor->second.successor != INVALID_SUCCESSOR() && predecessor->second.successor < to_insert->first)
		{
			successor = spinlock_for_value(group, predecessor->second.successor);
			predecessor = successor;
		}

		if (predecessor->second.predecessor == INVALID_PREDECESSOR())
		{
			if (predecessor->first < to_insert->first)
			{
				to_insert->second.predecessor = predecessor->first;
				to_insert->second.successor = predecessor->second.successor;

				if (predecessor->second.successor.compare_and_swap(group, INVALID_SUCCESSOR(), to_insert->first) == INVALID_SUCCESSOR())
				{
					m_tail.max(group, to_insert->first);
					return to_insert;
				}
			}
			else
			{
				to_insert->second.predecessor = predecessor->second.predecessor;
				to_insert->second.successor = predecessor->first;

				if (predecessor->second.predecessor.compare_and_swap(group, INVALID_PREDECESSOR(), to_insert->first) == INVALID_PREDECESSOR())
				{
					m_head.min(group, to_insert->first);
					return to_insert;
				}
			}
		}
		else
		{
			to_insert->second.predecessor = predecessor->first;
			to_insert->second.successor = predecessor->second.successor;

			if (predecessor->second.successor == INVALID_SUCCESSOR())
			{
				if (predecessor->second.successor.compare_and_swap(group, INVALID_SUCCESSOR(), to_insert->first) == INVALID_SUCCESSOR())
				{
					m_tail.max(group, to_insert->first);
					return to_insert;
				}
			}

			successor = spinlock_for_value(group, predecessor->second.successor);
			key_type old_successor = predecessor->second.successor;
			key_type old_predecessor = successor->second.predecessor;
			if (predecessor->second.successor.compare_and_swap(group, old_successor, to_insert->first) == old_successor)
			{
				if (successor->second.predecessor.compare_and_swap(group, old_predecessor, to_insert->first) == old_predecessor)
				{
					return to_insert;
				}
			}
		}
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::Map_iterator ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::spinlock_for_value(threads group, const key_type& key)
{
	Map_iterator result_it;
	do
	{
		result_it = m_bottom.find(group, key);
	} while (result_it == m_bottom.end());
	return result_it;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::walk_up(threads group, key_type key, size_type from)
{
	while (from != 0)
	{
		--from;
		key_type bits = extract_i_upper_bits(key, from);
		auto it = m_maps[from].find(group, bits);
		it->second.minimal_left.min(key);
		it->second.maximal_right.max(key);
		if (it->second.minimal_left != key && it->second.maximal_right != key)
			return;
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void ConcurrentXFastTrieBinary<Key, Value, UNIVERSE>::post_condition(threads group)
{
#ifdef GPU_XFASTTRIE_DEBUG
	for (int i = 0; i != SUBRANK; ++i)
	{
		auto& map = m_maps[i];
		for (auto it = map.begin(); it != map.end(); ++it)
		{
			if (i + 1 < SUBRANK)
			{
				auto left_child = m_maps[i + 1].find(group, it->first << 1);
				auto right_child = m_maps[i + 1].find(group, (it->first << 1) | 1);
				ENSURE(left_child != m_maps[i + 1].end() || right_child != m_maps[i + 1].end());
			}
		}
	}

	if (m_head != INVALID_PREDECESSOR())
	{
		ENSURE(m_head == m_tail || m_tail != INVALID_SUCCESSOR());
		auto buffer_start = &(*m_bottom.end()) - m_bottom.capacity();
		auto head = m_bottom.find(group, m_head);
		auto tail = m_bottom.find(group, m_tail);
		ENSURE(&(*head) >= buffer_start && &(*head) < &(*m_bottom.end()));
		ENSURE(&(*tail) >= buffer_start && &(*tail) < &(*m_bottom.end()));
	}
	else
		ENSURE(m_tail == INVALID_SUCCESSOR());

	if (has_data())
	{
		for (auto it = m_bottom.begin(); it != m_bottom.end(); ++it)
		{
			if (it->first != m_head)
			{
				auto predecessor_it = m_bottom.find(group, it->second.predecessor);
				ENSURE(it->first > predecessor_it->first);
				ENSURE(it->first == predecessor_it->second.successor);
			}
			if (it->first != m_tail)
			{
				auto successor_it = m_bottom.find(group, it->second.successor);
				ENSURE(it->first < successor_it->first);
				ENSURE(it->first == successor_it->second.predecessor);
			}
		}
	}
#endif // GPU_XFASTTRIE_DEBUG
}
