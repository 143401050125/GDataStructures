#include "hip/hip_runtime.h"
#include "concurrent-xfasttrie-group-parallel.cuh"

#include "algorithms/find.cuh"
#include "utility/limits.cuh"
#include "utility/print.cuh"
#include "utility/warp_value.cuh"

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::begin()
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::const_iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::begin() const
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::end()
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::const_iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::end() const
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::ConcurrentXFastTrieGroupParallel(block_threads block, allocator_type& allocator, unsigned int expected_number_of_elements)
{
	unsigned int power_of_two;
	if (expected_number_of_elements == 0)
		power_of_two = 10u;
	else
		power_of_two = __ffs(expected_number_of_elements) - 1u;

	for (int rank = 0; rank != NUMBER_OF_KEYSETS; ++rank)
	{
		unsigned int preallocate = rank * (GROUP_SIZE + 1u) < power_of_two ? 1u << (rank * (GROUP_SIZE + 1u) + 2u) : 1u << power_of_two;
		new (&m_maps[rank]) Keyset{ block, allocator, preallocate };
	}
	new (&m_bottom) Map{ block, allocator, 1u << power_of_two };

	if (block.thread_rank() == 0)
	{
		m_head.store_unatomically(INVALID_PREDECESSOR());
		m_tail.store_unatomically(INVALID_SUCCESSOR());
	}
	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::ConcurrentXFastTrieGroupParallel(threads group, allocator_type& allocator, unsigned int expected_number_of_elements)
{
	unsigned int power_of_two;
	if (expected_number_of_elements == 0)
		power_of_two = 10u;
	else
		power_of_two = __ffs(expected_number_of_elements) - 1u;

	for (int rank = 0; rank != NUMBER_OF_KEYSETS; ++rank)
	{
		unsigned int preallocate = rank * (GROUP_SIZE + 1u) < power_of_two ? 1u << (rank * (GROUP_SIZE + 1u) + 2u) : 1u << power_of_two;
		new (&m_maps[rank]) Keyset{ group, allocator, preallocate };
	}
	new (&m_bottom) Map{ group, allocator, 1u << power_of_two };

	if (group.thread_rank() == 0)
	{
		m_head.store_unatomically(INVALID_PREDECESSOR());
		m_tail.store_unatomically(INVALID_SUCCESSOR());
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::clear(block_threads block)
{
	threads tile32 = cooperative_groups::tiled_partition<32>(block);

	if (block.thread_rank() < 32)
		clear(tile32);

	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::clear(threads group)
{
	for (int rank = 0; rank != NUMBER_OF_KEYSETS; ++rank)
		m_maps[rank].clear(group);

	m_bottom.clear(group);

	m_head.store_unatomically(INVALID_PREDECESSOR());
	m_tail.store_unatomically(INVALID_SUCCESSOR());
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::find(key_type key)
{
	key &= (1u << UNIVERSE) - 1u;
	return m_bottom.find(key);
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::find(threads group, key_type key)
{
	key &= (1u << UNIVERSE) - 1u;
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::const_iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::find(threads group, key_type key) const
{
	key &= (1u << UNIVERSE) - 1u;
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::insert(threads group, key_type key, mapped_type value)
{
	key &= (1u << UNIVERSE) - 1u;

	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	auto old_values = find_or_update(group, key);
	key_type old_minimal = old_values.first;
	key_type old_maximal = old_values.second;

	gpu::warp_value::ensure(group, old_minimal);
	gpu::warp_value::ensure(group, old_maximal);

	if (!has_data())
	{
		auto current_it = insert_at_bottom(group, key, value, INVALID_PREDECESSOR(), INVALID_SUCCESSOR());
		key_type old_head = m_head.compare_and_swap(INVALID_PREDECESSOR(), key);
		if (old_head == INVALID_PREDECESSOR())
		{
			key_type old_tail = m_tail.compare_and_swap(INVALID_SUCCESSOR(), key);
			if (old_tail == INVALID_SUCCESSOR())
			{
				return current_it;
			}

			if (key > old_tail)
				return insert_after(group, old_tail, current_it);
			else
				return insert_before(group, old_tail, current_it);
		}

		if (key > old_head)
			return insert_after(group, old_head, current_it);
		else if (key < old_head)
			return insert_before(group, old_head, current_it);
		else
			return current_it;
	}
	else
	{
		if (key < m_head)
		{
			auto current_it = insert_at_bottom(group, key, value, INVALID_PREDECESSOR(), m_head);
			return insert_before(group, m_head, current_it);
		}
		else if (key > m_tail)
		{
			auto current_it = insert_at_bottom(group, key, value, m_tail, INVALID_SUCCESSOR());
			return insert_after(group, m_tail, current_it);
		}
		else
		{
			if (key < old_minimal)
			{
				//Map_iterator successor_it = spinlock_for_value(group, old_minimal);
				auto current_it = insert_at_bottom(group, key, value, old_minimal, old_minimal);
				return insert_before(group, old_minimal, current_it);
			}
			else if (key > old_maximal)
			{
				//Map_iterator predecessor_it = spinlock_for_value(group, old_maximal);
				auto current_it = insert_at_bottom(group, key, value, old_maximal, old_maximal);
				return insert_after(group, old_maximal, current_it);
			}
			else
			{
				auto current_it = insert_at_bottom(group, key, value, old_minimal, old_maximal);
				return insert_between(group, old_minimal, old_maximal, current_it);
			}
		}
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::size_type ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::maximal_size() const
{
	return 1 << RANK;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::predecessor(threads group, key_type key)
{
	if (!has_data())
		return end();

	if (key < m_head)
		return end();
	if (key >= m_tail)
		return m_bottom.find(group, m_tail);

	BinarySearchResult u = binary_search(group, key);
	if (u.bottom_it == m_bottom.end())
	{
		const keyset_iterator& it = u.it;
		unsigned int index = u.index;
		if (key < it->second.minimal_left[index])
		{
			auto predecessor_it = m_bottom.find(group, it->second.minimal_left[index]);
			return m_bottom.find(group, predecessor_it->second.predecessor);
		}
		else
			return m_bottom.find(group, it->second.maximal_right[index]);
	}
	else
		return u.bottom_it;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::size_type ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::size() const
{
	return m_bottom.size();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::successor(threads group, key_type key)
{
	if (!has_data())
		return end();

	if (key > m_tail)
		return end();
	if (key <= m_head)
		return m_bottom.find(group, m_head);

	BinarySearchResult u = binary_search(group, key);
	if (u.bottom_it == m_bottom.end())
	{
		const keyset_iterator& it = u.it;
		unsigned int index = u.index;

		if (key < it->second.minimal_left[index])
			return m_bottom.find(group, it->second.minimal_left[index]);
		else
		{
			auto predecessor_it = m_bottom.find(group, it->second.minimal_left[index]);
			return m_bottom.find(group, predecessor_it->second.successor);
		}
	}
	else
		return u.bottom_it;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::debug() const
{
	using gpu::print;
	for (int i = 0; i != NUMBER_OF_KEYSETS; ++i)
	{
		print("HashMap (", i, "): ");
		const auto& map = m_maps[i];
		map.for_each([this, i, NUMBER_OF_KEYSETS = NUMBER_OF_KEYSETS, TAIL_GROUP_SIZE = TAIL_GROUP_SIZE, GROUP_SIZE = GROUP_SIZE](const auto& it) {
			print("{", key_type(it->first), "|");
			print("0: (", key_type(it->second.minimal_left[0]), ", ", key_type(it->second.maximal_right[0]), ")");
			unsigned int group_size = (i == NUMBER_OF_KEYSETS - 1u) ? TAIL_GROUP_SIZE : GROUP_SIZE;
			if (group_size)
			{
				for (unsigned int g = 0u; g != group_size; ++g)
				{
					print(" ", g + 1, ": ");
					unsigned int offset = (1u << (g + 1u)) - 1u;
					for (unsigned int i = 0u; i != 1u << (g + 1u); ++i)
					{
						print("(", key_type(it->second.minimal_left[offset + i]), ", ", key_type(it->second.maximal_right[offset + i]), ")");
					}
				}
			}
		});
		print("\n");
	}

	print("Bottom: ");
	m_bottom.for_each([this](const auto& it) {
		auto& value = it->second;
		if (value.predecessor != INVALID_PREDECESSOR() && value.successor != INVALID_SUCCESSOR())
			print("{", key_type(it->first), "|", value.value, "=>[", key_type(value.predecessor), "|", key_type(value.successor), "]}");
		else if (value.predecessor != INVALID_PREDECESSOR())
			print("{", key_type(it->first), "|", value.value, "=>[", key_type(value.predecessor), "|#]}");
		else if (value.successor != INVALID_SUCCESSOR())
			print("{", key_type(it->first), "|", value.value, "=>[#|", key_type(value.successor), "]}");
		else
			print("{", key_type(it->first), "|", value.value, "=>[#|#]}");
	});

	print("\nHead/Tail: ");
	if (m_head != INVALID_PREDECESSOR())
		print("Head: ", key_type(m_head), " ");
	if (m_tail != INVALID_SUCCESSOR())
		print("Tail: ", key_type(m_tail), " ");
	print("\n");
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ auto ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::binary_search(threads group, key_type key) -> BinarySearchResult
{
	auto thid = group.thread_rank();
	bool has_value = false;
	unsigned int index;
	keyset_iterator found_it;
	Map_iterator bottom_it;
	if (thid < NUMBER_OF_KEYSETS)
	{
		key_type group_bits;
		key_type tail_bits;
		if (thid == NUMBER_OF_KEYSETS - 1u)
		{
			group_bits = key >> (TAIL_GROUP_SIZE + 1u);
			tail_bits = key & ((1u << (TAIL_GROUP_SIZE + 1u)) - 1u);
			tail_bits >>= 1u; // Avoid last bit
			found_it = m_maps[thid].find(group_bits);
		}
		else
		{
			unsigned int shift_value = (TAIL_GROUP_SIZE + 1u) + (NUMBER_OF_KEYSETS - thid - 2u) * (GROUP_SIZE + 1) + 1u; // Don't forget last bit
			group_bits = key >> shift_value;
			tail_bits = group_bits & ((1u << GROUP_SIZE) - 1u);
			group_bits = group_bits >> GROUP_SIZE; // We get the upper bits
			found_it = m_maps[thid].find(group_bits);
		}

		// We try to find out the last place such that T T T F <- We want the third T.
		has_value = found_it != m_maps[thid].end();

		if (has_value)
			index = get_highest_index(tail_bits, found_it->second, thid);
	}
	else if (thid == NUMBER_OF_KEYSETS)
	{
		bottom_it = m_bottom.find(key);
	}

	unsigned int matching_bits = group.ballot(has_value);
	if (matching_bits)
	{
		// The idea is that we want the min, max of the lowest node in the tree where there is data
		unsigned int insert_update_separation = 31u - __clz(matching_bits);
		index = group.shfl(index, insert_update_separation);
		found_it.shfl(group, insert_update_separation);
	}
	bottom_it.shfl(group, NUMBER_OF_KEYSETS);
	return { found_it, bottom_it, index };
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::key_type ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::extract_i_upper_bits(key_type key, int number_of_bits) const
{
	key_type result = key >> (SUBRANK - number_of_bits);
	return result;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ auto ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::find_or_update(threads group, key_type key) -> gpu::pair<key_type, key_type>
{
	auto thid = group.thread_rank();

	key_type old_minimum = INVALID_PREDECESSOR();
	key_type old_maximum = INVALID_SUCCESSOR();
	bool has_value = false;
	if (thid < NUMBER_OF_KEYSETS)
	{
		key_type group_bits;
		key_type tail_bits;
		keyset_iterator found_it;
		if (thid == NUMBER_OF_KEYSETS - 1u)
		{
			group_bits = key >> (TAIL_GROUP_SIZE + 1u);
			tail_bits = key & ((1u << (TAIL_GROUP_SIZE + 1u)) - 1u);
			tail_bits >>= 1u; // Avoid last bit
		}
		else
		{
			unsigned int shift_value = (TAIL_GROUP_SIZE + 1u) + (NUMBER_OF_KEYSETS - thid - 2u) * (GROUP_SIZE + 1) + 1u; // Don't forget last bit
			group_bits = key >> shift_value;
			tail_bits = group_bits & ((1u << GROUP_SIZE) - 1u);
			group_bits = group_bits >> GROUP_SIZE; // We get the upper bits
		}
		found_it = m_maps[thid].find(group_bits);

		// We try to find out the last place such that T T T F <- We want the third T.
		has_value = found_it != m_maps[thid].end();

		unsigned int matching_bits = group.ballot(has_value);
		unsigned int insert_update_separation = 32u - __clz(matching_bits);

		if (has_value)
		{
			unsigned int i = get_highest_index(tail_bits, found_it->second, thid);
			old_minimum = found_it->second.minimal_left[i];
			old_maximum = found_it->second.maximal_right[i];
		}

		if (thid >= insert_update_separation && !has_value) // Not just, should be related to group
		{
			m_maps[thid].insert_or_update(gpu::make_pair<key_type, Child>(group_bits, make_children(tail_bits, key, thid)),
				[thid, NUMBER_OF_KEYSETS = NUMBER_OF_KEYSETS, TAIL_GROUP_SIZE = TAIL_GROUP_SIZE, GROUP_SIZE = GROUP_SIZE]
			(Child& lhs, Child&& rhs) {
				unsigned int upper_bound = thid == NUMBER_OF_KEYSETS - 1u ? TAIL_GROUP_SIZE : GROUP_SIZE;
				for (unsigned int i = 0u; i != (1u << (upper_bound + 1u)) - 1u; ++i)
				{
					if (rhs.minimal_left[i] < lhs.minimal_left[i])
						lhs.minimal_left[i].min(rhs.minimal_left[i]);
					if (rhs.maximal_right[i] > lhs.maximal_right[i])
						lhs.maximal_right[i].max(rhs.maximal_right[i]);
				}
			});
		}
		else
		{
			while (found_it == m_maps[thid].end()) // Should be unlikely
				found_it = m_maps[thid].find(group_bits);

			update_key(found_it->second, tail_bits, key, thid);
		}
	}

	unsigned int matching_bits = group.ballot(has_value);
	if (matching_bits)
	{
		// The idea is that we want the min, max of the lowest node in the tree where there is data
		unsigned int insert_update_separation = 31u - __clz(matching_bits);
		old_minimum = group.shfl(old_minimum, insert_update_separation);
		old_maximum = group.shfl(old_maximum, insert_update_separation);
	}

	return { old_minimum, old_maximum };
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ unsigned int ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::get_highest_index(key_type tail_bits, const Child& value, unsigned int thid)
{
	unsigned int upper_bound = thid == NUMBER_OF_KEYSETS - 1u ? TAIL_GROUP_SIZE : GROUP_SIZE;
	unsigned int offset = (1u << upper_bound) - 1u;
	while (offset)
	{
		unsigned int position = offset + tail_bits;
		if (value.minimal_left[position] != INVALID_PREDECESSOR())
			return position;
		offset = offset >> 1u;
		tail_bits = tail_bits >> 1u;
	}
	return 0u;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ auto ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::make_children(key_type tail_bits, const key_type& key, unsigned int thid) -> Child
{
	Child child;
	for (unsigned int i = 0u; i != (1u << (GROUP + 1u)) - 1u; ++i)
	{
		child.minimal_left[i].store_unatomically(INVALID_PREDECESSOR());
		child.maximal_right[i].store_unatomically(INVALID_SUCCESSOR());
	}

	unsigned int upper_bound = thid == NUMBER_OF_KEYSETS - 1u ? TAIL_GROUP_SIZE : GROUP_SIZE;
	unsigned int offset = (1u << upper_bound) - 1u;
	while (offset)
	{
		unsigned int position = offset + tail_bits;
		child.minimal_left[position].store_unatomically(key);
		child.maximal_right[position].store_unatomically(key);
		offset = offset >> 1u;
		tail_bits = tail_bits >> 1u;
	}
	child.minimal_left[0u].store_unatomically(key);
	child.maximal_right[0u].store_unatomically(key);
	return child;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::update_key(Child& child, key_type tail_bits, const key_type& key, unsigned int thid)
{
	unsigned int upper_bound = thid == NUMBER_OF_KEYSETS - 1u ? TAIL_GROUP_SIZE : GROUP_SIZE;
	unsigned int offset = (1u << upper_bound) - 1u;
	while (offset)
	{
		unsigned int position = offset + tail_bits;
		if (key < child.minimal_left[position])
			child.minimal_left[position].min(key);
		if (key > child.maximal_right[position])
			child.maximal_right[position].max(key);
		offset = offset >> 1u;
		tail_bits = tail_bits >> 1u;
	}
	if (key < child.minimal_left[0u])
		child.minimal_left[0u].min(key);
	if (key > child.maximal_right[0u])
		child.maximal_right[0u].max(key);
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::Map_iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::insert_at_bottom(threads group, key_type key, mapped_type value, key_type predecessor, key_type successor)
{
	return m_bottom.insert_or_update(group, gpu::make_pair<key_type, Node>(key, { value, predecessor, successor }), [](auto& lhs, auto&& rhs) {
		lhs.value = std::move(rhs.value);
	});
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ bool ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::has_data() const
{
	return m_head != INVALID_PREDECESSOR() || m_tail != INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::key_type ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::INVALID_PREDECESSOR() const
{
	return gpu::numeric_limits<key_type>::max();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::key_type ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::INVALID_SUCCESSOR() const
{
	return gpu::numeric_limits<key_type>::min();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::insert_after(threads group, key_type key, Map_iterator to_insert)
{
	auto predecessor_it = spinlock_for_value(group, key);
	if (predecessor_it == m_bottom.end())
		return to_insert;

	int i = 0;
	//while (true)
	{
		auto last_predecessor_it = end();
		while (predecessor_it->second.successor != INVALID_SUCCESSOR() && predecessor_it->second.successor < to_insert->first && predecessor_it != last_predecessor_it)
		{
			last_predecessor_it = predecessor_it;
			predecessor_it = spinlock_for_value(group, predecessor_it->second.successor);
			++i;

			if (predecessor_it == m_bottom.end() || i > 20)
				return to_insert;
		}

		if (predecessor_it->second.successor == INVALID_SUCCESSOR())
		{
			to_insert->second.predecessor.store(group, predecessor_it->first);
			to_insert->second.successor.store(group, INVALID_SUCCESSOR());

			if (predecessor_it->second.successor.compare_and_swap(group, INVALID_SUCCESSOR(), to_insert->first) == INVALID_SUCCESSOR())
			{
				m_tail.max(group, to_insert->first);
				return to_insert;
			}

			//continue;
			return to_insert;
		}

		if (predecessor_it == m_bottom.end() || predecessor_it->second.successor == to_insert->first)
			return to_insert;

#ifdef GPU_XFASTTRIE_DEBUG
		ENSURE(predecessor_it->second.successor != INVALID_SUCCESSOR());
#endif // GPU_XFASTTRIE_DEBUG
		auto successor_it = spinlock_for_value(group, predecessor_it->second.successor);

		if (successor_it == m_bottom.end() || successor_it->second.predecessor == to_insert->first)
			return to_insert;

		key_type old_successor = predecessor_it->second.successor;
		key_type old_predecessor = successor_it->second.predecessor;

#ifdef GPU_XFASTTRIE_DEBUG
		ENSURE(old_successor >= to_insert->first && old_predecessor <= to_insert->first);
#endif // GPU_XFASTTRIE_DEBUG

		to_insert->second.predecessor.store(group, predecessor_it->first);
		to_insert->second.successor.store(group, successor_it->first);

		if (predecessor_it->second.successor.compare_and_swap(group, old_successor, to_insert->first) == old_successor)
		{
			if (successor_it->second.predecessor.compare_and_swap(group, old_predecessor, to_insert->first) == old_predecessor)
			{
				return to_insert;
			}
		}
		return to_insert;
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::insert_before(threads group, key_type key, Map_iterator to_insert)
{
	auto successor_it = spinlock_for_value(group, key);
	if (successor_it == m_bottom.end())
		return to_insert;

	int i = 0;
	//while (true)
	{
		auto last_successor_it = end();
		while (successor_it->second.predecessor != INVALID_PREDECESSOR() && successor_it->second.predecessor > to_insert->first && successor_it != last_successor_it)
		{
			last_successor_it = successor_it;
			successor_it = spinlock_for_value(group, successor_it->second.predecessor);
			++i;

			if (successor_it == m_bottom.end() || i > 20)
				return to_insert;
		}

		if (successor_it->second.predecessor == INVALID_PREDECESSOR())
		{
			to_insert->second.predecessor.store(group, INVALID_PREDECESSOR());
			to_insert->second.successor.store(group, successor_it->first);

			if (successor_it->second.predecessor.compare_and_swap(group, INVALID_PREDECESSOR(), to_insert->first) == INVALID_PREDECESSOR())
			{
				m_head.min(group, to_insert->first);
				return to_insert;
			}

			//continue;
			return to_insert;
		}

		if (successor_it == m_bottom.end() || successor_it->second.predecessor == to_insert->first)
			return to_insert;

#ifdef GPU_XFASTTRIE_DEBUG
		ENSURE(successor_it->second.predecessor != INVALID_PREDECESSOR());
#endif // GPU_XFASTTRIE_DEBUG
		auto predecessor_it = spinlock_for_value(group, successor_it->second.predecessor);
		if (predecessor_it == m_bottom.end())
			return to_insert;

		key_type old_successor = predecessor_it->second.successor;
		key_type old_predecessor = successor_it->second.predecessor;

#ifdef GPU_XFASTTRIE_DEBUG
		ENSURE(old_successor >= to_insert->first && old_predecessor <= to_insert->first);
#endif // GPU_XFASTTRIE_DEBUG

		to_insert->second.predecessor.store(group, predecessor_it->first);
		to_insert->second.successor.store(group, successor_it->first);

		if (predecessor_it->second.successor.compare_and_swap(group, old_successor, to_insert->first) == old_successor)
		{
			if (successor_it->second.predecessor.compare_and_swap(group, old_predecessor, to_insert->first) == old_predecessor)
			{
				return to_insert;
			}
		}
		return to_insert;
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::insert_between(threads group, key_type predecessor_it_key, key_type successor_it_key, Map_iterator to_insert)
{
	auto predecessor_it = spinlock_for_value(group, predecessor_it_key);
	auto successor_it = spinlock_for_value(group, successor_it_key);

	if (predecessor_it == m_bottom.end() || successor_it == m_bottom.end())
		return to_insert;

	int i = 0;
	//while (true)
	{
		while (predecessor_it->second.successor < to_insert->first)
		{
			predecessor_it = spinlock_for_value(group, predecessor_it->second.successor);
			++i;

			if (predecessor_it == m_bottom.end() || i > 20)
				return to_insert;
		}

		while (successor_it->second.predecessor > to_insert->first)
		{
			successor_it = spinlock_for_value(group, successor_it->second.predecessor);
			++i;

			if (successor_it == m_bottom.end() || i > 20)
				return to_insert;
		}

		key_type old_successor = predecessor_it->second.successor;
		key_type old_predecessor = successor_it->second.predecessor;

#ifdef GPU_XFASTTRIE_DEBUG
		ENSURE(old_successor >= to_insert->first && old_predecessor <= to_insert->first);
#endif // GPU_XFASTTRIE_DEBUG

		to_insert->second.predecessor.store(group, predecessor_it->first);
		to_insert->second.successor.store(group, successor_it->first);

		if (predecessor_it->second.successor.compare_and_swap(group, old_successor, to_insert->first) == old_successor)
		{
			if (successor_it->second.predecessor.compare_and_swap(group, old_predecessor, to_insert->first) == old_predecessor)
			{
				return to_insert;
			}
		}
		return to_insert;
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::Map_iterator ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::spinlock_for_value(threads group, const key_type& key)
{
	int i = 0;
	Map_iterator result_it;
	do
	{
		result_it = m_bottom.find(group, key);
		++i;
		if (i > 32)
			return m_bottom.end();
	} while (result_it == m_bottom.end());
	return result_it;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void ConcurrentXFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::post_condition(threads group)
{
#ifdef GPU_XFASTTRIE_DEBUG
	if (m_head != INVALID_PREDECESSOR())
	{
		ENSURE(m_head == m_tail || m_tail != INVALID_SUCCESSOR());
		auto buffer_start = &(*m_bottom.end()) - m_bottom.capacity();
		auto head = m_bottom.find(group, m_head);
		auto tail = m_bottom.find(group, m_tail);
		ENSURE(&(*head) >= buffer_start && &(*head) < &(*m_bottom.end()));
		ENSURE(&(*tail) >= buffer_start && &(*tail) < &(*m_bottom.end()));
	}
	else
		ENSURE(m_tail == INVALID_SUCCESSOR());

	/*if (has_data())
	{
		m_bottom.for_each([this, &group](const auto& it) {
			if (it->first != m_head)
			{
				auto predecessor_it = m_bottom.find(group, it->second.predecessor);
				ENSURE(it->first > predecessor_it->first);
				ENSURE(it->first == predecessor_it->second.successor);
			}
			if (it->first != m_tail)
			{
				auto successor_it = m_bottom.find(group, it->second.successor);
				ENSURE(it->first < successor_it->first);
				ENSURE(it->first == successor_it->second.predecessor);
			}
		});
	}*/
#endif // GPU_XFASTTRIE_DEBUG
}
