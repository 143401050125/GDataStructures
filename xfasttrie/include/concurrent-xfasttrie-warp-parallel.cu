#include "hip/hip_runtime.h"
#include "concurrent-xfasttrie-warp-parallel.cuh"

#include "algorithms/find.cuh"
#include "utility/print.cuh"
#include "utility/warp_value.cuh"

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::begin()
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::const_iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::begin() const
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::end()
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::const_iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::end() const
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::ConcurrentXFastTrieWarpParallel(block_threads block, allocator_type& allocator, unsigned int expected_number_of_elements)
{
	unsigned int power_of_two;
	if (expected_number_of_elements == 0)
		power_of_two = 10u;
	else
		power_of_two = __ffs(expected_number_of_elements) - 1u;

	for (int rank = 0; rank != SUBRANK; ++rank)
	{
		unsigned int preallocate = rank < power_of_two ? 1u << (rank + 1u) : 1u << power_of_two;
		new (&m_maps[rank]) Keyset{ block, allocator, preallocate };
	}
	new (&m_bottom) Map{ block, allocator, 1u << power_of_two };

	if (block.thread_rank() == 0)
	{
		m_head.store_unatomically(INVALID_PREDECESSOR());
		m_tail.store_unatomically(INVALID_SUCCESSOR());
	}
	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::ConcurrentXFastTrieWarpParallel(threads group, allocator_type& allocator, unsigned int expected_number_of_elements)
{
	unsigned int power_of_two;
	if (expected_number_of_elements == 0)
		power_of_two = 10u;
	else
		power_of_two = __ffs(expected_number_of_elements) - 1u;

	for (int rank = 0; rank != SUBRANK; ++rank)
	{
		unsigned int preallocate = rank < power_of_two ? 1u << (rank + 1u) : 1u << power_of_two;
		new (&m_maps[rank]) Keyset{ block, allocator, preallocate };
	}
	new (&m_bottom) Map{ block, allocator, 1u << power_of_two };

	m_head.store_unatomically(INVALID_PREDECESSOR());
	m_tail.store_unatomically(INVALID_SUCCESSOR());
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::clear(block_threads block)
{
	threads tile32 = cooperative_groups::tiled_partition<32>(block);

	if (block.thread_rank() < 32)
		clear(tile32);

	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::clear(threads group)
{
	for (int rank = 0; rank != SUBRANK; ++rank)
		m_maps[rank].clear(group);

	m_bottom.clear(group);

	m_head.store_unatomically(INVALID_PREDECESSOR());
	m_tail.store_unatomically(INVALID_SUCCESSOR());
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::find(key_type key)
{
	key &= (1u << UNIVERSE) - 1u;
	return m_bottom.find(key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::find(threads group, key_type key)
{
	key &= (1u << UNIVERSE) - 1u;
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::const_iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::find(threads group, key_type key) const
{
	key &= (1u << UNIVERSE) - 1u;
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::insert(threads group, key_type key, mapped_type value)
{
	key &= (1u << UNIVERSE) - 1u;

	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	auto old_values = find_or_update(group, key);
	key_type old_minimal = old_values.first;
	key_type old_maximal = old_values.second;

	gpu::warp_value::ensure(group, old_minimal);
	gpu::warp_value::ensure(group, old_maximal);

	if (!has_data())
	{
		auto current_it = insert_at_bottom(group, key, value, INVALID_PREDECESSOR(), INVALID_SUCCESSOR());
		key_type old_head = m_head.compare_and_swap(group, INVALID_PREDECESSOR(), key);
		if (old_head == INVALID_PREDECESSOR())
		{
			key_type old_tail = m_tail.compare_and_swap(group, INVALID_SUCCESSOR(), key);
			if (old_tail == INVALID_SUCCESSOR())
			{
				return current_it;
			}

			if (key > old_tail)
				return insert_after(group, old_tail, current_it);
			else
				return insert_before(group, old_tail, current_it);
		}

		if (key > old_head)
			return insert_after(group, old_head, current_it);
		else if (key < old_head)
			return insert_before(group, old_head, current_it);
		else
			return current_it;
	}
	else
	{
		if (key < m_head)
		{
			auto current_it = insert_at_bottom(group, key, value, INVALID_PREDECESSOR(), m_head);
			return insert_before(group, m_head, current_it);
		}
		else if (key > m_tail)
		{
			auto current_it = insert_at_bottom(group, key, value, m_tail, INVALID_SUCCESSOR());
			return insert_after(group, m_tail, current_it);
		}
		else
		{
			if (key < old_minimal)
			{
				//Map_iterator successor_it = spinlock_for_value(group, old_minimal);
				auto current_it = insert_at_bottom(group, key, value, old_minimal, old_minimal);
				return insert_before(group, old_minimal, current_it);
			}
			else if (key > old_maximal)
			{
				//Map_iterator predecessor_it = spinlock_for_value(group, old_maximal);
				auto current_it = insert_at_bottom(group, key, value, old_maximal, old_maximal);
				return insert_after(group, old_maximal, current_it);
			}
			else
			{
				auto current_it = insert_at_bottom(group, key, value, old_minimal, old_maximal);
				return insert_between(group, old_minimal, old_maximal, current_it);
			}
		}
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::size_type ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::maximal_size() const
{
	return 1 << RANK;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::predecessor(threads group, key_type key)
{
	if (!has_data())
		return end();

	if (key < m_head)
		return end();
	if (key >= m_tail)
		return m_bottom.find(group, m_tail);

	auto u = binary_search(group, key);
	if (u.second == m_bottom.end())
	{
		if (key < u.first->second.minimal_left)
		{
			auto predecessor_it = m_bottom.find(group, u.first->second.minimal_left);
			return m_bottom.find(group, predecessor_it->second.predecessor);
		}
		else
			return m_bottom.find(group, u.first->second.maximal_right);
	}
	else
		return u.second;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::size_type ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::size() const
{
	return m_bottom.size();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::successor(threads group, key_type key)
{
	if (!has_data())
		return end();

	if (key > m_tail)
		return end();
	if (key <= m_head)
		return m_bottom.find(group, m_head);

	auto u = binary_search(group, key);
	if (u.second == m_bottom.end())
	{
		if (key < u.first->second.minimal_left)
			return m_bottom.find(group, u.first->second.minimal_left);
		else
		{
			auto predecessor_it = m_bottom.find(group, u.first->second.minimal_left);
			return m_bottom.find(group, predecessor_it->second.successor);
		}
	}
	else
		return u.second;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::debug() const
{
	using gpu::print;
	for (int i = 0; i != SUBRANK; ++i)
	{
		print("HashMap (", i, "): ");
		const auto& map = m_maps[i];
		map.for_each([](const auto& it) {
			print("{", key_type(it->first), "|", key_type(it->second.minimal_left), ", ", key_type(it->second.maximal_right), "}");
		});
		print("\n");
	}

	print("Bottom: ");
	m_bottom.for_each([this](const auto& it) {
		auto& value = it->second;
		if (value.predecessor != INVALID_PREDECESSOR() && value.successor != INVALID_SUCCESSOR())
			print("{", key_type(it->first), "|", value.value, "=>[", key_type(value.predecessor), "|", key_type(value.successor), "]}");
		else if (value.predecessor != INVALID_PREDECESSOR())
			print("{", key_type(it->first), "|", value.value, "=>[", key_type(value.predecessor), "|#]}");
		else if (value.successor != INVALID_SUCCESSOR())
			print("{", key_type(it->first), "|", value.value, "=>[#|", key_type(value.successor), "]}");
		else
			print("{", key_type(it->first), "|", value.value, "=>[#|#]}");
	});

	print("\nHead/Tail: ");
	if (m_head != INVALID_PREDECESSOR())
		print("Head: ", key_type(m_head), " ");
	if (m_tail != INVALID_SUCCESSOR())
		print("Tail: ", key_type(m_tail), " ");
	print("\n");
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ auto ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::binary_search(threads group, key_type key) -> gpu::pair<keyset_iterator, Map_iterator>
{
	auto thid = group.thread_rank();
	keyset_iterator found_it;
	Map_iterator bottom_it;

	unsigned int warp_offset = 0u;
	do
	{
		bool has_value = false;
		if (warp_offset + thid < SUBRANK)
		{
			key_type bits = extract_i_upper_bits(key, warp_offset + thid);
			found_it = m_maps[warp_offset + thid].find(bits);

			// We try to find out the last place such that T T T F <- We want the third T.
			has_value = found_it != m_maps[warp_offset + thid].end();
		}
		else if (warp_offset + thid == SUBRANK)
		{
			bottom_it = m_bottom.find(key);
		}

		unsigned int matching_bits = group.ballot(has_value);
		if (matching_bits)
		{
			// The idea is that we want the min, max of the lowest node in the tree where there is data
			unsigned int insert_update_separation = 31u - __clz(matching_bits);
			found_it.shfl(group, insert_update_separation);
			break;
		}
		warp_offset += group.size();
	} while (warp_offset < SUBRANK);

	bottom_it.shfl(group, SUBRANK % 32);
	return { found_it, bottom_it };
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::key_type ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::extract_i_upper_bits(key_type key, int number_of_bits) const
{
	key_type result = key >> (SUBRANK - number_of_bits);
	return result;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ auto ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::find_or_update(threads group, key_type key) -> gpu::pair<key_type, key_type>
{
	auto thid = group.thread_rank();

	key_type old_minimum = INVALID_PREDECESSOR();
	key_type old_maximum = INVALID_SUCCESSOR();

	{
		unsigned int warp_offset = 0u;
		do
		{
			bool has_value = false;
			if (warp_offset + thid < SUBRANK)
			{
				key_type bits = extract_i_upper_bits(key, warp_offset + thid);
				keyset_iterator found_it = m_maps[warp_offset + thid].find(bits);

				// We try to find out the last place such that T T T F <- We want the third T.
				has_value = found_it != m_maps[warp_offset + thid].end();
				unsigned int matching_bits = group.ballot(has_value);
				unsigned int insert_update_separation = 32u - __clz(matching_bits);

				if (has_value)
				{
					old_minimum = found_it->second.minimal_left;
					old_maximum = found_it->second.maximal_right;
				}

				if (thid >= insert_update_separation && !has_value) // Not just, should be related to group
				{
					m_maps[warp_offset + thid].insert_or_update(gpu::make_pair<key_type, Children>(bits, { key, key }),
						[](Children& lhs, Children&& rhs) {
						if (rhs.minimal_left < lhs.minimal_left)
							lhs.minimal_left.min(rhs.minimal_left);
						else if (rhs.maximal_right > lhs.maximal_right)
							lhs.maximal_right.max(rhs.maximal_right);
					});
				}
				else
				{
					while (found_it == m_maps[warp_offset + thid].end()) // Should be unlikely
						found_it = m_maps[warp_offset + thid].find(bits);

					if (key < found_it->second.minimal_left)
						found_it->second.minimal_left.min(key);
					if (key > found_it->second.maximal_right)
						found_it->second.maximal_right.max(key);
				}
			}

			group.sync();
			unsigned int matching_bits = group.ballot(has_value);
			if (matching_bits)
			{
				// The idea is that we want the min, max of the lowest node in the tree where there is data
				unsigned int insert_update_separation = 31u - __clz(matching_bits);
				old_minimum = group.shfl(old_minimum, insert_update_separation);
				old_maximum = group.shfl(old_maximum, insert_update_separation);
			}

			warp_offset += group.size();
		} while (warp_offset < SUBRANK);
	}

	return { old_minimum, old_maximum };
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::Map_iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::insert_at_bottom(threads group, key_type key, mapped_type value, key_type predecessor, key_type successor)
{
	return m_bottom.insert_or_update(group, gpu::make_pair<key_type, Node>(key, { value, predecessor, successor }), [](auto& lhs, auto&& rhs) {
		lhs.value = std::move(rhs.value);
		/*if (rhs.predecessor < lhs.predecessor)
			lhs.predecessor.min(rhs.predecessor);
		else if (rhs.successor > lhs.successor)
			lhs.successor.max(rhs.successor);*/
	});
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ bool ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::has_data() const
{
	return m_head != INVALID_PREDECESSOR() || m_tail != INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::key_type ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::INVALID_PREDECESSOR() const
{
	return gpu::numeric_limits<key_type>::max();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::key_type ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::INVALID_SUCCESSOR() const
{
	return gpu::numeric_limits<key_type>::min();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::insert_after(threads group, key_type key, Map_iterator to_insert)
{
	auto predecessor_it = spinlock_for_value(group, key);
	if (predecessor_it == m_bottom.end())
		return to_insert;

	int i = 0;
	//while (true)
	{
		auto last_predecessor_it = end();
		while (predecessor_it->second.successor != INVALID_SUCCESSOR() && predecessor_it->second.successor < to_insert->first && predecessor_it != last_predecessor_it)
		{
			last_predecessor_it = predecessor_it;
			predecessor_it = spinlock_for_value(group, predecessor_it->second.successor);
			++i;

			if (predecessor_it == m_bottom.end() || i > 20)
				return to_insert;
		}

		if (predecessor_it->second.successor == INVALID_SUCCESSOR())
		{
			to_insert->second.predecessor.store(group, predecessor_it->first);
			to_insert->second.successor.store(group, INVALID_SUCCESSOR());

			if (predecessor_it->second.successor.compare_and_swap(group, INVALID_SUCCESSOR(), to_insert->first) == INVALID_SUCCESSOR())
			{
				m_tail.max(group, to_insert->first);
				return to_insert;
			}

			//continue;
			return to_insert;
		}

		if (predecessor_it == m_bottom.end() || predecessor_it->second.successor == to_insert->first)
			return to_insert;

#ifdef GPU_XFASTTRIE_DEBUG
		ENSURE(predecessor_it->second.successor != INVALID_SUCCESSOR());
#endif // GPU_XFASTTRIE_DEBUG
		auto successor_it = spinlock_for_value(group, predecessor_it->second.successor);

		if (successor_it == m_bottom.end() || successor_it->second.predecessor == to_insert->first)
			return to_insert;

		key_type old_successor = predecessor_it->second.successor;
		key_type old_predecessor = successor_it->second.predecessor;

#ifdef GPU_XFASTTRIE_DEBUG
		ENSURE(old_successor >= to_insert->first && old_predecessor <= to_insert->first);
#endif // GPU_XFASTTRIE_DEBUG

		to_insert->second.predecessor.store(group, predecessor_it->first);
		to_insert->second.successor.store(group, successor_it->first);

		if (predecessor_it->second.successor.compare_and_swap(group, old_successor, to_insert->first) == old_successor)
		{
			if (successor_it->second.predecessor.compare_and_swap(group, old_predecessor, to_insert->first) == old_predecessor)
			{
				return to_insert;
			}
		}
		return to_insert;
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::insert_before(threads group, key_type key, Map_iterator to_insert)
{
	auto successor_it = spinlock_for_value(group, key);
	if (successor_it == m_bottom.end())
		return to_insert;

	int i = 0;
	//while (true)
	{
		auto last_successor_it = end();
		while (successor_it->second.predecessor != INVALID_PREDECESSOR() && successor_it->second.predecessor > to_insert->first && successor_it != last_successor_it)
		{
			last_successor_it = successor_it;
			successor_it = spinlock_for_value(group, successor_it->second.predecessor);
			++i;

			if (successor_it == m_bottom.end() || i > 20)
				return to_insert;
		}

		if (successor_it->second.predecessor == INVALID_PREDECESSOR())
		{
			to_insert->second.predecessor.store(group, INVALID_PREDECESSOR());
			to_insert->second.successor.store(group, successor_it->first);

			if (successor_it->second.predecessor.compare_and_swap(group, INVALID_PREDECESSOR(), to_insert->first) == INVALID_PREDECESSOR())
			{
				m_head.min(group, to_insert->first);
				return to_insert;
			}

			//continue;
			return to_insert;
		}

		if (successor_it == m_bottom.end() || successor_it->second.predecessor == to_insert->first)
			return to_insert;

#ifdef GPU_XFASTTRIE_DEBUG
		ENSURE(successor_it->second.predecessor != INVALID_PREDECESSOR());
#endif // GPU_XFASTTRIE_DEBUG
		auto predecessor_it = spinlock_for_value(group, successor_it->second.predecessor);
		if (predecessor_it == m_bottom.end())
			return to_insert;

		key_type old_successor = predecessor_it->second.successor;
		key_type old_predecessor = successor_it->second.predecessor;

#ifdef GPU_XFASTTRIE_DEBUG
		ENSURE(old_successor >= to_insert->first && old_predecessor <= to_insert->first);
#endif // GPU_XFASTTRIE_DEBUG

		to_insert->second.predecessor.store(group, predecessor_it->first);
		to_insert->second.successor.store(group, successor_it->first);

		if (predecessor_it->second.successor.compare_and_swap(group, old_successor, to_insert->first) == old_successor)
		{
			if (successor_it->second.predecessor.compare_and_swap(group, old_predecessor, to_insert->first) == old_predecessor)
			{
				return to_insert;
			}
		}
		return to_insert;
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::insert_between(threads group, key_type predecessor_it_key, key_type successor_it_key, Map_iterator to_insert)
{
	auto predecessor_it = spinlock_for_value(group, predecessor_it_key);
	auto successor_it = spinlock_for_value(group, successor_it_key);

	if (predecessor_it == m_bottom.end() || successor_it == m_bottom.end())
		return to_insert;

	int i = 0;
	//while (true)
	{
		while (predecessor_it->second.successor < to_insert->first)
		{
			predecessor_it = spinlock_for_value(group, predecessor_it->second.successor);
			++i;

			if (predecessor_it == m_bottom.end() || i > 20)
				return to_insert;
		}

		while (successor_it->second.predecessor > to_insert->first)
		{
			successor_it = spinlock_for_value(group, successor_it->second.predecessor);
			++i;

			if (successor_it == m_bottom.end() || i > 20)
				return to_insert;
		}

		key_type old_successor = predecessor_it->second.successor;
		key_type old_predecessor = successor_it->second.predecessor;

#ifdef GPU_XFASTTRIE_DEBUG
		ENSURE(old_successor >= to_insert->first && old_predecessor <= to_insert->first);
#endif // GPU_XFASTTRIE_DEBUG

		to_insert->second.predecessor.store(group, predecessor_it->first);
		to_insert->second.successor.store(group, successor_it->first);

		if (predecessor_it->second.successor.compare_and_swap(group, old_successor, to_insert->first) == old_successor)
		{
			if (successor_it->second.predecessor.compare_and_swap(group, old_predecessor, to_insert->first) == old_predecessor)
			{
				return to_insert;
			}
		}
		return to_insert;
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::Map_iterator ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::spinlock_for_value(threads group, key_type key)
{
	int i = 0;
	Map_iterator result_it;
	do
	{
		result_it = m_bottom.find(group, key);
		++i;
		if (i > 32)
			return m_bottom.end();
	} while (result_it == m_bottom.end());
	return result_it;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void ConcurrentXFastTrieWarpParallel<Key, Value, UNIVERSE>::post_condition(threads group)
{
#ifdef GPU_XFASTTRIE_DEBUG
	for (int i = 0; i != SUBRANK; ++i)
	{
		auto& map = m_maps[i];
		map.for_each([this, group, i](const auto& it) {
			if (i + 1 < SUBRANK)
			{
				auto left_child = m_maps[i + 1].find(group, it->first << 1);
				auto right_child = m_maps[i + 1].find(group, (it->first << 1) | 1);
				ENSURE(left_child != m_maps[i + 1].end() || right_child != m_maps[i + 1].end());
			}
		});
	}

	if (m_head != INVALID_PREDECESSOR())
	{
		ENSURE(m_head == m_tail || m_tail != INVALID_SUCCESSOR());
	}
	else
		ENSURE(m_tail == INVALID_SUCCESSOR());

	if (has_data())
	{
		m_bottom.for_each([this, group](const auto& it) {
			if (it->first != m_head)
			{
				auto predecessor_it = m_bottom.find(group, it->second.predecessor);
				ENSURE(it->first > predecessor_it->first);
				ENSURE(it->first == predecessor_it->second.successor);
			}
			if (it->first != m_tail)
			{
				auto successor_it = m_bottom.find(group, it->second.successor);
				ENSURE(it->first < successor_it->first);
				ENSURE(it->first == successor_it->second.predecessor);
			}
		});
	}
#endif // GPU_XFASTTRIE_DEBUG
}
