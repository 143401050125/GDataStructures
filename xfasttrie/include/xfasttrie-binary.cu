#include "hip/hip_runtime.h"
#include "xfasttrie-binary.cuh"

#include "utility/limits.cuh"
#include "utility/print.cuh"

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::iterator XFastTrieBinary<Key, Value, UNIVERSE>::begin()
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::const_iterator XFastTrieBinary<Key, Value, UNIVERSE>::begin() const
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::iterator XFastTrieBinary<Key, Value, UNIVERSE>::end()
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::const_iterator XFastTrieBinary<Key, Value, UNIVERSE>::end() const
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ XFastTrieBinary<Key, Value, UNIVERSE>::XFastTrieBinary(block_threads block, gpu::default_allocator& allocator)
{
	threads tile32 = cooperative_groups::tiled_partition<32>(block);
	auto thid = block.thread_rank();

	if (thid < tile32.size())
	{
		unsigned int max_allocate = 1 << 18u;
		for (int rank = 0; rank != SUBRANK; ++rank)
		{
			unsigned int preallocate = rank < 17u ? 1u << (rank + 2u) : max_allocate;
			m_maps[rank] = Keyset{ tile32, allocator, preallocate };
		}
		m_bottom = Map{ tile32, allocator, max_allocate };

		m_head = INVALID_PREDECESSOR();
		m_tail = INVALID_SUCCESSOR();
	}
	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ XFastTrieBinary<Key, Value, UNIVERSE>::XFastTrieBinary(threads group, gpu::default_allocator& allocator)
{
	unsigned int max_allocate = 1 << 18u;
	for (int rank = 0; rank != SUBRANK; ++rank)
	{
		unsigned int preallocate = rank < 17u ? 1u << (rank + 2u) : max_allocate;
		m_maps[rank] = Keyset{ group, allocator, preallocate };
	}
	m_bottom = Map{ group, allocator, max_allocate };

	m_head = INVALID_PREDECESSOR();
	m_tail = INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieBinary<Key, Value, UNIVERSE>::clear(block_threads block)
{
	threads tile32 = cooperative_groups::tiled_partition<32>(block);

	if (block.thread_rank() < 32)
		clear(tile32);

	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieBinary<Key, Value, UNIVERSE>::clear(threads group)
{
	for (int rank = 0; rank != SUBRANK; ++rank)
		m_maps[rank].clear(group);

	m_bottom.clear(group);

	m_head = INVALID_PREDECESSOR();
	m_tail = INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::iterator XFastTrieBinary<Key, Value, UNIVERSE>::find(key_type key)
{
	return m_bottom.find(key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::iterator XFastTrieBinary<Key, Value, UNIVERSE>::find(threads group, key_type key)
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::const_iterator XFastTrieBinary<Key, Value, UNIVERSE>::find(threads group, key_type key) const
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::iterator XFastTrieBinary<Key, Value, UNIVERSE>::insert(threads group, key_type key, value_type value)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	auto u = binary_search(group, key);
	auto old_it = m_maps[u.second].end(); // since when we reallocate data, pointer will change
	auto old_minimal = u.first->second.minimal_left;
	auto old_maximal = u.first->second.maximal_right;

	size_type loop = u.second;
	while (loop != SUBRANK)
	{
		key_type bits = extract_i_upper_bits(key, loop);
		auto found_it = m_maps[loop].find(group, bits);
		auto end_it = m_maps[loop].end();
		if (found_it == end_it)
		{
			m_maps[loop].insert(group, gpu::make_pair<key_type, Children>(bits, { key, key }));
		}
		++loop;
	}

	walk_up(group, key, u.second);

	if (!has_data())
	{
		auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, INVALID_PREDECESSOR(), INVALID_SUCCESSOR() }));
		m_head = key;
		m_tail = key;
		post_condition(group);
		return current_it;
	}
	else
	{
		if (key < m_head)
		{
			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, INVALID_PREDECESSOR(), m_head }));
			auto it = m_bottom.find(group, m_head);
			it->second.predecessor = key;
			m_head = key;
			post_condition(group);
			return current_it;
		}
		else if (key > m_tail)
		{
			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, m_tail, INVALID_SUCCESSOR() }));
			auto it = m_bottom.find(group, m_tail);
			it->second.successor = key;
			m_tail = key;
			post_condition(group);
			return current_it;
		}
		else
		{
			if (key < old_minimal)
			{
				auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value,{}, old_minimal }));
				auto successor_it = m_bottom.find(group, old_minimal);
				current_it->second.predecessor = successor_it->second.predecessor;
				auto predecessor_it = m_bottom.find(group, successor_it->second.predecessor);

				if (group.thread_rank() == 0)
				{
					successor_it->second.predecessor = key;
					predecessor_it->second.successor = key;
				}
				group.sync();
				post_condition(group);
				return current_it;
			}
			else if (key > old_maximal)
			{
				auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, old_maximal,{} }));
				auto predecessor_it = m_bottom.find(group, old_maximal);
				current_it->second.successor = predecessor_it->second.successor;
				auto successor_it = m_bottom.find(group, predecessor_it->second.successor);

				if (group.thread_rank() == 0)
				{
					successor_it->second.predecessor = key;
					predecessor_it->second.successor = key;
				}
				group.sync();
				post_condition(group);
				return current_it;
			}
			else
			{
				/*if (group.thread_rank() == 3)
				{
					gpu::print(" > ", key, " o ", old_minimal, " p ", old_maximal, "\n");
				}
				group.sync();
				if (group.thread_rank() == 0)
				{
					gpu::print(" > ", key, " o ", old_minimal, " p ", old_maximal, "\n");
				}
				group.sync();

				ENSURE(false);*/
				return end();
			}
		}
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::size_type XFastTrieBinary<Key, Value, UNIVERSE>::maximal_size() const
{
	return 1 << RANK;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::iterator XFastTrieBinary<Key, Value, UNIVERSE>::predecessor(threads group, key_type key)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	if (!has_data())
		return end();

	if (key < m_head)
		return end();
	if (key >= m_tail)
		return m_bottom.find(group, m_tail);

	auto u = binary_search(group, key);
	if (key < u.first->second.minimal_left)
	{
		auto predecessor_it = m_bottom.find(group, u.first->second.minimal_left);
		return m_bottom.find(group, predecessor_it->second.predecessor);
	}
	else
		return m_bottom.find(group, u.first->second.maximal_right);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::size_type XFastTrieBinary<Key, Value, UNIVERSE>::size() const
{
	return m_bottom.size();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::iterator XFastTrieBinary<Key, Value, UNIVERSE>::successor(threads group, key_type key)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	if (!has_data())
		return end();

	if (key > m_tail)
		return end();
	if (key <= m_head)
		return m_bottom.find(group, m_head);

	auto u = binary_search(group, key);
	if (key < u.first->second.minimal_left)
		return m_bottom.find(group, u.first->second.minimal_left);
	else
	{
		auto predecessor_it = m_bottom.find(group, u.first->second.minimal_left);
		return m_bottom.find(group, predecessor_it->second.successor);
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieBinary<Key, Value, UNIVERSE>::debug() const
{
	using gpu::print;
	for (int i = 0; i != SUBRANK; ++i)
	{
		print("HashMap (", i, "): ");
		const auto& map = m_maps[i];
		for (auto it = map.begin(); it != map.end(); ++it)
		{
			print("{", it->first, "|", it->second.minimal_left, ", ", it->second.maximal_right, "}");
		}
		print("\n");
	}

	print("Bottom: ");
	for (auto it = m_bottom.begin(); it != m_bottom.end(); ++it)
	{
		auto& value = it->second;
		if (value.predecessor != INVALID_PREDECESSOR() && value.successor != INVALID_SUCCESSOR())
			print("{", it->first, "|", value.value, "=>[", value.predecessor, "|", value.successor, "]}");
		else if (value.predecessor != INVALID_PREDECESSOR())
			print("{", it->first, "|", value.value, "=>[", value.predecessor, "|#]}");
		else if (value.successor != INVALID_SUCCESSOR())
			print("{", it->first, "|", value.value, "=>[#|", value.successor, "]}");
		else
			print("{", it->first, "|", value.value, "=>[#|#]}");
	}

	print("\nHead/Tail: ");
	if (m_head != INVALID_PREDECESSOR())
		print("Head: ", m_head, " ");
	if (m_tail != INVALID_SUCCESSOR())
		print("Tail: ", m_tail, " ");
	print("\n");
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ auto XFastTrieBinary<Key, Value, UNIVERSE>::binary_search(threads group, key_type key) -> gpu::pair<keyset_iterator, size_type>
{
	int low = 0;
	int high = SUBRANK - 1;
	keyset_iterator u = m_maps[low].end();
	while (low <= high)
	{
		int mid = (low + high) / 2;
		key_type bits = extract_i_upper_bits(key, mid);
		auto v = m_maps[mid].find(group, bits);
		if (v == m_maps[mid].end())
		{
			high = mid - 1;
		}
		else
		{
			u = v;
			low = mid + 1;
		}
	}

	return { u, low };
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::key_type XFastTrieBinary<Key, Value, UNIVERSE>::extract_i_upper_bits(key_type key, int number_of_bits) const
{
	key_type result = key >> (SUBRANK - number_of_bits);
	return result;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::iterator XFastTrieBinary<Key, Value, UNIVERSE>::get_predecessor(threads group, key_type key, keyset_iterator u)
{
	auto& value = u->second;
	return m_bottom.find(group, value.minimal_left);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::iterator XFastTrieBinary<Key, Value, UNIVERSE>::get_successor(threads group, key_type key, keyset_iterator u)
{
	auto& value = u->second;
	return m_bottom.find(group, value.maximal_right);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ bool XFastTrieBinary<Key, Value, UNIVERSE>::has_data() const
{
	return m_head != INVALID_PREDECESSOR() && m_tail != INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::key_type XFastTrieBinary<Key, Value, UNIVERSE>::INVALID_PREDECESSOR() const
{
	return gpu::numeric_limits<key_type>::max();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieBinary<Key, Value, UNIVERSE>::key_type XFastTrieBinary<Key, Value, UNIVERSE>::INVALID_SUCCESSOR() const
{
	return gpu::numeric_limits<key_type>::min();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieBinary<Key, Value, UNIVERSE>::walk_up(threads group, key_type key, size_type from)
{
	while (from != 0)
	{
		--from;
		key_type bits = extract_i_upper_bits(key, from);
		auto it = m_maps[from].find(group, bits);
		it->second.minimal_left = min(it->second.minimal_left, key);
		it->second.maximal_right = max(it->second.maximal_right, key);
		if (it->second.minimal_left != key && it->second.maximal_right != key)
			return;
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieBinary<Key, Value, UNIVERSE>::post_condition(threads group)
{
#ifdef GPU_XFASTTRIE_DEBUG
	for (int i = 0; i != SUBRANK; ++i)
	{
		auto& map = m_maps[i];
		for (auto it = map.begin(); it != map.end(); ++it)
		{
			if (i + 1 < SUBRANK)
			{
				auto left_child = m_maps[i + 1].find(group, it->first << 1);
				auto right_child = m_maps[i + 1].find(group, (it->first << 1) | 1);
				ENSURE(left_child != m_maps[i + 1].end() || right_child != m_maps[i + 1].end());
			}
		}
	}

	if (m_head != INVALID_PREDECESSOR())
	{
		ENSURE(m_head == m_tail || m_tail != INVALID_SUCCESSOR());
		auto buffer_start = &(*m_bottom.end()) - m_bottom.capacity();
		auto head = m_bottom.find(group, m_head);
		auto tail = m_bottom.find(group, m_tail);
		ENSURE(&(*head) >= buffer_start && &(*head) < &(*m_bottom.end()));
		ENSURE(&(*tail) >= buffer_start && &(*tail) < &(*m_bottom.end()));
	}
	else
		ENSURE(m_tail == INVALID_SUCCESSOR());

	if (has_data())
	{
		for (auto it = m_bottom.begin(); it != m_bottom.end(); ++it)
		{
			if (it->first != m_head)
			{
				auto predecessor_it = m_bottom.find(group, it->second.predecessor);
				ENSURE(it->first > predecessor_it->first);
				ENSURE(it->first == predecessor_it->second.successor);
			}
			if (it->first != m_tail)
			{
				auto successor_it = m_bottom.find(group, it->second.successor);
				ENSURE(it->first < successor_it->first);
				ENSURE(it->first == successor_it->second.predecessor);
			}
		}
	}
#endif // GPU_XFASTTRIE_DEBUG
}
