#include "hip/hip_runtime.h"
#include "xfasttrie-group-parallel.cuh"

#include "algorithms/find.cuh"
#include "utility/limits.cuh"
#include "utility/print.cuh"

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::begin()
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::const_iterator XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::begin() const
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::end()
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::const_iterator XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::end() const
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::XFastTrieGroupParallel(block_threads block, gpu::default_allocator& allocator)
{
	threads tile32 = cooperative_groups::tiled_partition<32>(block);
	auto thid = block.thread_rank();

	if (thid < tile32.size())
	{
		unsigned int max_allocate = 1 << 18u;
		for (int rank = 0; rank != NUMBER_OF_KEYSETS; ++rank)
		{
			unsigned int preallocate = rank * GROUP_SIZE < 17u ? 1u << (rank * GROUP_SIZE + 2u) : max_allocate;
			m_maps[rank] = Keyset{ tile32, allocator, preallocate };
		}
		m_bottom = Map{ tile32, allocator, max_allocate };

		m_head = INVALID_PREDECESSOR();
		m_tail = INVALID_SUCCESSOR();
	}
	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::XFastTrieGroupParallel(threads group, gpu::default_allocator& allocator)
{
	unsigned int max_allocate = 1 << 18u;
	for (int rank = 0; rank != NUMBER_OF_KEYSETS; ++rank)
	{
		unsigned int preallocate = rank * GROUP_SIZE < 17u ? 1u << (rank * GROUP_SIZE + 2u) : max_allocate;
		m_maps[rank] = Keyset{ group, allocator, preallocate };
	}
	m_bottom = Map{ group, allocator, max_allocate };

	m_head = INVALID_PREDECESSOR();
	m_tail = INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::clear(block_threads block)
{
	threads tile32 = cooperative_groups::tiled_partition<32>(block);

	if (block.thread_rank() < 32)
		clear(tile32);

	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::clear(threads group)
{
	for (int rank = 0; rank != NUMBER_OF_KEYSETS; ++rank)
		m_maps[rank].clear(group);

	m_bottom.clear(group);

	m_head = INVALID_PREDECESSOR();
	m_tail = INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::find(key_type key)
{
	return m_bottom.find(key);
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::find(threads group, key_type key)
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::const_iterator XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::find(threads group, key_type key) const
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::insert(threads group, key_type key, mapped_type value)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	auto old_values = find_or_update(group, key);
	key_type old_minimal = old_values.first;
	key_type old_maximal = old_values.second;

	if (!has_data())
	{
		auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, INVALID_PREDECESSOR(), INVALID_SUCCESSOR() }));
		m_head = key;
		m_tail = key;
		post_condition(group);
		return current_it;
	}
	else
	{
		if (key < m_head)
		{
			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, INVALID_PREDECESSOR(), m_head }));
			auto it = m_bottom.find(group, m_head);
			it->second.predecessor = key;
			m_head = key;
			post_condition(group);
			return current_it;
		}
		else if (key > m_tail)
		{
			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, m_tail, INVALID_SUCCESSOR() }));
			auto it = m_bottom.find(group, m_tail);
			it->second.successor = key;
			m_tail = key;
			post_condition(group);
			return current_it;
		}
		else
		{
			if (key < old_minimal)
			{
				auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value,{}, old_minimal }));
				auto successor_it = m_bottom.find(group, old_minimal);
				current_it->second.predecessor = successor_it->second.predecessor;
				auto predecessor_it = m_bottom.find(group, successor_it->second.predecessor);

				if (group.thread_rank() == 0)
				{
					successor_it->second.predecessor = key;
					predecessor_it->second.successor = key;
				}
				group.sync();
				post_condition(group);
				return current_it;
			}
			else if (key > old_maximal)
			{
				auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, old_maximal,{} }));
				auto predecessor_it = m_bottom.find(group, old_maximal);
				current_it->second.successor = predecessor_it->second.successor;
				auto successor_it = m_bottom.find(group, predecessor_it->second.successor);

				if (group.thread_rank() == 0)
				{
					successor_it->second.predecessor = key;
					predecessor_it->second.successor = key;
				}
				group.sync();
				post_condition(group);
				return current_it;
			}
			else
			{
				/*if (group.thread_rank() == 3)
				{
				gpu::print(" > ", key, " o ", old_minimal, " p ", old_maximal, "\n");
				}
				group.sync();
				if (group.thread_rank() == 0)
				{
				gpu::print(" > ", key, " o ", old_minimal, " p ", old_maximal, "\n");
				}
				group.sync();

				ENSURE(false);*/
				return end();
			}
		}
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::size_type XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::maximal_size() const
{
	return 1 << RANK;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::predecessor(threads group, key_type key)
{
	if (!has_data())
		return end();

	if (key < m_head)
		return end();
	if (key >= m_tail)
		return m_bottom.find(group, m_tail);

	BinarySearchResult u = binary_search(group, key);
	if (u.bottom_it == m_bottom.end())
	{
		const keyset_iterator& it = u.it;
		unsigned int index = u.index;
		if (key < it->second.minimal_left[index])
		{
			auto predecessor_it = m_bottom.find(group, it->second.minimal_left[index]);
			return m_bottom.find(group, predecessor_it->second.predecessor);
		}
		else
			return m_bottom.find(group, it->second.maximal_right[index]);
	}
	else
		return u.bottom_it;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::size_type XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::size() const
{
	return m_bottom.size();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::iterator XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::successor(threads group, key_type key)
{
	if (!has_data())
		return end();

	if (key > m_tail)
		return end();
	if (key <= m_head)
		return m_bottom.find(group, m_head);

	BinarySearchResult u = binary_search(group, key);
	if (u.bottom_it == m_bottom.end())
	{
		const keyset_iterator& it = u.it;
		unsigned int index = u.index;

		if (key < it->second.minimal_left[index])
			return m_bottom.find(group, it->second.minimal_left[index]);
		else
		{
			auto predecessor_it = m_bottom.find(group, it->second.minimal_left[index]);
			return m_bottom.find(group, predecessor_it->second.successor);
		}
	}
	else
		return u.bottom_it;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::debug() const
{
	using gpu::print;
	for (int i = 0; i != NUMBER_OF_KEYSETS; ++i)
	{
		print("HashMap (", i, "): ");
		const auto& map = m_maps[i];
		for (auto it = map.begin(); it != map.end(); ++it)
		{
			print("{", it->first, "|");
			print("0: (", it->second.minimal_left[0], ", ", it->second.maximal_right[0], ")");
			unsigned int group_size = (i == NUMBER_OF_KEYSETS - 1u) ? TAIL_GROUP_SIZE : GROUP_SIZE;
			if (group_size)
			{
				for (unsigned int g = 0u; g != group_size; ++g)
				{
					print(" ", g + 1, ": ");
					unsigned int offset = (1u << (g + 1u)) - 1u;
					for (unsigned int i = 0u; i != 1u << (g + 1u); ++i)
					{
						print("(", it->second.minimal_left[offset + i], ", ", it->second.maximal_right[offset + i], ")");
					}
				}
			}
		}
		print("\n");
	}

	print("Bottom: ");
	for (auto it = m_bottom.begin(); it != m_bottom.end(); ++it)
	{
		auto& value = it->second;
		if (value.predecessor != INVALID_PREDECESSOR() && value.successor != INVALID_SUCCESSOR())
			print("{", it->first, "|", value.value, "=>[", value.predecessor, "|", value.successor, "]}");
		else if (value.predecessor != INVALID_PREDECESSOR())
			print("{", it->first, "|", value.value, "=>[", value.predecessor, "|#]}");
		else if (value.successor != INVALID_SUCCESSOR())
			print("{", it->first, "|", value.value, "=>[#|", value.successor, "]}");
		else
			print("{", it->first, "|", value.value, "=>[#|#]}");
	}

	print("\nHead/Tail: ");
	if (m_head != INVALID_PREDECESSOR())
		print("Head: ", m_head, " ");
	if (m_tail != INVALID_SUCCESSOR())
		print("Tail: ", m_tail, " ");
	print("\n");
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ auto XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::binary_search(threads group, key_type key) -> BinarySearchResult
{
	auto thid = group.thread_rank();
	bool has_no_value = true;
	unsigned int index;
	keyset_iterator found_it;
	Map_iterator bottom_it;
	if (thid < NUMBER_OF_KEYSETS)
	{
		key_type group_bits;
		key_type tail_bits;
		if (thid == NUMBER_OF_KEYSETS - 1u)
		{
			group_bits = key >> (TAIL_GROUP_SIZE + 1u);
			tail_bits = key & ((1u << (TAIL_GROUP_SIZE + 1u)) - 1u);
			tail_bits >>= 1u; // Avoid last bit
			found_it = m_maps[thid].find(group_bits);
		}
		else
		{
			unsigned int shift_value = (TAIL_GROUP_SIZE + 1u) + (NUMBER_OF_KEYSETS - thid - 2u) * (GROUP_SIZE + 1) + 1u; // Don't forget last bit
			group_bits = key >> shift_value;
			tail_bits = group_bits & ((1u << GROUP_SIZE) - 1u);
			group_bits = group_bits >> GROUP_SIZE; // We get the upper bits
			found_it = m_maps[thid].find(group_bits);
		}

		// We try to find out the last place such that T T T F <- We want the third T.
		has_no_value = found_it == m_maps[thid].end();

		if (!has_no_value)
			index = get_highest_index(tail_bits, found_it->second, thid);
	}
	else if (thid == NUMBER_OF_KEYSETS)
	{
		bottom_it = m_bottom.find(key);
	}

	unsigned int matching_bits = group.ballot(has_no_value);
	if (matching_bits)
	{
		// The idea is that we want the min, max of the lowest node in the tree where there is data
		unsigned int insert_update_separation = __ffs(matching_bits) - 2u;
		index = group.shfl(index, insert_update_separation);
		found_it.shfl(group, insert_update_separation);
	}
	bottom_it.shfl(group, NUMBER_OF_KEYSETS);
	return { found_it, bottom_it, index };
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::key_type XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::extract_i_upper_bits(key_type key, int number_of_bits) const
{
	key_type result = key >> (SUBRANK - number_of_bits);
	return result;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ auto XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::find_or_update(threads group, key_type key) -> gpu::pair<key_type, key_type>
{
	auto thid = group.thread_rank();

	{
		bool resize = false;
		if (thid < NUMBER_OF_KEYSETS)
		{
			resize = m_maps[thid].pending_resize();
		}
		group.sync();
		unsigned int matching_bits = group.ballot(resize);
		while (matching_bits)
		{
			unsigned int rank = __ffs(matching_bits) - 1u;
			m_maps[rank].resize(group);
			matching_bits ^= (1 << rank); // Bits are put in little endian
		}
	}

	key_type old_minimum = INVALID_PREDECESSOR();
	key_type old_maximum = INVALID_SUCCESSOR();
	bool has_no_value = true;
	if (thid < NUMBER_OF_KEYSETS)
	{
		key_type group_bits;
		key_type tail_bits;
		keyset_iterator found_it;
		if (thid == NUMBER_OF_KEYSETS - 1u)
		{
			group_bits = key >> (TAIL_GROUP_SIZE + 1u);
			tail_bits = key & ((1u << (TAIL_GROUP_SIZE + 1u)) - 1u);
			tail_bits >>= 1u; // Avoid last bit
			found_it = m_maps[thid].find(group_bits);
		}
		else
		{
			unsigned int shift_value = (TAIL_GROUP_SIZE + 1u) + (NUMBER_OF_KEYSETS - thid - 2u) * (GROUP_SIZE + 1) + 1u; // Don't forget last bit
			group_bits = key >> shift_value;
			tail_bits = group_bits & ((1u << GROUP_SIZE) - 1u);
			group_bits = group_bits >> GROUP_SIZE; // We get the upper bits
			found_it = m_maps[thid].find(group_bits);
		}

		// We try to find out the last place such that T T T F <- We want the third T.
		has_no_value = found_it == m_maps[thid].end();
		unsigned int matching_bits = group.ballot(has_no_value);
		unsigned int insert_update_separation = __ffs(matching_bits);

		if (!has_no_value)
		{
			unsigned int i = get_highest_index(tail_bits, found_it->second, thid);
			old_minimum = found_it->second.minimal_left[i];
			old_maximum = found_it->second.maximal_right[i];
		}

		if (thid >= insert_update_separation - 1 && has_no_value) // Not just, should be related to group
		{
			m_maps[thid].insert(gpu::make_pair<key_type, Child>(group_bits, make_children(tail_bits, key, thid)));
		}
		else
		{
			update_key(found_it->second, tail_bits, key, thid);
		}
	}

	group.sync();

	unsigned int matching_bits = group.ballot(has_no_value);
	if (matching_bits)
	{
		// The idea is that we want the min, max of the lowest node in the tree where there is data
		unsigned int insert_update_separation = __ffs(matching_bits) - 2u;
		old_minimum = group.shfl(old_minimum, insert_update_separation);
		old_maximum = group.shfl(old_maximum, insert_update_separation);
	}

	return { old_minimum, old_maximum };
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ unsigned int XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::get_highest_index(key_type tail_bits, const Child& value, unsigned int thid)
{
	unsigned int upper_bound = thid == NUMBER_OF_KEYSETS - 1u ? TAIL_GROUP_SIZE : GROUP_SIZE;
	unsigned int offset = (1u << upper_bound) - 1u;
	while (offset)
	{
		unsigned int position = offset + tail_bits;
		if (value.minimal_left[position] != INVALID_PREDECESSOR())
			return position;
		offset = offset >> 1u;
		tail_bits = tail_bits >> 1u;
	}
	return 0u;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ auto XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::make_children(key_type tail_bits, const key_type& key, unsigned int thid) -> Child
{
	Child child;
	for (unsigned int i = 0u; i != (1u << (GROUP + 1u)) - 1u; ++i)
	{
		child.minimal_left[i] = INVALID_PREDECESSOR();
		child.maximal_right[i] = INVALID_SUCCESSOR();
	}

	unsigned int upper_bound = thid == NUMBER_OF_KEYSETS - 1u ? TAIL_GROUP_SIZE : GROUP_SIZE;
	unsigned int offset = (1u << upper_bound) - 1u;
	while (offset)
	{
		unsigned int position = offset + tail_bits;
		child.minimal_left[position] = key;
		child.maximal_right[position] = key;
		offset = offset >> 1u;
		tail_bits = tail_bits >> 1u;
	}
	child.minimal_left[0u] = key;
	child.maximal_right[0u] = key;
	return child;
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::update_key(Child& child, key_type tail_bits, const key_type& key, unsigned int thid)
{
	unsigned int upper_bound = thid == NUMBER_OF_KEYSETS - 1u ? TAIL_GROUP_SIZE : GROUP_SIZE;
	unsigned int offset = (1u << upper_bound) - 1u;
	while (offset)
	{
		unsigned int position = offset + tail_bits;
		child.minimal_left[position] = min(child.minimal_left[position], key);
		child.maximal_right[position] = max(child.maximal_right[position], key);
		offset = offset >> 1u;
		tail_bits = tail_bits >> 1u;
	}
	child.minimal_left[0u] = min(child.minimal_left[0u], key);
	child.maximal_right[0u] = max(child.maximal_right[0u], key);
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ bool XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::has_data() const
{
	return m_head != INVALID_PREDECESSOR() && m_tail != INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::key_type XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::INVALID_PREDECESSOR() const
{
	return gpu::numeric_limits<key_type>::max();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ typename XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::key_type XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::INVALID_SUCCESSOR() const
{
	return gpu::numeric_limits<key_type>::min();
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::walk_up(threads group, key_type key, size_type from)
{
	while (from != 0)
	{
		--from;
		key_type bits = extract_i_upper_bits(key, from);
		auto it = m_maps[from].find(group, bits);
		it->second.minimal_left = min(it->second.minimal_left, key);
		it->second.maximal_right = max(it->second.maximal_right, key);
		if (it->second.minimal_left != key && it->second.maximal_right != key)
			return;
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE, std::size_t GROUP>
__device__ void XFastTrieGroupParallel<Key, Value, UNIVERSE, GROUP>::post_condition(threads group)
{
#ifdef GPU_XFASTTRIE_DEBUG
	if (m_head != INVALID_PREDECESSOR())
	{
		ENSURE(m_head == m_tail || m_tail != INVALID_SUCCESSOR());
		auto buffer_start = &(*m_bottom.end()) - m_bottom.capacity();
		auto head = m_bottom.find(group, m_head);
		auto tail = m_bottom.find(group, m_tail);
		ENSURE(&(*head) >= buffer_start && &(*head) < &(*m_bottom.end()));
		ENSURE(&(*tail) >= buffer_start && &(*tail) < &(*m_bottom.end()));
	}
	else
		ENSURE(m_tail == INVALID_SUCCESSOR());

	if (has_data())
	{
		for (auto it = m_bottom.begin(); it != m_bottom.end(); ++it)
		{
			if (it->first != m_head)
			{
				auto predecessor_it = m_bottom.find(group, it->second.predecessor);
				ENSURE(it->first > predecessor_it->first);
				ENSURE(it->first == predecessor_it->second.successor);
			}
			if (it->first != m_tail)
			{
				auto successor_it = m_bottom.find(group, it->second.successor);
				ENSURE(it->first < successor_it->first);
				ENSURE(it->first == successor_it->second.predecessor);
			}
		}
	}
#endif // GPU_XFASTTRIE_DEBUG
}
