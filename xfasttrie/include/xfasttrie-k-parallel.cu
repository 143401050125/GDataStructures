#include "hip/hip_runtime.h"
#include "xfasttrie-k-parallel.cuh"

#include "algorithms/find.cuh"
#include "containers/array.cuh"
#include "utility/print.cuh"

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::iterator XFastTrieKParallel<Key, Value, UNIVERSE>::begin()
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::const_iterator XFastTrieKParallel<Key, Value, UNIVERSE>::begin() const
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::iterator XFastTrieKParallel<Key, Value, UNIVERSE>::end()
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::const_iterator XFastTrieKParallel<Key, Value, UNIVERSE>::end() const
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ XFastTrieKParallel<Key, Value, UNIVERSE>::XFastTrieKParallel(threads group, gpu::default_allocator& allocator)
{
	unsigned int max_allocate = 1024;
	for (int rank = 0; rank != SUBRANK; ++rank)
	{
		unsigned int preallocate = rank < 10 ? 1 << (rank + 2) : max_allocate;
		m_maps[rank] = Keyset{ group, allocator, preallocate };
	}
	m_bottom = Map{ group, allocator, max_allocate };

	m_head = m_bottom.end();
	m_tail = m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieKParallel<Key, Value, UNIVERSE>::clear(threads group)
{
	for (int rank = 0; rank != SUBRANK; ++rank)
		m_maps[rank].clear(group);

	m_bottom.clear(group);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::iterator XFastTrieKParallel<Key, Value, UNIVERSE>::find(threads group, key_type key)
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::const_iterator XFastTrieKParallel<Key, Value, UNIVERSE>::find(threads group, key_type key) const
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::iterator XFastTrieKParallel<Key, Value, UNIVERSE>::insert(threads group, key_type key, value_type value)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	auto u = binary_search(group, key);

	size_type loop = u.second;
	while (loop != SUBRANK)
	{
		key_type bits = extract_i_upper_bits(key, loop);
		auto found_it = m_maps[loop].find(group, bits);
		auto end_it = m_maps[loop].end();
		if (found_it == end_it)
		{
			m_maps[loop].insert(group, gpu::make_pair<key_type, key_type>(bits, key));
		}
		++loop;
	}

	if (!m_head && !m_tail)
	{
		auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, m_bottom.end(), m_bottom.end() }));
		m_head = current_it;
		m_tail = current_it;
		post_condition(group);
		return current_it;
	}
	else if (u.first == m_maps[u.second].end())
	{
		if (key < m_head->first)
		{
			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, m_bottom.end(), m_head }));
			m_head->second.predecessor = current_it;
			m_head = current_it;
			post_condition(group);
			return current_it;
		}
		else
		{
			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, m_tail, m_bottom.end() }));
			m_tail->second.successor = current_it;
			m_tail = current_it;
			post_condition(group);
			return current_it;
		}
	}
	else
	{
		const auto& predecessor_key = u.first->second;
		auto predecessor_it = m_bottom.find(group, predecessor_key);
		if (key > predecessor_key)
		{
			if (key > m_tail->first)
			{
				auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, m_tail, m_bottom.end() }));
				m_tail->second.successor = current_it;
				m_tail = current_it;
				post_condition(group);
				return current_it;
			}
			else
			{
				auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, predecessor_it, predecessor_it->second.successor }));
				current_it->second.successor->second.predecessor = current_it;
				predecessor_it->second.successor = current_it;
				post_condition(group);
				return current_it;
			}
		}
		else
		{
			while (predecessor_it->second.predecessor && predecessor_it->second.predecessor->first > key)
				predecessor_it = predecessor_it->second.predecessor; // In case of 6 7 and we add 5

			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, predecessor_it->second.predecessor, predecessor_it }));
			if (predecessor_it->second.predecessor)
				current_it->second.predecessor->second.successor = current_it;
			predecessor_it->second.predecessor = current_it;

			if (m_head == predecessor_it)
				m_head = current_it;

			post_condition(group);
			return current_it;
		}
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::iterator XFastTrieKParallel<Key, Value, UNIVERSE>::predecessor(threads group, key_type key)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	if (!m_head)
		return end();

	if (key < m_head->first)
		return end();
	if (key >= m_tail->first)
		return m_tail;

	auto u = binary_search(group, key);

	return get_predecessor(group, key, u.first);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::size_type XFastTrieKParallel<Key, Value, UNIVERSE>::size() const
{
	return 1 << RANK;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::iterator XFastTrieKParallel<Key, Value, UNIVERSE>::successor(threads group, key_type key)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	if (!m_tail)
		return end();

	if (key > m_tail->first)
		return end();
	if (key <= m_head->first)
		return m_head;

	auto u = binary_search(group, key);

	return get_successor(group, key, u.first);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ auto XFastTrieKParallel<Key, Value, UNIVERSE>::binary_search(threads group, key_type key) -> gpu::pair<keyset_iterator, size_type>
{
	group.sync();

	__shared__ gpu::pair<keyset_iterator, size_type> result;
	__shared__ gpu::array<keyset_iterator, SUBRANK> results_iterator;
	auto thid = group.thread_rank();
	auto warp_id = thid / 32;
	results_iterator[warp_id] = m_maps[0].end();

	/*if (group.thread_rank() == 0)
		debug();
	group.sync();*/

	cooperative_groups::thread_block_tile<32> warp = cooperative_groups::tiled_partition<32>(group);

	if (thid < 32 * SUBRANK)
	{
		int offset = 0;
		do
		{
			key_type bits = extract_i_upper_bits(key, warp_id + offset);
			auto found_it = m_maps[warp_id + offset].find(warp, bits);
			warp.sync();
			if (warp.thread_rank() == 0)
				results_iterator[warp_id + offset] = found_it;
			/*group.sync();
			if (warp.thread_rank() == 0)
				printf("%d %d %d %d %p %p ", thid, warp_id, offset, group.size(), &(*results_iterator[0]), &(*results_iterator[1]));*/
			offset += group.size() / 32;
		} while (warp_id + offset < SUBRANK);
		group.sync();

		if (warp.thread_rank() == 0)
			printf("%d %d %d %p %p ", thid, warp_id, offset, &(*results_iterator[0]), &(*results_iterator[1]));

		if (warp_id == 0)
		{
			// We try to find out the last place such that T T T F <- We want the third T.
			// So we search the first F
			auto res = gpu::find_if(warp, results_iterator.begin(), results_iterator.end(), [this, thid](const keyset_iterator& it) {
				return it == this->m_maps[thid % 32].end();
			}); // We need to find the first invalid

			//printf("%d %p ", group.thread_rank(), res);

			// If there are none -> F F F F then we return head.
			if (res == results_iterator.begin() && thid == 0)
				result = { *results_iterator.begin(), 0 };
			// Otherwise we return the previous one, as in T F, we get 1, we need to return 0
			else if (thid == 0)
			{
				//printf("%p %p ; ", results_iterator.begin(), res);
				--res;
				assert(res >= results_iterator.begin());
				result = std::move(gpu::make_pair<keyset_iterator, size_type>(*res, res - results_iterator.begin()));
			}
		}
	}
	group.sync();

	gpu::pair<keyset_iterator, size_type> copy = result;
	return copy;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieKParallel<Key, Value, UNIVERSE>::debug() const
{
	using gpu::print;
	for (int i = 0; i != SUBRANK; ++i)
	{
		print("HashMap (", i, "): ");
		const auto& map = m_maps[i];
		for (auto it = map.begin(); it != map.end(); ++it)
		{
			print("{", it->first, "|", it->second, "}");
		}
		print("\n");
	}

	print("Bottom: ");
	for (auto it = m_bottom.begin(); it != m_bottom.end(); ++it)
	{
		auto& value = it->second;
		if (value.predecessor && value.successor)
			print("{", it->first, "|", value.value, "=>[", value.predecessor->first, "|", value.successor->first, "]}");
		else if (value.predecessor)
			print("{", it->first, "|", value.value, "=>[", value.predecessor->first, "|#]}");
		else if (value.successor)
			print("{", it->first, "|", value.value, "=>[#|", value.successor->first, "]}");
		else
			print("{", it->first, "|", value.value, "=>[#|#]}");
	}

	print("\nHead/Tail: ");
	if (m_head)
		print("Head: ", m_head->second.value, " ");
	if (m_tail)
		print("Tail: ", m_tail->second.value, " ");
	print("\n");
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::key_type XFastTrieKParallel<Key, Value, UNIVERSE>::extract_i_upper_bits(key_type key, int number_of_bits) const
{
	key_type result = key >> (SUBRANK - number_of_bits);
	return result;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::iterator XFastTrieKParallel<Key, Value, UNIVERSE>::get_predecessor(threads group, key_type key, keyset_iterator u)
{
	auto& value = u->second;
	if (key < value)
	{
		auto it = m_bottom.find(group, value);
		if (it->second.predecessor)
			return it->second.predecessor;
		else
			return end();
	}
	return m_bottom.find(group, value);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieKParallel<Key, Value, UNIVERSE>::iterator XFastTrieKParallel<Key, Value, UNIVERSE>::get_successor(threads group, key_type key, keyset_iterator u)
{
	auto& value = u->second;
	auto it = m_bottom.find(group, value);
	if (it->first > key)
		return it;
	if (it->second.successor)
		return it->second.successor;
	else
		return end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieKParallel<Key, Value, UNIVERSE>::post_condition(threads group)
{
	for (int i = 0; i != SUBRANK; ++i)
	{
		auto& map = m_maps[i];
		for (auto it = map.begin(); it != map.end(); ++it)
		{
			if (i + 1 < SUBRANK)
			{
				auto left_child = m_maps[i + 1].find(group, it->first << 1);
				auto right_child = m_maps[i + 1].find(group, (it->first << 1) | 1);
				assert(left_child != m_maps[i + 1].end() || right_child != m_maps[i + 1].end());
			}
		}
	}

	if (m_head)
		assert(m_tail);
	else
		assert(!m_tail);

	if (m_head)
	{
		for (auto it = m_bottom.begin(); it != m_bottom.end(); ++it)
		{
			if (it->second.predecessor)
			{
				if (it != m_head)
					assert(it->first > it->second.predecessor->first);
				assert(it->second.predecessor->second.successor == it);
			}

			if (it->second.successor)
			{
				if (it != m_tail)
					assert(it->first < it->second.successor->first);
				assert(it->second.successor->second.predecessor == it);
			}
		}
	}
	group.sync();
}
