#include "hip/hip_runtime.h"
#include "xfasttrie-warp-parallel.cuh"

#include "algorithms/find.cuh"
#include "utility/print.cuh"

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator XFastTrieWarpParallel<Key, Value, UNIVERSE>::begin()
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::const_iterator XFastTrieWarpParallel<Key, Value, UNIVERSE>::begin() const
{
	return m_bottom.begin();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator XFastTrieWarpParallel<Key, Value, UNIVERSE>::end()
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::const_iterator XFastTrieWarpParallel<Key, Value, UNIVERSE>::end() const
{
	return m_bottom.end();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ XFastTrieWarpParallel<Key, Value, UNIVERSE>::XFastTrieWarpParallel(block_threads block, gpu::default_allocator& allocator)
{
	threads tile32 = cooperative_groups::tiled_partition<32>(block);
	auto thid = block.thread_rank();

	if (thid < tile32.size())
	{
		//unsigned int max_allocate = 4096u;
		unsigned int max_allocate = 1 << 18u;
		for (int rank = 0; rank != SUBRANK; ++rank)
		{
			unsigned int preallocate = rank < 17u ? 1u << (rank + 2u) : max_allocate;
			m_maps[rank] = Keyset{ tile32, allocator, preallocate };
		}
		m_bottom = Map{ tile32, allocator, max_allocate };

		m_head = INVALID_PREDECESSOR();
		m_tail = INVALID_SUCCESSOR();
	}
	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ XFastTrieWarpParallel<Key, Value, UNIVERSE>::XFastTrieWarpParallel(threads group, gpu::default_allocator& allocator)
{
	//unsigned int max_allocate = 4096u;
	unsigned int max_allocate = 1 << 18u;
	for (int rank = 0; rank != SUBRANK; ++rank)
	{
		unsigned int preallocate = rank < 17u ? 1u << (rank + 2u) : max_allocate;
		m_maps[rank] = Keyset{ group, allocator, preallocate };
	}
	m_bottom = Map{ group, allocator, max_allocate };

	m_head = INVALID_PREDECESSOR();
	m_tail = INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieWarpParallel<Key, Value, UNIVERSE>::clear(block_threads block)
{
	threads tile32 = cooperative_groups::tiled_partition<32>(block);

	if (block.thread_rank() < 32)
		clear(tile32);

	block.sync();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieWarpParallel<Key, Value, UNIVERSE>::clear(threads group)
{
	for (int rank = 0; rank != SUBRANK; ++rank)
		m_maps[rank].clear(group);

	m_bottom.clear(group);

	m_head = INVALID_PREDECESSOR();
	m_tail = INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator XFastTrieWarpParallel<Key, Value, UNIVERSE>::find(key_type key)
{
	return m_bottom.find(key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator XFastTrieWarpParallel<Key, Value, UNIVERSE>::find(threads group, key_type key)
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::const_iterator XFastTrieWarpParallel<Key, Value, UNIVERSE>::find(threads group, key_type key) const
{
	return m_bottom.find(group, key);
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator XFastTrieWarpParallel<Key, Value, UNIVERSE>::insert(threads group, key_type key, mapped_type value)
{
	auto it = m_bottom.find(group, key);
	if (it != m_bottom.end())
		return it;

	auto old_values = find_or_update(group, key);
	key_type old_minimal = old_values.first;
	key_type old_maximal = old_values.second;

	if (!has_data())
	{
		auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, INVALID_PREDECESSOR(), INVALID_SUCCESSOR() }));
		m_head = key;
		m_tail = key;
		post_condition(group);
		return current_it;
	}
	else
	{
		if (key < m_head)
		{
			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, INVALID_PREDECESSOR(), m_head }));
			auto it = m_bottom.find(group, m_head);
			it->second.predecessor = key;
			m_head = key;
			post_condition(group);
			return current_it;
		}
		else if (key > m_tail)
		{
			auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, m_tail, INVALID_SUCCESSOR() }));
			auto it = m_bottom.find(group, m_tail);
			it->second.successor = key;
			m_tail = key;
			post_condition(group);
			return current_it;
		}
		else
		{
			if (key < old_minimal)
			{
				auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, {}, old_minimal }));
				auto successor_it = m_bottom.find(group, old_minimal);
				current_it->second.predecessor = successor_it->second.predecessor;
				auto predecessor_it = m_bottom.find(group, successor_it->second.predecessor);

				if (group.thread_rank() == 0)
				{
					successor_it->second.predecessor = key;
					predecessor_it->second.successor = key;
				}
				group.sync();
				post_condition(group);
				return current_it;
			}
			else if (key > old_maximal)
			{
				auto current_it = m_bottom.insert(group, gpu::make_pair<key_type, Node>(key, { value, old_maximal, {} }));
				auto predecessor_it = m_bottom.find(group, old_maximal);
				current_it->second.successor = predecessor_it->second.successor;
				auto successor_it = m_bottom.find(group, predecessor_it->second.successor);

				if (group.thread_rank() == 0)
				{
					successor_it->second.predecessor = key;
					predecessor_it->second.successor = key;
				}
				group.sync();
				post_condition(group);
				return current_it;
			}
			else
			{
				/*if (group.thread_rank() == 3)
				{
					gpu::print(" > ", key, " o ", old_minimal, " p ", old_maximal, "\n");
				}
				group.sync();
				if (group.thread_rank() == 0)
				{
					gpu::print(" > ", key, " o ", old_minimal, " p ", old_maximal, "\n");
				}
				group.sync();

				ENSURE(false);*/
				return end();
			}
		}
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::size_type XFastTrieWarpParallel<Key, Value, UNIVERSE>::maximal_size() const
{
	return 1 << RANK;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator XFastTrieWarpParallel<Key, Value, UNIVERSE>::predecessor(threads group, key_type key)
{
	if (!has_data())
		return end();

	if (key < m_head)
		return end();
	if (key >= m_tail)
		return m_bottom.find(group, m_tail);

	auto u = binary_search(group, key);
	if (u.second == m_bottom.end())
	{
		if (key < u.first->second.minimal_left)
		{
			auto predecessor_it = m_bottom.find(group, u.first->second.minimal_left);
			return m_bottom.find(group, predecessor_it->second.predecessor);
		}
		else
			return m_bottom.find(group, u.first->second.maximal_right);
	}
	else
		return u.second;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::size_type XFastTrieWarpParallel<Key, Value, UNIVERSE>::size() const
{
	return m_bottom.size();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::iterator XFastTrieWarpParallel<Key, Value, UNIVERSE>::successor(threads group, key_type key)
{
	if (!has_data())
		return end();

	if (key > m_tail)
		return end();
	if (key <= m_head)
		return m_bottom.find(group, m_head);

	auto u = binary_search(group, key);
	if (u.second == m_bottom.end())
	{
		if (key < u.first->second.minimal_left)
			return m_bottom.find(group, u.first->second.minimal_left);
		else
		{
			auto predecessor_it = m_bottom.find(group, u.first->second.minimal_left);
			return m_bottom.find(group, predecessor_it->second.successor);
		}
	}
	else
		return u.second;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieWarpParallel<Key, Value, UNIVERSE>::debug() const
{
	using gpu::print;
	for (int i = 0; i != SUBRANK; ++i)
	{
		print("HashMap (", i, "): ");
		const auto& map = m_maps[i];
		for (auto it = map.begin(); it != map.end(); ++it)
		{
			print("{", it->first, "|", it->second.minimal_left, ", ", it->second.maximal_right, "}");
		}
		print("\n");
	}

	print("Bottom: ");
	for (auto it = m_bottom.begin(); it != m_bottom.end(); ++it)
	{
		auto& value = it->second;
		if (value.predecessor != INVALID_PREDECESSOR() && value.successor != INVALID_SUCCESSOR())
			print("{", it->first, "|", value.value, "=>[", value.predecessor, "|", value.successor, "]}");
		else if (value.predecessor != INVALID_PREDECESSOR())
			print("{", it->first, "|", value.value, "=>[", value.predecessor, "|#]}");
		else if (value.successor != INVALID_SUCCESSOR())
			print("{", it->first, "|", value.value, "=>[#|", value.successor, "]}");
		else
			print("{", it->first, "|", value.value, "=>[#|#]}");
	}

	print("\nHead/Tail: ");
	if (m_head != INVALID_PREDECESSOR())
		print("Head: ", m_head, " ");
	if (m_tail != INVALID_SUCCESSOR())
		print("Tail: ", m_tail, " ");
	print("\n");
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ auto XFastTrieWarpParallel<Key, Value, UNIVERSE>::binary_search(threads group, key_type key) -> gpu::pair<keyset_iterator, Map_iterator>
{
	auto thid = group.thread_rank();
	keyset_iterator found_it;
	Map_iterator bottom_it;

	unsigned int warp_offset = 0u;
	do
	{
		bool has_no_value = true;
		if (warp_offset + thid < SUBRANK)
		{
			key_type bits = extract_i_upper_bits(key, warp_offset + thid);
			found_it = m_maps[warp_offset + thid].find(bits);

			// We try to find out the last place such that T T T F <- We want the third T.
			has_no_value = found_it == m_maps[warp_offset + thid].end();
		}
		else if (warp_offset + thid == SUBRANK)
		{
			bottom_it = m_bottom.find(key);
		}

		unsigned int matching_bits = group.ballot(has_no_value);
		if (matching_bits)
		{
			// The idea is that we want the min, max of the lowest node in the tree where there is data
			unsigned int insert_update_separation = __ffs(matching_bits) - 2u;
			found_it.shfl(group, insert_update_separation);
			break;
		}
		warp_offset += group.size();
	} while (warp_offset < SUBRANK);

	bottom_it.shfl(group, SUBRANK % 32);
	return { found_it, bottom_it };
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::key_type XFastTrieWarpParallel<Key, Value, UNIVERSE>::extract_i_upper_bits(key_type key, int number_of_bits) const
{
	key_type result = key >> (SUBRANK - number_of_bits);
	return result;
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ auto XFastTrieWarpParallel<Key, Value, UNIVERSE>::find_or_update(threads group, key_type key) -> gpu::pair<key_type, key_type>
{
	auto thid = group.thread_rank();

	{
		unsigned int warp_offset = 0u;
		do
		{
			bool resize = false;
			if (warp_offset + thid < SUBRANK)
			{
				resize = m_maps[warp_offset + thid].pending_resize();
			}
			group.sync();
			unsigned int matching_bits = group.ballot(resize);
			while (matching_bits)
			{
				unsigned int rank = __ffs(matching_bits) - 1u;
				m_maps[warp_offset + rank].resize(group);
				matching_bits ^= (1 << rank); // Bits are put in little endian
			}
			warp_offset += group.size();
		} while (warp_offset < SUBRANK);
	}

	key_type old_minimum = INVALID_PREDECESSOR();
	key_type old_maximum = INVALID_SUCCESSOR();

	{
		unsigned int warp_offset = 0u;
		do
		{
			bool has_no_value = true;
			if (warp_offset + thid < SUBRANK)
			{
				key_type bits = extract_i_upper_bits(key, warp_offset + thid);
				keyset_iterator found_it = m_maps[warp_offset + thid].find(bits);

				// We try to find out the last place such that T T T F <- We want the third T.
				has_no_value = found_it == m_maps[warp_offset + thid].end();
				unsigned int matching_bits = group.ballot(has_no_value);
				unsigned int insert_update_separation = __ffs(matching_bits);

				if (!has_no_value)
				{
					old_minimum = found_it->second.minimal_left;
					old_maximum = found_it->second.maximal_right;
				}

				if (thid >= insert_update_separation - 1 && has_no_value) // Not just, should be related to group
				{
					m_maps[warp_offset + thid].insert(gpu::make_pair<key_type, Children>(bits, { key, key }));
				}
				else
				{
					found_it->second.minimal_left = min(found_it->second.minimal_left, key);
					found_it->second.maximal_right = max(found_it->second.maximal_right, key);
				}
			}

			group.sync();

			unsigned int matching_bits = group.ballot(has_no_value);
			if (matching_bits)
			{
				// The idea is that we want the min, max of the lowest node in the tree where there is data
				unsigned int insert_update_separation = __ffs(matching_bits) - 2u;
				old_minimum = group.shfl(old_minimum, insert_update_separation);
				old_maximum = group.shfl(old_maximum, insert_update_separation);
			}

			warp_offset += group.size();
		} while (warp_offset < SUBRANK);
	}

	return { old_minimum, old_maximum };
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ bool XFastTrieWarpParallel<Key, Value, UNIVERSE>::has_data() const
{
	return m_head != INVALID_PREDECESSOR() && m_tail != INVALID_SUCCESSOR();
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::key_type XFastTrieWarpParallel<Key, Value, UNIVERSE>::INVALID_PREDECESSOR() const
{
	return key_type{ -1 };
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ typename XFastTrieWarpParallel<Key, Value, UNIVERSE>::key_type XFastTrieWarpParallel<Key, Value, UNIVERSE>::INVALID_SUCCESSOR() const
{
	return key_type{ 0 };
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieWarpParallel<Key, Value, UNIVERSE>::walk_up(threads group, key_type key, size_type from)
{
	while (from != 0)
	{
		--from;
		key_type bits = extract_i_upper_bits(key, from);
		auto it = m_maps[from].find(group, bits);
		it->second.minimal_left = min(it->second.minimal_left, key);
		it->second.maximal_right = max(it->second.maximal_right, key);
		if (it->second.minimal_left != key && it->second.maximal_right != key)
			return;
	}
}

template <typename Key, typename Value, std::size_t UNIVERSE>
__device__ void XFastTrieWarpParallel<Key, Value, UNIVERSE>::post_condition(threads group)
{
#ifdef GPU_XFASTTRIE_DEBUG
	for (int i = 0; i != SUBRANK; ++i)
	{
		auto& map = m_maps[i];
		for (auto it = map.begin(); it != map.end(); ++it)
		{
			if (i + 1 < SUBRANK)
			{
				auto left_child = m_maps[i + 1].find(group, it->first << 1);
				auto right_child = m_maps[i + 1].find(group, (it->first << 1) | 1);
				ENSURE(left_child != m_maps[i + 1].end() || right_child != m_maps[i + 1].end());
			}
		}
	}

	if (m_head != INVALID_PREDECESSOR())
	{
		ENSURE(m_head == m_tail || m_tail != INVALID_SUCCESSOR());
		auto buffer_start = &(*m_bottom.end()) - m_bottom.capacity();
		auto head = m_bottom.find(group, m_head);
		auto tail = m_bottom.find(group, m_tail);
		ENSURE(&(*head) >= buffer_start && &(*head) < &(*m_bottom.end()));
		ENSURE(&(*tail) >= buffer_start && &(*tail) < &(*m_bottom.end()));
	}
	else
		ENSURE(m_tail == INVALID_SUCCESSOR());

	if (has_data())
	{
		for (auto it = m_bottom.begin(); it != m_bottom.end(); ++it)
		{
			if (it->first != m_head)
			{
				auto predecessor_it = m_bottom.find(group, it->second.predecessor);
				ENSURE(it->first > predecessor_it->first);
				ENSURE(it->first == predecessor_it->second.successor);
			}
			if (it->first != m_tail)
			{
				auto successor_it = m_bottom.find(group, it->second.successor);
				ENSURE(it->first < successor_it->first);
				ENSURE(it->first == successor_it->second.predecessor);
			}
		}
	}
#endif // GPU_XFASTTRIE_DEBUG
}
