
#include <hip/hip_runtime.h>
//#include "concurrent-xfasttrie-binary.cuh"
//#include "Catch2/catch.hpp"
//#include "cuda/api_wrappers.h"
//
//#include "concurrent-xfasttrie-common.cuh"
//
//using key_type = unsigned int;
//using mapped_type = int;
//using XFastTrie = ConcurrentXFastTrieBinary<key_type, mapped_type, 3>;
//
//SCENARIO("CONCURRENT-X-FAST-TRIE-BINARY", "[XFASTTRIE][CONCURRENTBINARY]")
//{
//	unsigned int NUMBER_OF_WARPS = 2u;
//	int memory_size_allocated = 1u << 29u;
//	unsigned int to_insert = 1u << 5u;
//	auto current_device = cuda::device::current::get();
//	auto d_memory = cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated);
//	auto d_allocator = cuda::memory::device::make_unique<allocator_type>(current_device);
//
//	GIVEN("A XFastTrie")
//	{
//		auto d_xfasttrie = cuda::memory::device::make_unique<XFastTrie>(current_device);
//
//		cuda::launch(initialize_allocator<XFastTrie>,
//			{ 1u, NUMBER_OF_WARPS * 32u },
//			d_allocator.get(), d_memory.get(), memory_size_allocated, d_xfasttrie.get(), to_insert
//		);
//
//		WHEN("We add elements in increasing order")
//		{
//			THEN("It should be good")
//			{
//				cuda::launch(test_insert_increasing_order<XFastTrie>,
//					{ 1u, NUMBER_OF_WARPS * 32u },
//					d_xfasttrie.get()
//				);
//			}
//		}
//
//		WHEN("We add elements with dulpicates")
//		{
//			THEN("It should be good")
//			{
//				cuda::launch(test_insert_with_duplicates<XFastTrie>,
//				{ 1u, NUMBER_OF_WARPS * 32u },
//					d_xfasttrie.get()
//				);
//			}
//		}
//
//		WHEN("We add elements in random order")
//		{
//			THEN("It should be good")
//			{
//				cuda::launch(test_insert_random<XFastTrie>,
//				{ 1u, NUMBER_OF_WARPS * 32u },
//					d_xfasttrie.get()
//				);
//			}
//		}
//	}
//}
