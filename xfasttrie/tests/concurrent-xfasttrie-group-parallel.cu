#include "hip/hip_runtime.h"
#include "concurrent-xfasttrie-group-parallel.cuh"
#include "Catch2/catch.hpp"
#include "cuda/api_wrappers.h"

#include "concurrent-xfasttrie-common.cuh"

#include <ctime>
#include <iostream>

using key_type = gpu::UInt32;
using mapped_type = int;
using XFastTrie = ConcurrentXFastTrieGroupParallel<key_type, mapped_type, 32, 2>;

SCENARIO("CONCURRENT-X-FAST-TRIE-GROUP-PARALLEL", "[XFASTTRIE][CONCURRENTGROUPPARALLEL]")
{
	unsigned int NUMBER_OF_BLOCKS = 32u;
	unsigned int NUMBER_OF_WARPS = 16u;
	const unsigned int memory_size_allocated = 1u << 31u;
	unsigned int to_insert = 1u << 18u;
	auto current_device = cuda::device::current::get();
	auto d_memory = cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated);
	auto d_allocator = cuda::memory::device::make_unique<allocator_type>(current_device);

	GIVEN("A XFastTrie")
	{
		auto d_xfasttrie = cuda::memory::device::make_unique<XFastTrie>(current_device);

		cuda::launch(initialize_allocator<XFastTrie>,
			{ 1u, NUMBER_OF_WARPS * 32u },
			d_allocator.get(), d_memory.get(), memory_size_allocated, d_xfasttrie.get(), to_insert
		);

		/*WHEN("We add elements in increasing order")
		{
			THEN("It should be good")
			{
				cuda::launch(test_insert_increasing_order<XFastTrie>,
					{ 1u, NUMBER_OF_WARPS * 32u },
					d_xfasttrie.get()
				);
			}
		}

		WHEN("We add elements with dulpicates")
		{
			THEN("It should be good")
			{
				cuda::launch(test_insert_with_duplicates<XFastTrie>,
				{ 1u, NUMBER_OF_WARPS * 32u },
					d_xfasttrie.get()
				);
			}
		}*/

		WHEN("We add elements in random order")
		{
			THEN("It should be good")
			{
				std::cout << to_insert << std::endl;
				const std::clock_t begin_time = std::clock();
				cuda::launch(test_insert_random<XFastTrie>,
				{ NUMBER_OF_BLOCKS * 1u, NUMBER_OF_WARPS * 32u },
					d_xfasttrie.get(), to_insert
				);
				cuda::device::current::get().synchronize();
				std::cout << float(std::clock() - begin_time) / CLOCKS_PER_SEC;
				fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(hipPeekAtLastError()), __FILE__, __LINE__);
				cuda::device::current::get().synchronize();
				cuda::launch(test_retrieve_size<XFastTrie>,
				{ 1u, 1u },
					d_xfasttrie.get(), to_insert
				);
				cuda::device::current::get().synchronize();
			}
		}
	}
}
