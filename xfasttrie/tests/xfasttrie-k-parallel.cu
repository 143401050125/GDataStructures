
#include <hip/hip_runtime.h>
//#include "xfasttrie-k-parallel.cuh"
//#include "Catch2/catch.hpp"
//#include "cuda/api_wrappers.h"
//
//#include "allocators/default_allocator.cuh"
//#include <cassert>
//#include <cooperative_groups.h>
//
//using XTrie = XFastTrieKParallel<unsigned char, int>;
//using XTrieKey = typename XTrie::key_type;
//using XTrie3 = XFastTrieKParallel<unsigned char, int, 3>;
//using XTrie3Key = typename XTrie3::key_type;
//using BigXTrie = XFastTrieKParallel<int, int>;
//using BigXTrieKey = typename BigXTrie::key_type;
//
//__global__ void XFastTrieKParallel_initialize_allocator_small(gpu::default_allocator* allocator, char* memory, int memory_size, XTrie3* xtrie)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//	if (block.thread_rank() == 0)
//		new (allocator) gpu::default_allocator(memory, memory_size);
//	block.sync();
//	new (xtrie) XTrie3(block, *allocator);
//}
//
//__global__ void XFastTrieKParallel_initialize_allocator(gpu::default_allocator* allocator, char* memory, int memory_size, XTrie* xtrie)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//	if (block.thread_rank() == 0)
//		new (allocator) gpu::default_allocator(memory, memory_size);
//	block.sync();
//	new (xtrie) XTrie(block, *allocator);
//}
//
//__global__ void XFastTrieKParallel_initialize_allocator_big(gpu::default_allocator* allocator, char* memory, int memory_size, BigXTrie* xtrie)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//	if (block.thread_rank() == 0)
//		new (allocator) gpu::default_allocator(memory, memory_size);
//	block.sync();
//	new (xtrie) BigXTrie(block, *allocator);
//}
//
//template <typename Key, typename Value, std::size_t Universe>
//__device__ void XFastTrieKParallel_ensure_value(const XFastTrieKParallel<Key, Value, Universe>& trie, typename XFastTrieKParallel<Key, Value, Universe>::iterator it, int expected_value)
//{
//	assert(it != trie.end());
//	assert(it->second == expected_value);
//}
//
//__global__ void XFastTrieKParallel_test_insert_find_2(XTrie* triePtr)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//
//	XTrie& trie = *triePtr;
//	auto convert = [](int value) -> XTrieKey { return value;  };
//	trie.insert(block, convert(3), 3);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(3)), 3);
//	trie.insert(block, convert(1), 1);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(1)), 1);
//	trie.insert(block, convert(6), 6);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(6)), 6);
//	trie.insert(block, convert(5), 5);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(5)), 5);
//}
//
//__global__ void XFastTrieKParallel_test_insert_find(XTrie* triePtr)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//
//	XTrie& trie = *triePtr;
//	auto convert = [](int value) -> XTrieKey { return value;  };
//	trie.insert(block, convert(255), 255);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(255)), 255);
//	trie.insert(block, convert(13), 13);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(13)), 13);
//	trie.insert(block, convert(251), 251);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(251)), 251);
//	trie.insert(block, convert(15), 15);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(15)), 15);
//}
//
//__global__ void XFastTrieKParallel_test_insert_find_small(XTrie3* triePtr)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//
//	XTrie3& trie = *triePtr;
//	auto convert = [](int value) -> XTrie3Key { return value;  };
//	trie.insert(block, convert(3), 3);
//	if (block.thread_rank() == 0)
//		trie.debug();
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(3)), 3);
//	trie.insert(block, convert(1), 1);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(1)), 1);
//	trie.insert(block, convert(6), 6);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(6)), 6);
//	printf("==");
//	trie.insert(block, convert(5), 5);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(5)), 5);
//	trie.insert(block, convert(7), 7);
//	auto it = trie.find(block, convert(7));
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(7)), 7);
//	trie.insert(block, convert(4), 4);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(4)), 4);
//	trie.insert(block, convert(0), 0);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(0)), 0);
//}
//
//__global__ void XFastTrieKParallel_test_insert_find_small_2(XTrie3* triePtr)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//
//	XTrie3& trie = *triePtr;
//	auto convert = [](int value) -> XTrie3Key { return value;  };
//	trie.insert(block, convert(7), 7);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(7)), 7);
//	trie.insert(block, convert(0), 0);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(0)), 0);
//	trie.insert(block, convert(3), 3);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(3)), 3);
//	trie.insert(block, convert(5), 5);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(5)), 5);
//	trie.insert(block, convert(7), 7);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(7)), 7);
//	trie.insert(block, convert(4), 4);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(4)), 4);
//	trie.insert(block, convert(0), 0);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(0)), 0);
//}
//
//__global__ void XFastTrieKParallel_test_insert_find_small_increasing_order(XTrie3* triePtr)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//
//	XTrie3& trie = *triePtr;
//	auto convert = [](int value) -> XTrie3Key { return value;  };
//	for (int i = 0; i != trie.size(); ++i)
//	{
//		trie.insert(block, convert(i), i);
//		XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(i)), i);
//	}
//}
//
//__global__ void XFastTrieKParallel_test_insert_find_small_decreasing_order(XTrie3* triePtr)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//
//	XTrie3& trie = *triePtr;
//	auto convert = [](int value) -> XTrie3Key { return value;  };
//	for (int i = trie.size() - 1; i != 0; --i)
//	{
//		trie.insert(block, convert(i), i);
//		XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(i)), i);
//	}
//}
//
//__global__ void XFastTrieKParallel_test_predecessor_successor_small(XTrie3* triePtr)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//
//	XTrie3& trie = *triePtr;
//	auto convert = [](int value) -> XTrie3Key { return value;  };
//	assert(trie.predecessor(block, convert(4)) == trie.end());
//	assert(trie.successor(block, convert(4)) == trie.end());
//
//	trie.insert(block, convert(2), 2);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(3)), 2);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(4)), 2);
//	XFastTrieKParallel_ensure_value(trie, trie.successor(block, convert(1)), 2);
//	assert(trie.predecessor(block, convert(1)) == trie.end());
//	assert(trie.successor(block, convert(3)) == trie.end());
//
//	trie.insert(block, convert(3), 3);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(3)), 3);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(4)), 3);
//	XFastTrieKParallel_ensure_value(trie, trie.successor(block, convert(2)), 2);
//}
//
//__global__ void XFastTrieKParallel_test_insert_find_big(BigXTrie* triePtr)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//
//	BigXTrie& trie = *triePtr;
//	auto convert = [](int value) -> BigXTrieKey { return value;  };
//	trie.insert(block, convert(3), 3);
//	/*if (block.thread_rank() == 0)
//		trie.debug();*/
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(3)), 3);
//	trie.insert(block, convert(1), 1);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(1)), 1);
//	trie.insert(block, convert(6), 6);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(6)), 6);
//	trie.insert(block, convert(5), 5);
//	XFastTrieKParallel_ensure_value(trie, trie.find(block, convert(5)), 5);
//}
//
//__global__ void XFastTrieKParallel_test_predecessor_successor(XTrie* triePtr)
//{
//	cooperative_groups::thread_block block = cooperative_groups::this_thread_block();
//
//	XTrie& trie = *triePtr;
//	auto convert = [](int value) -> XTrieKey { return value;  };
//	assert(trie.predecessor(block, convert(128)) == trie.end());
//	assert(trie.successor(block, convert(128)) == trie.end());
//
//	trie.insert(block, convert(2), 2);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(128)), 2);
//	XFastTrieKParallel_ensure_value(trie, trie.successor(block, convert(1)), 2);
//	assert(trie.predecessor(block, convert(1)) == trie.end());
//	assert(trie.successor(block, convert(3)) == trie.end());
//
//	trie.insert(block, convert(13), 13);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(128)), 13);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(13)), 13);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(12)), 2);
//	XFastTrieKParallel_ensure_value(trie, trie.successor(block, convert(3)), 13);
//	assert(trie.successor(block, convert(128)) == trie.end());
//
//	trie.insert(block, convert(251), 251);
//	if (block.thread_rank() == 0)
//		trie.debug();
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(128)), 13);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(253)), 251);
//	XFastTrieKParallel_ensure_value(trie, trie.successor(block, convert(1)), 2);
//	assert(trie.predecessor(block, convert(1)) == trie.end());
//	XFastTrieKParallel_ensure_value(trie, trie.successor(block, convert(3)), 13);
//	XFastTrieKParallel_ensure_value(trie, trie.successor(block, convert(128)), 251);
//	XFastTrieKParallel_ensure_value(trie, trie.successor(block, convert(248)), 251);
//	assert(trie.successor(block, convert(252)) == trie.end());
//
//	trie.insert(block, convert(190), 190);
//	if (block.thread_rank() == 0)
//		trie.debug();
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(189)), 13);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(190)), 190);
//	XFastTrieKParallel_ensure_value(trie, trie.predecessor(block, convert(250)), 190);
//	XFastTrieKParallel_ensure_value(trie, trie.successor(block, convert(191)), 251);
//}
//
//SCENARIO("X-FAST-TRIE-K-PARALLEL", "[XFASTTRIE][KPARALLEL]")
//{
//	int memory_size_allocated = 4 * 1024 * 1024;
//	auto current_device = cuda::device::current::get();
//	auto d_memory = cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated);
//	auto d_allocator = cuda::memory::device::make_unique<gpu::default_allocator>(current_device);
//	unsigned int number_of_warps = 2u;
//
//	GIVEN("A X-fast trie for 2^3")
//	{
//		auto d_xtrie3 = cuda::memory::device::make_unique<XTrie3>(current_device);
//		cuda::launch(
//			XFastTrieKParallel_initialize_allocator_small,
//			{ 1u, 1u },
//			d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie3.get()
//		);
//
//		WHEN("We add different values")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					XFastTrieKParallel_test_insert_find_small,
//					{ 1u, number_of_warps * 32u },
//					d_xtrie3.get()
//				);
//			}
//		}
//
//
//		WHEN("We try again")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					XFastTrieKParallel_test_insert_find_small_2,
//					{ 1u, number_of_warps * 32u },
//					d_xtrie3.get()
//				);
//			}
//		}
//
//		WHEN("We try again in increasing order")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					XFastTrieKParallel_test_insert_find_small_increasing_order,
//					{ 1u, number_of_warps * 32u },
//					d_xtrie3.get()
//				);
//			}
//		}
//
//		WHEN("We try again in decreasing order")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					XFastTrieKParallel_test_insert_find_small_decreasing_order,
//					{ 1u, number_of_warps * 32u },
//					d_xtrie3.get()
//				);
//			}
//		}
//
//		WHEN("We add different values")
//		{
//			THEN("Predecessor and successor should be conformed")
//			{
//				cuda::launch(
//					XFastTrieKParallel_test_predecessor_successor_small,
//					{ 1u, number_of_warps * 32u },
//					d_xtrie3.get()
//				);
//			}
//		}
//	}
//
//	GIVEN("A X-fast trie for 2^8")
//	{
//		auto d_xtrie = cuda::memory::device::make_unique<XTrie>(current_device);
//		cuda::launch(
//			XFastTrieKParallel_initialize_allocator,
//			{ 1u, number_of_warps * 32u },
//			d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie.get()
//		);
//
//		WHEN("We add different values")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					XFastTrieKParallel_test_insert_find,
//					{ 1u, number_of_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We add different values")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					XFastTrieKParallel_test_insert_find_2,
//					{ 1u, number_of_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We add different values")
//		{
//			THEN("Predecessor and successor should be conformed")
//			{
//				cuda::launch(
//					XFastTrieKParallel_test_predecessor_successor,
//					{ 1u, number_of_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//	}
//
//	GIVEN("A X-fast trie for 2^32")
//	{
//		auto d_xtrie = cuda::memory::device::make_unique<BigXTrie>(current_device);
//		cuda::launch(
//			XFastTrieKParallel_initialize_allocator_big,
//			{ 1u, number_of_warps * 32u },
//			d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie.get()
//		);
//
//		WHEN("We add different values")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					XFastTrieKParallel_test_insert_find_big,
//					{ 1u, number_of_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//	}
//}
