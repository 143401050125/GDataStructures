
#include <hip/hip_runtime.h>
//#include "xfasttrie-warp-parallel.cuh"
//#include "Catch2/catch.hpp"
//#include "cuda/api_wrappers.h"
//
//#include "xfasttrie-common.cuh"
//
//using XTrie = XFastTrieWarpParallel<unsigned char, int>;
//using XTrieKey = typename XTrie::key_type;
//using XTrie3 = XFastTrieWarpParallel<unsigned char, int, 3>;
//using XTrie3Key = typename XTrie3::key_type;
//using MediumXTrie = XFastTrieWarpParallel<int, int, 16>;
//using MediumXTrieKey = typename MediumXTrie::key_type;
//using BigXTrie = XFastTrieWarpParallel<int, int>;
//using BigXTrieKey = typename BigXTrie::key_type;
//using HugeXTrie = XFastTrieWarpParallel<gpu::UInt64, int>;
//using HugeXTrieKey = typename HugeXTrie::key_type;
//
//SCENARIO("X-FAST-TRIE-WARP-PARALLEL", "[XFASTTRIE][WARPPARALLEL]")
//{
//	int memory_size_allocated = 1024u * 1024 * 1024;
//	auto current_device = cuda::device::current::get();
//	auto d_memory = cuda::memory::device::make_unique<char[]>(current_device, memory_size_allocated);
//	auto d_allocator = cuda::memory::device::make_unique<gpu::default_allocator>(current_device);
//	unsigned int number_warps = 1u;
//
//	GIVEN("A X-fast trie for 2^3")
//	{
//		auto d_xtrie3 = cuda::memory::device::make_unique<XTrie3>(current_device);
//		cuda::launch(
//			initialize_allocator<XTrie3>,
//			{ 1u, number_warps * 32u },
//			d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie3.get()
//		);
//
//		WHEN("We add different values")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_find<XTrie3>,
//					{ 1u, number_warps * 32u },
//					d_xtrie3.get()
//				);
//			}
//		}
//
//		WHEN("We try in increasing order")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_increasing_order<XTrie3>,
//					{ 1u, number_warps * 32u },
//					d_xtrie3.get()
//				);
//			}
//		}
//
//		WHEN("We try again in decreasing order")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_decreasing_order<XTrie3>,
//					{ 1u, number_warps * 32u },
//					d_xtrie3.get()
//				);
//			}
//		}
//
//		WHEN("We add different values")
//		{
//			THEN("Predecessor and successor should be conformed")
//			{
//				cuda::launch(
//					test_predecessor_successor<XTrie3>,
//					{ 1u, number_warps * 32u },
//					d_xtrie3.get()
//				);
//			}
//		}
//	}
//
//	GIVEN("A X-fast trie for 2^8")
//	{
//		auto d_xtrie = cuda::memory::device::make_unique<XTrie>(current_device);
//		cuda::launch(
//			initialize_allocator<XTrie>,
//			{ 1u, number_warps * 32u },
//			d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie.get()
//		);
//
//		WHEN("We add different values")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_find<XTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We try in increasing order")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_increasing_order<XTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We try again in decreasing order")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_decreasing_order<XTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We add different values")
//		{
//			THEN("Predecessor and successor should be conformed")
//			{
//				cuda::launch(
//					test_predecessor_successor<XTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We add random values")
//		{
//			THEN("It should be ok")
//			{
//				cuda::launch(
//					test_random<XTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get(), 10
//				);
//			}
//		}
//	}
//
//	GIVEN("A X-fast trie for 2^32")
//	{
//		auto d_xtrie = cuda::memory::device::make_unique<BigXTrie>(current_device);
//		cuda::launch(
//			initialize_allocator<BigXTrie>,
//			{ 1u, number_warps * 32u },
//			d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie.get()
//		);
//
//		WHEN("We add different values")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_find<BigXTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We try in increasing order")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_increasing_order<BigXTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We try again in decreasing order")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_decreasing_order<BigXTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We add different values")
//		{
//			THEN("Predecessor and successor should be conformed")
//			{
//				cuda::launch(
//					test_predecessor_successor<BigXTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We add random values")
//		{
//			THEN("It should be ok")
//			{
//				cuda::launch(
//					test_random<BigXTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get(), 10
//				);
//			}
//		}
//	}
//
//	GIVEN("A X-fast trie for 2^64")
//	{
//		auto d_xtrie = cuda::memory::device::make_unique<HugeXTrie>(current_device);
//		cuda::launch(
//			initialize_allocator<HugeXTrie>,
//			{ 1u, number_warps * 32u },
//			d_allocator.get(), d_memory.get(), memory_size_allocated, d_xtrie.get()
//		);
//
//		WHEN("We add different values")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_find<HugeXTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We try in increasing order")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_increasing_order<HugeXTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We try again in decreasing order")
//		{
//			THEN("We should be able to retrieve them")
//			{
//				cuda::launch(
//					test_insert_decreasing_order<HugeXTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We add different values")
//		{
//			THEN("Predecessor and successor should be conformed")
//			{
//				cuda::launch(
//					test_predecessor_successor<HugeXTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get()
//				);
//			}
//		}
//
//		WHEN("We add random values")
//		{
//			THEN("It should be ok")
//			{
//				cuda::launch(
//					test_random<HugeXTrie>,
//					{ 1u, number_warps * 32u },
//					d_xtrie.get(), 10
//				);
//			}
//		}
//	}
//}
